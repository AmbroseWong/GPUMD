#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class defining the simulation box.
------------------------------------------------------------------------------*/

#include "box.cuh"
#include "utilities/error.cuh"
#include <cmath>

double Box::get_volume(void)
{
  double volume;
  if (triclinic) {
    volume = abs(
      cpu_h[0] * (cpu_h[4] * cpu_h[8] - cpu_h[5] * cpu_h[7]) +
      cpu_h[1] * (cpu_h[5] * cpu_h[6] - cpu_h[3] * cpu_h[8]) +
      cpu_h[2] * (cpu_h[3] * cpu_h[7] - cpu_h[4] * cpu_h[6]));
  } else {
    volume = cpu_h[0] * cpu_h[1] * cpu_h[2];
  }
  return volume;
}

void Box::get_inverse(void)
{
  cpu_h[9] = cpu_h[4] * cpu_h[8] - cpu_h[5] * cpu_h[7];
  cpu_h[10] = cpu_h[2] * cpu_h[7] - cpu_h[1] * cpu_h[8];
  cpu_h[11] = cpu_h[1] * cpu_h[5] - cpu_h[2] * cpu_h[4];
  cpu_h[12] = cpu_h[5] * cpu_h[6] - cpu_h[3] * cpu_h[8];
  cpu_h[13] = cpu_h[0] * cpu_h[8] - cpu_h[2] * cpu_h[6];
  cpu_h[14] = cpu_h[2] * cpu_h[3] - cpu_h[0] * cpu_h[5];
  cpu_h[15] = cpu_h[3] * cpu_h[7] - cpu_h[4] * cpu_h[6];
  cpu_h[16] = cpu_h[1] * cpu_h[6] - cpu_h[0] * cpu_h[7];
  cpu_h[17] = cpu_h[0] * cpu_h[4] - cpu_h[1] * cpu_h[3];
  double det;
  if (triclinic) {
    det = cpu_h[0] * (cpu_h[4] * cpu_h[8] - cpu_h[5] * cpu_h[7]) +
          cpu_h[1] * (cpu_h[5] * cpu_h[6] - cpu_h[3] * cpu_h[8]) +
          cpu_h[2] * (cpu_h[3] * cpu_h[7] - cpu_h[4] * cpu_h[6]);
  } else {
    det = cpu_h[0] * cpu_h[1] * cpu_h[2];
  }
  for (int n = 9; n < 18; n++) {
    cpu_h[n] /= det;
  }
}

void static get_num_bins_one_direction(
  const int pbc, const double rc, const double box_length, int& num_bins, bool& use_ON2)
{
  if (pbc) {
    num_bins = floor(box_length / rc);
    if (num_bins < 3) {
      use_ON2 = true;
    }
  } else {
    num_bins = 1;
  }
}

bool Box::get_num_bins(const double rc, int num_bins[]) const
{
  bool use_ON2 = false;
  if (triclinic) {
    use_ON2 = true;
  } else {
    get_num_bins_one_direction(pbc_x, rc, cpu_h[0], num_bins[0], use_ON2);
    get_num_bins_one_direction(pbc_y, rc, cpu_h[1], num_bins[1], use_ON2);
    get_num_bins_one_direction(pbc_z, rc, cpu_h[2], num_bins[2], use_ON2);
  }
  if (num_bins[0] * num_bins[1] * num_bins[2] < 50) {
    use_ON2 = true;
  }
  return use_ON2;
}
