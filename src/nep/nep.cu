#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The neuroevolution potential (NEP)
------------------------------------------------------------------------------*/

#include "error.cuh"
#include "gpu_vector.cuh"
#include "mic.cuh"
#include "neighbor.cuh"
#include "nep.cuh"

NEP::NEP(
  int num_neurons_2b,
  float r1_2b,
  float r2_2b,
  int num_neurons_3b,
  float r1_3b,
  float r2_3b,
  int num_neurons_mb,
  int n_max,
  int L_max)
{
  // 2body
  ann2b.dim = 1;
  ann2b.num_neurons_per_layer = num_neurons_2b;
  para2b.r1 = r1_2b;
  para2b.r2 = r2_2b;
  para2b.r2inv = 1.0f / para2b.r2;
  para2b.pi_factor = 3.1415927f / (r2_2b - r1_2b);
  // 3body
  ann3b.dim = 3;
  ann3b.num_neurons_per_layer = num_neurons_3b;
  para3b.r1 = r1_3b;
  para3b.r2 = r2_3b;
  para3b.r2inv = 1.0f / para3b.r2;
  para3b.pi_factor = 3.1415927f / (r2_3b - r1_3b);
  // manybody
  paramb.n_max = n_max;
  paramb.L_max = L_max;
  paramb.r1 = r1_2b; // manybody has the same cutoff as twobody
  paramb.r2 = r2_2b; // manybody has the same cutoff as twobody
  paramb.r2inv = 1.0f / paramb.r2;
  paramb.pi_factor = 3.1415927f / (paramb.r2 - paramb.r1);
  annmb.dim = (n_max + 1) * (L_max + 1);
  annmb.num_neurons_per_layer = num_neurons_mb;
};

void NEP::initialize(int N, int MAX_ATOM_NUMBER)
{
  if (ann3b.num_neurons_per_layer > 0) {
    nep_data.f12x3b.resize(N * MAX_ATOM_NUMBER);
    nep_data.f12y3b.resize(N * MAX_ATOM_NUMBER);
    nep_data.f12z3b.resize(N * MAX_ATOM_NUMBER);
    nep_data.NN3b.resize(N);
    nep_data.NL3b.resize(N * MAX_ATOM_NUMBER);
  }
  if (annmb.num_neurons_per_layer > 0) {
    nep_data.Fp.resize(N * annmb.dim);
  }
}

void NEP::update_potential(const float* parameters)
{
  int offset = 0;
  if (ann2b.num_neurons_per_layer > 0) {
    update_potential(parameters, offset, ann2b);
  }
  if (ann3b.num_neurons_per_layer > 0) {
    if (ann2b.num_neurons_per_layer > 0) {
      offset = ann2b.num_neurons_per_layer * (ann2b.num_neurons_per_layer + 4) + 1;
    }
    update_potential(parameters, offset, ann3b);
  }
}

void NEP::update_potential(const float* parameters, const int offset, NEP::ANN& ann)
{
  for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
    for (int d = 0; d < ann.dim; ++d) {
      ann.w0[n * ann.dim + d] = parameters[n * ann.dim + d + offset];
    }
    ann.b0[n] = parameters[n + ann.num_neurons_per_layer * ann.dim + offset];
    for (int m = 0; m < ann.num_neurons_per_layer; ++m) {
      int nm = n * ann.num_neurons_per_layer + m;
      ann.w1[nm] = parameters[nm + ann.num_neurons_per_layer * (ann.dim + 1) + offset];
    }
    ann.b1[n] = parameters
      [n + ann.num_neurons_per_layer * (ann.num_neurons_per_layer + (ann.dim + 1)) + offset];
    ann.w2[n] = parameters
      [n + ann.num_neurons_per_layer * (ann.num_neurons_per_layer + (ann.dim + 2)) + offset];
  }
  ann.b2 =
    parameters[ann.num_neurons_per_layer * (ann.num_neurons_per_layer + (ann.dim + 3)) + offset];
}

static __device__ void apply_ann(const NEP::ANN& ann, float* q, float& p123, float* f123)
{
  // energy
  float x1[10] = {0.0f}; // states of the 1st hidden layer neurons
  float x2[10] = {0.0f}; // states of the 2nd hidden layer neurons
  for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
    float w0_times_q = 0.0f;
    for (int d = 0; d < ann.dim; ++d) {
      w0_times_q += ann.w0[n * ann.dim + d] * q[d];
    }
    x1[n] = tanh(w0_times_q - ann.b0[n]);
  }
  for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
    for (int m = 0; m < ann.num_neurons_per_layer; ++m) {
      x2[n] += ann.w1[n * ann.num_neurons_per_layer + m] * x1[m];
    }
    x2[n] = tanh(x2[n] - ann.b1[n]);
  }
  for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
    p123 += ann.w2[n] * x2[n];
  }
  p123 -= ann.b2;

  // energy gradient (compute it component by component)
  for (int d = 0; d < ann.dim; ++d) {
    float y1[10] = {0.0f}; // derivatives of the states of the 1st hidden layer neurons
    float y2[10] = {0.0f}; // derivatives of the states of the 2nd hidden layer neurons
    for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
      y1[n] = (1.0f - x1[n] * x1[n]) * ann.w0[n * ann.dim + d];
    }
    for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
      for (int m = 0; m < ann.num_neurons_per_layer; ++m) {
        y2[n] += ann.w1[n * ann.num_neurons_per_layer + m] * y1[m];
      }
      y2[n] *= 1.0f - x2[n] * x2[n];
    }
    for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
      f123[d] += ann.w2[n] * y2[n];
    }
  }
}

static __device__ void find_fc(float r1, float r2, float pi_factor, float d12, float& fc)
{
  if (d12 < r1) {
    fc = 1.0f;
  } else if (d12 < r2) {
    fc = 0.5f * cos(pi_factor * (d12 - r1)) + 0.5f;
  } else {
    fc = 0.0f;
  }
}

static __device__ void
find_fc_and_fcp(float r1, float r2, float pi_factor, float d12, float& fc, float& fcp)
{
  if (d12 < r1) {
    fc = 1.0f;
    fcp = 0.0f;
  } else if (d12 < r2) {
    fc = 0.5f * cos(pi_factor * (d12 - r1)) + 0.5f;
    fcp = -sin(pi_factor * (d12 - r1)) * pi_factor * 0.5f;
  } else {
    fc = 0.0f;
    fcp = 0.0f;
  }
}

static __global__ void find_force_2body(
  int number_of_particles,
  int* Na,
  int* Na_sum,
  int* g_NN2b,
  int* g_NL2b,
  int* g_type,
  NEP::Para2B para2b,
  NEP::ANN ann2b,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_virial,
  float* g_pe)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = g_box + 18 * blockIdx.x;
    int neighbor_number = g_NN2b[n1];

    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];

    float pe = 0.0f;
    float fx = 0.0f;
    float fy = 0.0f;
    float fz = 0.0f;
    float virial_xx = 0.0f;
    float virial_yy = 0.0f;
    float virial_zz = 0.0f;
    float virial_xy = 0.0f;
    float virial_yz = 0.0f;
    float virial_zx = 0.0f;

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_NL2b[n1 + number_of_particles * i1];

      float x12 = g_x[n2] - x1;
      float y12 = g_y[n2] - y1;
      float z12 = g_z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);

      float p2 = 0.0f, f2[1] = {0.0f};
      float q[1] = {d12 / para2b.r2};
      apply_ann(ann2b, q, p2, f2);
      f2[0] /= para2b.r2;
      float fc, fcp;
      find_fc_and_fcp(para2b.r1, para2b.r2, para2b.pi_factor, d12, fc, fcp);
      p2 *= fc;
      f2[0] = (f2[0] * fc + p2 * fcp) / d12;

      fx += x12 * f2[0];
      fy += y12 * f2[0];
      fz += z12 * f2[0];
      virial_xx -= x12 * x12 * f2[0] * 0.5f;
      virial_yy -= y12 * y12 * f2[0] * 0.5f;
      virial_zz -= z12 * z12 * f2[0] * 0.5f;
      virial_xy -= x12 * y12 * f2[0] * 0.5f;
      virial_yz -= y12 * z12 * f2[0] * 0.5f;
      virial_zx -= z12 * x12 * f2[0] * 0.5f;
      pe += p2 * 0.5f;
    }

    g_fx[n1] = fx;
    g_fy[n1] = fy;
    g_fz[n1] = fz;
    g_virial[n1 + number_of_particles * 0] = virial_xx;
    g_virial[n1 + number_of_particles * 1] = virial_yy;
    g_virial[n1 + number_of_particles * 2] = virial_zz;
    g_virial[n1 + number_of_particles * 3] = virial_xy;
    g_virial[n1 + number_of_particles * 4] = virial_yz;
    g_virial[n1 + number_of_particles * 5] = virial_zx;
    g_pe[n1] = pe;
  }
}

static __global__ void find_neighbor_list_3body(
  int number_of_particles,
  int* Na,
  int* Na_sum,
  int* g_NN2b,
  int* g_NL2b,
  NEP::Para3B para3b,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  int* g_NN3b,
  int* g_NL3b)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = g_box + 18 * blockIdx.x;
    int neighbor_number = g_NN2b[n1];

    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];

    int count = 0;

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_NL2b[n1 + number_of_particles * i1];

      float x12 = g_x[n2] - x1;
      float y12 = g_y[n2] - y1;
      float z12 = g_z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);
      float d12sq = x12 * x12 + y12 * y12 + z12 * z12;

      if (d12sq < para3b.r2 * para3b.r2) {
        g_NL3b[n1 + number_of_particles * (count++)] = n2;
      }
    }

    g_NN3b[n1] = count;
  }
}

static __global__ void find_partial_force_3body(
  int number_of_particles,
  int* Na,
  int* Na_sum,
  int* g_neighbor_number,
  int* g_neighbor_list,
  int* g_type,
  NEP::Para3B para3b,
  NEP::ANN ann3b,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  float* g_potential,
  float* g_f12x,
  float* g_f12y,
  float* g_f12z)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;

  if (n1 < N2) {
    const float* __restrict__ h = g_box + 18 * blockIdx.x;
    int neighbor_number = g_neighbor_number[n1];
    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];
    float pot_energy = 0.0f;

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * number_of_particles + n1;
      int n2 = g_neighbor_list[index];
      float x12 = g_x[n2] - x1;
      float y12 = g_y[n2] - y1;
      float z12 = g_z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      float d12inv = 1.0f / d12;
      float fc12, fcp12;
      find_fc_and_fcp(para3b.r1, para3b.r2, para3b.pi_factor, d12, fc12, fcp12);

      float p12 = 0.0f, f12[3] = {0.0f, 0.0f, 0.0f};

      for (int i2 = 0; i2 < neighbor_number; ++i2) {
        int n3 = g_neighbor_list[n1 + number_of_particles * i2];
        if (n3 == n2) {
          continue;
        }
        float x13 = g_x[n3] - x1;
        float y13 = g_y[n3] - y1;
        float z13 = g_z[n3] - z1;
        dev_apply_mic(h, x13, y13, z13);
        float d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
        float fc13;
        find_fc(para3b.r1, para3b.r2, para3b.pi_factor, d13, fc13);

        float x23 = x13 - x12;
        float y23 = y13 - y12;
        float z23 = z13 - z12;
        float d23 = sqrt(x23 * x23 + y23 * y23 + z23 * z23);
        float d23inv = 1.0f / d23;
        float q[3] = {d12 + d13, (d12 - d13) * (d12 - d13), d23};
        float p123 = 0.0f, f123[3] = {0.0f, 0.0f, 0.0f};
        apply_ann(ann3b, q, p123, f123);

        p12 += p123 * fc12 * fc13;
        float tmp = p123 * fcp12 * fc13 + (f123[0] + f123[1] * (d12 - d13) * 2.0f) * fc12 * fc13;
        f12[0] += 2.0f * (tmp * x12 * d12inv - f123[2] * fc12 * fc13 * x23 * d23inv);
        f12[1] += 2.0f * (tmp * y12 * d12inv - f123[2] * fc12 * fc13 * y23 * d23inv);
        f12[2] += 2.0f * (tmp * z12 * d12inv - f123[2] * fc12 * fc13 * z23 * d23inv);
      }
      pot_energy += p12;
      g_f12x[index] = f12[0];
      g_f12y[index] = f12[1];
      g_f12z[index] = f12[2];
    }
    g_potential[n1] += pot_energy;
  }
}

static __global__ void find_force_3body(
  int number_of_particles,
  int* Na,
  int* Na_sum,
  int* g_neighbor_number,
  int* g_neighbor_list,
  const float* __restrict__ g_f12x,
  const float* __restrict__ g_f12y,
  const float* __restrict__ g_f12z,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_virial)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_virial_xx = 0.0f;
    float s_virial_yy = 0.0f;
    float s_virial_zz = 0.0f;
    float s_virial_xy = 0.0f;
    float s_virial_yz = 0.0f;
    float s_virial_zx = 0.0f;
    const float* __restrict__ h = g_box + 18 * blockIdx.x;
    int neighbor_number = g_neighbor_number[n1];
    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * number_of_particles + n1;
      int n2 = g_neighbor_list[index];
      int neighbor_number_2 = g_neighbor_number[n2];

      float x12 = g_x[n2] - x1;
      float y12 = g_y[n2] - y1;
      float z12 = g_z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);

      float f12x = g_f12x[index];
      float f12y = g_f12y[index];
      float f12z = g_f12z[index];
      int offset = 0;
      for (int k = 0; k < neighbor_number_2; ++k) {
        if (n1 == g_neighbor_list[n2 + number_of_particles * k]) {
          offset = k;
          break;
        }
      }
      index = offset * number_of_particles + n2;
      float f21x = g_f12x[index];
      float f21y = g_f12y[index];
      float f21z = g_f12z[index];

      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;

      s_virial_xx -= x12 * (f12x - f21x) * 0.5f;
      s_virial_yy -= y12 * (f12y - f21y) * 0.5f;
      s_virial_zz -= z12 * (f12z - f21z) * 0.5f;
      s_virial_xy -= x12 * (f12y - f21y) * 0.5f;
      s_virial_yz -= y12 * (f12z - f21z) * 0.5f;
      s_virial_zx -= z12 * (f12x - f21x) * 0.5f;
    }

    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    g_virial[n1] += s_virial_xx;
    g_virial[n1 + number_of_particles] += s_virial_yy;
    g_virial[n1 + number_of_particles * 2] += s_virial_zz;
    g_virial[n1 + number_of_particles * 3] += s_virial_xy;
    g_virial[n1 + number_of_particles * 4] += s_virial_yz;
    g_virial[n1 + number_of_particles * 5] += s_virial_zx;
  }
}

static __device__ float find_Tn(const int n, const int x)
{
  if (n == 0) {
    return 1.0f;
  } else if (n == 1) {
    return x;
  } else {
    float t0 = 1.0f;
    float t1 = x;
    float t2;
    for (int m = 2; m <= n; ++m) {
      t2 = 2.0f * t1 + t0;
      t0 = t1;
      t1 = t2;
    }
    return t2;
  }
}

static __global__ void find_energy_manybody(
  int number_of_particles,
  int* Na,
  int* Na_sum,
  int* g_NN,
  int* g_NL,
  int* g_type,
  NEP::ParaMB paramb,
  NEP::ANN annmb,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  float* g_pe,
  float* g_Fp)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = g_box + 18 * blockIdx.x;
    int neighbor_number = g_NN[n1];

    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];

    float q[27] = {0.0f};
    for (int n = 0; n < paramb.n_max; ++n) {
      float tmp_sum[10] = {0.0f};
      for (int i1 = 0; i1 < neighbor_number; ++i1) {
        int n2 = g_NL[n1 + number_of_particles * i1];
        float x12 = g_x[n2] - x1;
        float y12 = g_y[n2] - y1;
        float z12 = g_z[n2] - z1;
        dev_apply_mic(h, x12, y12, z12);
        float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
        float fc12;
        find_fc(paramb.r1, paramb.r2, paramb.pi_factor, d12, fc12);
        float fn = fc12 * find_Tn(n, 2 * d12 * paramb.r2inv - 1.0f);

        float d12inv = 1.0f / d12;
        x12 *= d12inv;
        y12 *= d12inv;
        z12 *= d12inv;
        tmp_sum[0] += fn;
        tmp_sum[1] += x12 * fn;
        tmp_sum[2] += y12 * fn;
        tmp_sum[3] += z12 * fn;
        tmp_sum[4] += x12 * x12 * fn;
        tmp_sum[5] += y12 * y12 * fn;
        tmp_sum[6] += z12 * z12 * fn;
        tmp_sum[7] += x12 * y12 * fn;
        tmp_sum[8] += x12 * z12 * fn;
        tmp_sum[9] += y12 * z12 * fn;
      }
      q[n * 3 + 0] = tmp_sum[0] * tmp_sum[0];
      q[n * 3 + 1] = tmp_sum[1] * tmp_sum[1] + tmp_sum[2] * tmp_sum[2] + tmp_sum[3] * tmp_sum[3];
      q[n * 3 + 2] = tmp_sum[7] * tmp_sum[7] + tmp_sum[8] * tmp_sum[8] + tmp_sum[9] * tmp_sum[9];
      q[n * 3 + 2] *= 2.0f;
      q[n * 3 + 2] += tmp_sum[4] * tmp_sum[4] + tmp_sum[5] * tmp_sum[5] + tmp_sum[6] * tmp_sum[6];
    }

    float F, Fp[27];
    apply_ann(annmb, q, F, Fp);
    g_pe[n1] += F;
    for (int d = 0; d < annmb.dim; ++d) {
      g_Fp[d * number_of_particles + n1] = Fp[d];
    }
  }
}

static __global__ void find_partial_force_manybody(
  int number_of_particles,
  int* Na,
  int* Na_sum,
  int* g_NN,
  int* g_NL,
  int* g_type,
  NEP::ParaMB paramb,
  NEP::ANN annmb,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  const float* __restrict__ g_Fp,
  const float* __restrict__ g_sum_f,
  const float* __restrict__ g_sum_fx,
  const float* __restrict__ g_sum_fy,
  const float* __restrict__ g_sum_fz,
  const float* __restrict__ g_sum_fxx,
  const float* __restrict__ g_sum_fyy,
  const float* __restrict__ g_sum_fzz,
  const float* __restrict__ g_sum_fxy,
  const float* __restrict__ g_sum_fxz,
  const float* __restrict__ g_sum_fyz,
  float* g_f12x,
  float* g_f12y,
  float* g_f12z)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = g_box + 18 * blockIdx.x;
    int neighbor_number = g_NN[n1];

    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * number_of_particles + n1;
      int n2 = g_NL[index];

      float x12 = g_x[n2] - x1;
      float y12 = g_y[n2] - y1;
      float z12 = g_z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      float fc12;
      find_fc(paramb.r1, paramb.r2, paramb.pi_factor, d12, fc12);
      float d12inv = 1.0f / d12;

      float f12[3];
      for (int n = 0; n < paramb.n_max; ++n) {
        float fn = fc12 * find_Tn(n, 2 * d12 * paramb.r2inv - 1.0f);
        float fnp = 0.0f; // TODO

        // x
        float dqdx_n0 = 2.0f * g_sum_f[n1 + number_of_particles * n] * fnp * x12;
        f12[0] += g_Fp[(n * 3 + 0) * number_of_particles + n1] * dqdx_n0;
        float dqdx_n1 = 2.0f * g_sum_fx[n1 + number_of_particles * n] *
                          (fnp * x12 * x12 + fn * (1.0f - x12 * x12) * d12inv) +
                        2.0f * g_sum_fy[n1 + number_of_particles * n] * (fnp * x12 * y12) +
                        2.0f * g_sum_fz[n1 + number_of_particles * n] * (fnp * x12 * z12);
        f12[0] += g_Fp[(n * 3 + 1) * number_of_particles + n1] * dqdx_n1;
        float dqdx_n2 = 0; // TODO
        f12[0] += g_Fp[(n * 3 + 2) * number_of_particles + n1] * dqdx_n2;

        // y
        float dqdy_n0 = 2.0f * g_sum_f[n1 + number_of_particles * n] * fnp * y12;
        f12[1] += g_Fp[(n * 3 + 0) * number_of_particles + n1] * dqdy_n0;
        float dqdy_n1 = 2.0f * g_sum_fx[n1 + number_of_particles * n] * (fnp * y12 * x12) +
                        2.0f * g_sum_fy[n1 + number_of_particles * n] *
                          (fnp * y12 * y12 + fn * (1.0f - y12 * y12) * d12inv) +
                        2.0f * g_sum_fz[n1 + number_of_particles * n] * (fnp * y12 * z12);
        f12[1] += g_Fp[(n * 3 + 1) * number_of_particles + n1] * dqdy_n1;
        float dqdy_n2 = 0; // TODO
        f12[1] += g_Fp[(n * 3 + 2) * number_of_particles + n1] * dqdy_n2;

        // z
        float dqdz_n0 = 2.0f * g_sum_f[n1 + number_of_particles * n] * fnp * z12;
        f12[2] += g_Fp[(n * 3 + 0) * number_of_particles + n1] * dqdz_n0;
        float dqdz_n1 = 2.0f * g_sum_fx[n1 + number_of_particles * n] * (fnp * z12 * x12) +
                        2.0f * g_sum_fy[n1 + number_of_particles * n] * (fnp * z12 * y12) +
                        2.0f * g_sum_fz[n1 + number_of_particles * n] *
                          (fnp * z12 * z12 + fn * (1.0f - z12 * z12) * d12inv);
        f12[2] += g_Fp[(n * 3 + 1) * number_of_particles + n1] * dqdz_n1;
        float dqdz_n2 = 0; // TODO
        f12[2] += g_Fp[(n * 3 + 2) * number_of_particles + n1] * dqdz_n2;
      }
      g_f12x[index] = f12[0];
      g_f12y[index] = f12[1];
      g_f12z[index] = f12[2];
    }
  }
}

static __global__ void
initialize_properties(int N, float* g_pe, float* g_fx, float* g_fy, float* g_fz, float* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    g_pe[n1] = 0.0f;
    g_fx[n1] = 0.0f;
    g_fy[n1] = 0.0f;
    g_fz[n1] = 0.0f;
    g_virial[n1 + 0 * N] = 0.0f;
    g_virial[n1 + 1 * N] = 0.0f;
    g_virial[n1 + 2 * N] = 0.0f;
    g_virial[n1 + 3 * N] = 0.0f;
    g_virial[n1 + 4 * N] = 0.0f;
    g_virial[n1 + 5 * N] = 0.0f;
  }
}

void NEP::find_force(
  int Nc,
  int N,
  int* Na,
  int* Na_sum,
  int max_Na,
  int* type,
  float* h,
  Neighbor* neighbor,
  float* r,
  GPU_Vector<float>& f,
  GPU_Vector<float>& virial,
  GPU_Vector<float>& pe)
{
  if (ann2b.num_neurons_per_layer > 0) {
    find_force_2body<<<Nc, max_Na>>>(
      N, Na, Na_sum, neighbor->NN, neighbor->NL, type, para2b, ann2b, r, r + N, r + N * 2, h,
      f.data(), f.data() + N, f.data() + N * 2, virial.data(), pe.data());
    CUDA_CHECK_KERNEL
  } else {
    initialize_properties<<<(N - 1) / 64 + 1, 64>>>(
      N, pe.data(), f.data(), f.data() + N, f.data() + N * 2, virial.data());
    CUDA_CHECK_KERNEL
  }

  if (ann3b.num_neurons_per_layer > 0) {
    find_neighbor_list_3body<<<Nc, max_Na>>>(
      N, Na, Na_sum, neighbor->NN, neighbor->NL, para3b, r, r + N, r + N * 2, h,
      nep_data.NN3b.data(), nep_data.NL3b.data());
    CUDA_CHECK_KERNEL

    find_partial_force_3body<<<Nc, max_Na>>>(
      N, Na, Na_sum, nep_data.NN3b.data(), nep_data.NL3b.data(), type, para3b, ann3b, r, r + N,
      r + N * 2, h, pe.data(), nep_data.f12x3b.data(), nep_data.f12y3b.data(),
      nep_data.f12z3b.data());
    CUDA_CHECK_KERNEL

    find_force_3body<<<Nc, max_Na>>>(
      N, Na, Na_sum, nep_data.NN3b.data(), nep_data.NL3b.data(), nep_data.f12x3b.data(),
      nep_data.f12y3b.data(), nep_data.f12z3b.data(), r, r + N, r + N * 2, h, f.data(),
      f.data() + N, f.data() + N * 2, virial.data());
    CUDA_CHECK_KERNEL
  }

  if (annmb.num_neurons_per_layer > 0) {
    find_energy_manybody<<<Nc, max_Na>>>(
      N, Na, Na_sum, neighbor->NN, neighbor->NL, type, paramb, annmb, r, r + N, r + N * 2, h,
      pe.data(), nep_data.Fp.data());
  }
}
