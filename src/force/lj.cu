#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the RDIP potential.
------------------------------------------------------------------------------*/

#include "lj.cuh"
#include "utilities/error.cuh"
#define BLOCK_SIZE_FORCE 128

LJ::LJ(FILE* fid, int num_types)
{
  printf("Use the RDIP potential.\n");
  int count = fscanf(
    fid, "%f%f%f%f%f%f%f%f%f%f", &lj_para.A, &lj_para.B, &lj_para.C, &lj_para.D1, &lj_para.D2,
    &lj_para.z0, &lj_para.alpha, &lj_para.lambda1, &lj_para.lambda2, &lj_para.rc);
  PRINT_SCANF_ERROR(count, 10, "Reading error for RDIP potential.");
  lj_para.z02 = lj_para.z0 * lj_para.z0;
  lj_para.Az06 = lj_para.A * lj_para.z02 * lj_para.z02 * lj_para.z02;
  rc = lj_para.rc;
}

LJ::~LJ(void) {}

static __global__ void gpu_find_force(
  LJ_Para lj,
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const int* g_type,
  const int shift,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial,
  double* g_potential)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  double s_fx = 0.0;                                   // force_x
  double s_fy = 0.0;                                   // force_y
  double s_fz = 0.0;                                   // force_z
  double s_pe = 0.0;                                   // potential energy
  double s_sxx = 0.0;                                  // virial_stress_xx
  double s_sxy = 0.0;                                  // virial_stress_xy
  double s_sxz = 0.0;                                  // virial_stress_xz
  double s_syx = 0.0;                                  // virial_stress_yx
  double s_syy = 0.0;                                  // virial_stress_yy
  double s_syz = 0.0;                                  // virial_stress_yz
  double s_szx = 0.0;                                  // virial_stress_zx
  double s_szy = 0.0;                                  // virial_stress_zy
  double s_szz = 0.0;                                  // virial_stress_zz

  if (n1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];

      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float x12 = float(x12double);
      float y12 = float(y12double);
      float z12 = float(z12double);

      float rhosq = x12 * x12 + y12 * y12;
      float d12sq = rhosq + z12 * z12;
      float d12 = sqrt(d12sq);
      float d12inv = 1 / d12;
      float d12inv2 = d12inv * d12inv;
      float d12inv4 = d12inv2 * d12inv2;
      float d12inv6 = d12inv4 * d12inv2;
      float d12inv8 = d12inv6 * d12inv2;

      float D_factor = lj.C * (1 + lj.D1 * rhosq + lj.D2 * rhosq * rhosq);
      float exp_alpha = lj.B * exp(-lj.alpha * (d12 - lj.z0));
      float exp_lambda = exp(-lj.lambda1 * rhosq - lj.lambda2 * (z12 * z12 - lj.z02));

      float tmp = -6.0f * lj.Az06 * d12inv8 - lj.alpha * exp_alpha * d12inv;
      float f12x = tmp * x12;
      float f12y = tmp * y12;
      float f12z = tmp * z12;
      tmp = 2 * exp_lambda * ((lj.D1 + 2 * lj.D2 * rhosq) * lj.C - lj.lambda1 * D_factor);
      f12x += tmp * x12;
      f12y += tmp * y12;
      tmp = -2 * lj.lambda2 * D_factor * exp_lambda;
      f12z += tmp * z12;

      s_pe += 0.5f * (lj.Az06 * d12inv6 + exp_alpha + D_factor * exp_lambda);
      s_fx += f12x;
      s_fy += f12y;
      s_fz += f12z;
      f12x *= 0.5f;
      f12y *= 0.5f;
      f12z *= 0.5f;
      s_sxx -= x12 * f12x;
      s_sxy -= x12 * f12y;
      s_sxz -= x12 * f12z;
      s_syx -= y12 * f12x;
      s_syy -= y12 * f12y;
      s_syz -= y12 * f12z;
      s_szx -= z12 * f12x;
      s_szy -= z12 * f12y;
      s_szz -= z12 * f12z;
    }
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;
    g_potential[n1] += s_pe;
  }
}

void LJ::compute(
  const int type_shift,
  const Box& box,
  const Neighbor& neighbor,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();
  int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

  gpu_find_force<<<grid_size, BLOCK_SIZE_FORCE>>>(
    lj_para, number_of_atoms, N1, N2, box, neighbor.NN_local.data(), neighbor.NL_local.data(),
    type.data(), type_shift, position_per_atom.data(), position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2, force_per_atom.data(),
    force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms,
    virial_per_atom.data(), potential_per_atom.data());
  CUDA_CHECK_KERNEL
}
