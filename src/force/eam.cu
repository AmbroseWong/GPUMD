#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The EAM potential. Currently two analytical versions:
[1] X. W. Zhou et al. Phys. Rev. B 69, 144113 (2004).
[2] X. D. Dai et al. JPCM 18, 4527 (2006).
------------------------------------------------------------------------------*/

#include "eam.cuh"
#include "neighbor.cuh"
#include "utilities/error.cuh"
#define BLOCK_SIZE_FORCE 64

EAM::EAM(FILE* fid, char* name, int num_types, const int number_of_atoms)
{
  if (strcmp(name, "eam_zhou_2004") == 0) {
    initialize_eam2004zhou(fid, num_types);
  } else if (strcmp(name, "eam_dai_2006") == 0) {
    initialize_eam2006dai(fid);
    if (num_types > 1) {
      PRINT_INPUT_ERROR(
        "eam_dai_2006 has not been implemented for systems with two or more atom types.");
    }
  }

  eam_data.Fp.resize(number_of_atoms);
  eam_data.NN.resize(number_of_atoms);
  eam_data.NL.resize(number_of_atoms * 400); // very safe for EAM
  eam_data.cell_count.resize(number_of_atoms);
  eam_data.cell_count_sum.resize(number_of_atoms);
  eam_data.cell_contents.resize(number_of_atoms);
}

void EAM::initialize_eam2004zhou(FILE* fid, int num_types)
{
  printf("Use %d-element EAM potential with element(s):", num_types);
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for EAM potential.");
    printf(" %s", atom_symbol);
  }
  printf("\n");

  potential_model = 0;

  rc = 0.0;
  for (int type = 0; type < num_types; ++type) {
    double x[21];
    for (int n = 0; n < 21; n++) {
      int count = fscanf(fid, "%lf", &x[n]);
      PRINT_SCANF_ERROR(count, 1, "Reading error for EAM potential.");
    }
    eam2004zhou.re_inv[type] = 1.0 / x[0];
    eam2004zhou.fe[type] = x[1];
    eam2004zhou.rho_e_inv[type] = 1.0 / x[2];
    eam2004zhou.rho_s_inv[type] = 1.0 / x[3];
    eam2004zhou.alpha[type] = x[4];
    eam2004zhou.beta[type] = x[5];
    eam2004zhou.A[type] = x[6];
    eam2004zhou.B[type] = x[7];
    eam2004zhou.kappa[type] = x[8];
    eam2004zhou.lambda[type] = x[9];
    eam2004zhou.Fn0[type] = x[10];
    eam2004zhou.Fn1[type] = x[11];
    eam2004zhou.Fn2[type] = x[12];
    eam2004zhou.Fn3[type] = x[13];
    eam2004zhou.F0[type] = x[14];
    eam2004zhou.F1[type] = x[15];
    eam2004zhou.F2[type] = x[16];
    eam2004zhou.F3[type] = x[17];
    eam2004zhou.eta[type] = x[18];
    eam2004zhou.Fe[type] = x[19];
    eam2004zhou.rc[type] = x[20];
    eam2004zhou.rho_n[type] = x[2] * 0.85;
    eam2004zhou.rho_0[type] = x[2] * 1.15;
    eam2004zhou.rho_n_inv[type] = 1.0 / eam2004zhou.rho_n[type];
    if (rc < eam2004zhou.rc[type]) {
      rc = eam2004zhou.rc[type];
    }
  }
}

void EAM::initialize_eam2006dai(FILE* fid)
{
  printf("Use 1-element EAM potential with element(s):");
  char atom_symbol[10];
  int count = fscanf(fid, "%s", atom_symbol);
  PRINT_SCANF_ERROR(count, 1, "Reading error for EAM potential.");
  printf(" %s\n", atom_symbol);

  potential_model = 1;

  double x[9];
  for (int n = 0; n < 9; n++) {
    int count = fscanf(fid, "%lf", &x[n]);
    PRINT_SCANF_ERROR(count, 1, "Reading error for EAM potential.");
  }
  eam2006dai.A = x[0];
  eam2006dai.d = x[1];
  eam2006dai.c = x[2];
  eam2006dai.c0 = x[3];
  eam2006dai.c1 = x[4];
  eam2006dai.c2 = x[5];
  eam2006dai.c3 = x[6];
  eam2006dai.c4 = x[7];
  eam2006dai.B = x[8];
  eam2006dai.rc = (eam2006dai.c > eam2006dai.d) ? eam2006dai.c : eam2006dai.d;
  rc = eam2006dai.rc;
}

EAM::~EAM(void)
{
  // nothing
}

// pair function (phi and phip have been intentionally halved here)
static __device__ void
find_phi(const EAM2004Zhou& eam, const int type, const float d12, float& phi, float& phip)
{
  float r_ratio = d12 * eam.re_inv[type];
  float tmp1 = (r_ratio - eam.kappa[type]) * (r_ratio - eam.kappa[type]);   // 2
  tmp1 *= tmp1;                                                             // 4
  tmp1 *= tmp1 * tmp1 * tmp1 * tmp1;                                        // 20
  float tmp2 = (r_ratio - eam.lambda[type]) * (r_ratio - eam.lambda[type]); // 2
  tmp2 *= tmp2;                                                             // 4
  tmp2 *= tmp2 * tmp2 * tmp2 * tmp2;                                        // 20
  float phi1 = 0.5f * eam.A[type] * exp(-eam.alpha[type] * (r_ratio - 1.0f)) / (1.0f + tmp1);
  float phi2 = 0.5f * eam.B[type] * exp(-eam.beta[type] * (r_ratio - 1.0f)) / (1.0f + tmp2);
  phi = phi1 - phi2;
  phip = (phi2 * eam.re_inv[type]) *
           (eam.beta[type] + 20.0f * tmp2 / (r_ratio - eam.lambda[type]) / (1.0f + tmp2)) -
         (phi1 * eam.re_inv[type]) *
           (eam.alpha[type] + 20.0f * tmp1 / (r_ratio - eam.kappa[type]) / (1.0f + tmp1));
}

// density function f(r)
static __device__ void find_f(const EAM2004Zhou& eam, const int type, const float d12, float& f)
{
  float r_ratio = d12 * eam.re_inv[type];
  float tmp = (r_ratio - eam.lambda[type]) * (r_ratio - eam.lambda[type]); // 2
  tmp *= tmp;                                                              // 4
  tmp *= tmp * tmp * tmp * tmp;                                            // 20
  f = eam.fe[type] * exp(-eam.beta[type] * (r_ratio - 1.0f)) / (1.0f + tmp);
}

// derivative of the density function f'(r)
static __device__ void find_fp(const EAM2004Zhou& eam, const int type, const float d12, float& fp)
{
  float r_ratio = d12 * eam.re_inv[type];
  float tmp = (r_ratio - eam.lambda[type]) * (r_ratio - eam.lambda[type]); // 2
  tmp *= tmp;                                                              // 4
  tmp *= tmp * tmp * tmp * tmp;                                            // 20
  float f = eam.fe[type] * exp(-eam.beta[type] * (r_ratio - 1.0f)) / (1.0f + tmp);
  fp = -(f * eam.re_inv[type]) *
       (eam.beta[type] + 20.0f * tmp / (r_ratio - eam.lambda[type]) / (1.0f + tmp));
}

static __device__ void
find_f_and_fp(const EAM2004Zhou& eam, const int type, const float d12, float& f, float& fp)
{
  float r_ratio = d12 * eam.re_inv[type];
  float tmp = (r_ratio - eam.lambda[type]) * (r_ratio - eam.lambda[type]); // 2
  tmp *= tmp;                                                              // 4
  tmp *= tmp * tmp * tmp * tmp;                                            // 20
  f = eam.fe[type] * exp(-eam.beta[type] * (r_ratio - 1.0f)) / (1.0f + tmp);
  fp = -(f * eam.re_inv[type]) *
       (eam.beta[type] + 20.0f * tmp / (r_ratio - eam.lambda[type]) / (1.0f + tmp));
}

// pair function for EAM2004Zhou
static __device__ void find_phi(
  const EAM2004Zhou& eam,
  const int type1,
  const int type2,
  const float d12,
  float& phi,
  float& phip)
{
  if (type1 == type2) {
    find_phi(eam, type1, d12, phi, phip);
  } else {
    float phi1, phip1;
    find_phi(eam, type1, d12, phi1, phip1);
    float phi2, phip2;
    find_phi(eam, type2, d12, phi2, phip2);
    float f1, fp1;
    find_f_and_fp(eam, type1, d12, f1, fp1);
    float f2, fp2;
    find_f_and_fp(eam, type2, d12, f2, fp2);
    float f1inv = 1.0f / f1;
    float f2inv = 1.0f / f2;
    phi = 0.5f * (phi1 * f2 * f1inv + phi2 * f1 * f2inv);
    phip = (phip1 * f2 + phi1 * (fp2 - f2 * fp1 * f1inv)) * f1inv;
    phip += (phip2 * f1 + phi2 * (fp1 - f1 * fp2 * f2inv)) * f2inv;
    phip *= 0.5f;
  }
}

// embedding function
static __device__ void
find_F(const EAM2004Zhou& eam, const int type, const float rho, float& F, float& Fp)
{
  if (rho < eam.rho_n[type]) {
    float x = rho * eam.rho_n_inv[type] - 1.0f;
    F = ((eam.Fn3[type] * x + eam.Fn2[type]) * x + eam.Fn1[type]) * x + eam.Fn0[type];
    Fp = ((3.0f * eam.Fn3[type] * x + 2.0f * eam.Fn2[type]) * x + eam.Fn1[type]) / eam.rho_n[type];
  } else if (rho < eam.rho_0[type]) {
    float x = rho * eam.rho_e_inv[type] - 1.0f;
    F = ((eam.F3[type] * x + eam.F2[type]) * x + eam.F1[type]) * x + eam.F0[type];
    Fp = ((3.0f * eam.F3[type] * x + 2.0f * eam.F2[type]) * x + eam.F1[type]) * eam.rho_e_inv[type];
  } else {
    float x = rho * eam.rho_s_inv[type];
    float x_eta = pow(x, eam.eta[type]);
    F = eam.Fe[type] * (1.0f - eam.eta[type] * log(x)) * x_eta;
    Fp = (eam.eta[type] / rho) * (F - eam.Fe[type] * x_eta);
  }
}

// pair function (phi and phip have been intentionally halved here)
static __device__ void find_phi(const EAM2006Dai& fs, const float d12, float& phi, float& phip)
{
  if (d12 > fs.c) {
    phi = 0.0f;
    phip = 0.0f;
  } else {
    float tmp = ((((fs.c4 * d12 + fs.c3) * d12 + fs.c2) * d12 + fs.c1) * d12 + fs.c0);

    phi = 0.5f * (d12 - fs.c) * (d12 - fs.c) * tmp;

    phip = 2.0f * (d12 - fs.c) * tmp;
    phip += (((4.0f * fs.c4 * d12 + 3.0f * fs.c3) * d12 + 2.0f * fs.c2) * d12 + fs.c1) *
            (d12 - fs.c) * (d12 - fs.c);
    phip *= 0.5f;
  }
}

// density function f(r)
static __device__ void find_f(const EAM2006Dai& fs, const float d12, float& f)
{
  if (d12 > fs.d) {
    f = 0.0f;
  } else {
    float tmp = (d12 - fs.d) * (d12 - fs.d);
    f = tmp + fs.B * fs.B * tmp * tmp;
  }
}

// derivative of the density function f'(r)
static __device__ void find_fp(const EAM2006Dai& fs, const float d12, float& fp)
{
  if (d12 > fs.d) {
    fp = 0.0f;
  } else {
    float tmp = 2.0f * (d12 - fs.d);
    fp = tmp * (1.0f + fs.B * fs.B * tmp * (d12 - fs.d));
  }
}

// embedding function
static __device__ void find_F(const EAM2006Dai& fs, const float rho, float& F, float& Fp)
{
  float sqrt_rho = sqrt(rho);
  F = -fs.A * sqrt_rho;
  Fp = -fs.A * 0.5f / sqrt_rho;
}

// Calculate the embedding energy and its derivative
template <int potential_model>
static __global__ void find_force_eam_step1(
  const EAM2004Zhou eam2004zhou,
  const EAM2006Dai eam2006dai,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  float* g_Fp,
  double* g_pe)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index

  if (n1 < N2) {
    int NN = g_NN[n1];

    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    // Calculate the density
    float rho = 0.0f;
    for (int i1 = 0; i1 < NN; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      float rho12 = 0.0f;
      if (potential_model == 0) {
        find_f(eam2004zhou, g_type[n2], d12, rho12); // density is contributed by n2
      }
      if (potential_model == 1) {
        find_f(eam2006dai, d12, rho12);
      }
      rho += rho12;
    }

    // Calculate the embedding energy F and its derivative Fp
    float F, Fp;
    if (potential_model == 0)
      find_F(eam2004zhou, g_type[n1], rho, F, Fp); // embedding energy is for n1
    if (potential_model == 1)
      find_F(eam2006dai, rho, F, Fp);

    g_pe[n1] += F; // many-body potential energy
    g_Fp[n1] = Fp;
  }
}

// Force evaluation kernel
template <int potential_model>
static __global__ void find_force_eam_step2(
  const EAM2004Zhou eam2004zhou,
  const EAM2006Dai eam2006dai,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* g_type,
  const float* __restrict__ g_Fp,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial,
  double* g_pe)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  float s_fx = 0.0f;  // force_x
  float s_fy = 0.0f;  // force_y
  float s_fz = 0.0f;  // force_z
  float s_pe = 0.0f;  // potential energy
  float s_sxx = 0.0f; // virial_stress_xx
  float s_sxy = 0.0f; // virial_stress_xy
  float s_sxz = 0.0f; // virial_stress_xz
  float s_syx = 0.0f; // virial_stress_yx
  float s_syy = 0.0f; // virial_stress_yy
  float s_syz = 0.0f; // virial_stress_yz
  float s_szx = 0.0f; // virial_stress_zx
  float s_szy = 0.0f; // virial_stress_zy
  float s_szz = 0.0f; // virial_stress_zz

  if (n1 < N2) {
    int type1 = g_type[n1];
    int NN = g_NN[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    float Fp1 = g_Fp[n1];

    for (int i1 = 0; i1 < NN; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      int type2 = g_type[n2];
      float Fp2 = g_Fp[n2];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);

      float phi, phip, fp1, fp2;
      if (potential_model == 0) {
        find_phi(eam2004zhou, type1, type2, d12, phi, phip);
        if (type1 == type2) {
          find_fp(eam2004zhou, type1, d12, fp1);
          fp2 = fp1;
        } else {
          find_fp(eam2004zhou, type1, d12, fp1);
          find_fp(eam2004zhou, type2, d12, fp2);
        }
      }
      if (potential_model == 1) {
        find_phi(eam2006dai, d12, phi, phip);
        find_fp(eam2006dai, d12, fp1);
        fp2 = fp1;
      }

      float d12inv = 1.0f / d12;
      phip *= d12inv;
      fp1 *= d12inv;
      fp2 *= d12inv;
      float f12x = x12 * (phip + Fp1 * fp2);
      float f12y = y12 * (phip + Fp1 * fp2);
      float f12z = z12 * (phip + Fp1 * fp2);
      float f21x = -x12 * (phip + Fp2 * fp1);
      float f21y = -y12 * (phip + Fp2 * fp1);
      float f21z = -z12 * (phip + Fp2 * fp1);

      // two-body potential energy
      s_pe += phi;

      // per atom force
      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;

      // per-atom virial
      s_sxx += x12 * f21x;
      s_sxy += x12 * f21y;
      s_sxz += x12 * f21z;
      s_syx += y12 * f21x;
      s_syy += y12 * f21y;
      s_syz += y12 * f21z;
      s_szx += z12 * f21x;
      s_szy += z12 * f21y;
      s_szz += z12 * f21z;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * N] += s_sxx;
    g_virial[n1 + 1 * N] += s_syy;
    g_virial[n1 + 2 * N] += s_szz;
    g_virial[n1 + 3 * N] += s_sxy;
    g_virial[n1 + 4 * N] += s_sxz;
    g_virial[n1 + 5 * N] += s_syz;
    g_virial[n1 + 6 * N] += s_syx;
    g_virial[n1 + 7 * N] += s_szx;
    g_virial[n1 + 8 * N] += s_szy;

    // save potential energy
    g_pe[n1] += s_pe;
  }
}

// Force evaluation wrapper
void EAM::compute(
  Box& box,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();
  int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

#ifdef USE_FIXED_NEIGHBOR
  static int num_calls = 0;
#endif
#ifdef USE_FIXED_NEIGHBOR
  if (num_calls++ == 0) {
#endif
    find_neighbor(
      N1, N2, rc, box, type, position_per_atom, eam_data.cell_count, eam_data.cell_count_sum,
      eam_data.cell_contents, eam_data.NN, eam_data.NL);
#ifdef USE_FIXED_NEIGHBOR
  }
#endif

  if (potential_model == 0) {
    find_force_eam_step1<0><<<grid_size, BLOCK_SIZE_FORCE>>>(
      eam2004zhou, eam2006dai, number_of_atoms, N1, N2, box, eam_data.NN.data(), eam_data.NL.data(),
      type.data(), position_per_atom.data(), position_per_atom.data() + number_of_atoms,
      position_per_atom.data() + number_of_atoms * 2, eam_data.Fp.data(),
      potential_per_atom.data());
    CUDA_CHECK_KERNEL

    find_force_eam_step2<0><<<grid_size, BLOCK_SIZE_FORCE>>>(
      eam2004zhou, eam2006dai, number_of_atoms, N1, N2, box, eam_data.NN.data(), eam_data.NL.data(),
      type.data(), eam_data.Fp.data(), position_per_atom.data(),
      position_per_atom.data() + number_of_atoms, position_per_atom.data() + number_of_atoms * 2,
      force_per_atom.data(), force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms, virial_per_atom.data(),
      potential_per_atom.data());
    CUDA_CHECK_KERNEL
  }

  if (potential_model == 1) {
    find_force_eam_step1<1><<<grid_size, BLOCK_SIZE_FORCE>>>(
      eam2004zhou, eam2006dai, number_of_atoms, N1, N2, box, eam_data.NN.data(), eam_data.NL.data(),
      type.data(), position_per_atom.data(), position_per_atom.data() + number_of_atoms,
      position_per_atom.data() + number_of_atoms * 2, eam_data.Fp.data(),
      potential_per_atom.data());
    CUDA_CHECK_KERNEL

    find_force_eam_step2<1><<<grid_size, BLOCK_SIZE_FORCE>>>(
      eam2004zhou, eam2006dai, number_of_atoms, N1, N2, box, eam_data.NN.data(), eam_data.NL.data(),
      type.data(), eam_data.Fp.data(), position_per_atom.data(),
      position_per_atom.data() + number_of_atoms, position_per_atom.data() + number_of_atoms * 2,
      force_per_atom.data(), force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms, virial_per_atom.data(),
      potential_per_atom.data());
    CUDA_CHECK_KERNEL
  }
}
