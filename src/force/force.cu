#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The driver class calculating force and related quantities.
------------------------------------------------------------------------------*/

#include "eam.cuh"
#include "fcp.cuh"
#include "force.cuh"
#include "lj.cuh"
#include "nep3.cuh"
#include "nep3_multigpu.cuh"
#include "potential.cuh"
#include "tersoff1988.cuh"
#include "tersoff1989.cuh"
#include "tersoff_mini.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/read_file.cuh"
#include <vector>

#define BLOCK_SIZE 128

Force::Force(void) { is_fcp = false; }

void Force::parse_potential(
  char** param, int num_param, char* input_dir, const Box& box, const int number_of_atoms)
{
  static int num_calls = 0;
  if (num_calls++ != 0) {
    PRINT_INPUT_ERROR("potential keyword can only be used once.\n");
  }

  if (num_param != 2) {
    PRINT_INPUT_ERROR("potential should have 1 parameter.\n");
  }

  FILE* fid_potential = my_fopen(param[1], "r");
  char potential_name[20];
  int count = fscanf(fid_potential, "%s", potential_name);
  if (count != 1) {
    PRINT_INPUT_ERROR("reading error for potential file.");
  }
  int num_types = get_number_of_types(fid_potential);

  // determine the potential
  if (strcmp(potential_name, "tersoff_1989") == 0) {
    potential.reset(new Tersoff1989(fid_potential, num_types, number_of_atoms));
  } else if (strcmp(potential_name, "tersoff_1988") == 0) {
    potential.reset(new Tersoff1988(fid_potential, num_types, number_of_atoms));
  } else if (strcmp(potential_name, "tersoff_mini") == 0) {
    potential.reset(new Tersoff_mini(fid_potential, num_types, number_of_atoms));
  } else if (strcmp(potential_name, "eam_zhou_2004") == 0) {
    potential.reset(new EAM(fid_potential, potential_name, num_types, number_of_atoms));
  } else if (strcmp(potential_name, "eam_dai_2006") == 0) {
    potential.reset(new EAM(fid_potential, potential_name, num_types, number_of_atoms));
  } else if (strcmp(potential_name, "fcp") == 0) {
    potential.reset(new FCP(fid_potential, input_dir, num_types, number_of_atoms, box));
    is_fcp = true;
  } else if (
    strcmp(potential_name, "nep") == 0 || strcmp(potential_name, "nep_zbl") == 0 ||
    strcmp(potential_name, "nep3") == 0 || strcmp(potential_name, "nep3_zbl") == 0) {
    int num_gpus;
    CHECK(hipGetDeviceCount(&num_gpus));
#ifdef ZHEYONG
    num_gpus = 3;
#endif
    if (num_gpus == 1) {
      potential.reset(new NEP3(param[1], number_of_atoms));
    } else {
      potential.reset(new NEP3_MULTIGPU(num_gpus, param[1], number_of_atoms));
    }
  } else if (strcmp(potential_name, "lj") == 0) {
    potential.reset(new LJ(fid_potential, num_types, number_of_atoms));
  } else {
    PRINT_INPUT_ERROR("illegal potential model.\n");
  }

  fclose(fid_potential);

  potential->N1 = 0;
  potential->N2 = number_of_atoms;
}

int Force::get_number_of_types(FILE* fid_potential)
{
  int num_of_types;
  int count = fscanf(fid_potential, "%d", &num_of_types);
  PRINT_SCANF_ERROR(count, 1, "Reading error for number of types.");
  return num_of_types;
}

static __global__ void gpu_add_driving_force(
  int N,
  double fe_x,
  double fe_y,
  double fe_z,
  double* g_sxx,
  double* g_sxy,
  double* g_sxz,
  double* g_syx,
  double* g_syy,
  double* g_syz,
  double* g_szx,
  double* g_szy,
  double* g_szz,
  double* g_fx,
  double* g_fy,
  double* g_fz)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    g_fx[i] += fe_x * g_sxx[i] + fe_y * g_syx[i] + fe_z * g_szx[i];
    g_fy[i] += fe_x * g_sxy[i] + fe_y * g_syy[i] + fe_z * g_szy[i];
    g_fz[i] += fe_x * g_sxz[i] + fe_y * g_syz[i] + fe_z * g_szz[i];
  }
}

// get the total force
static __global__ void gpu_sum_force(int N, double* g_fx, double* g_fy, double* g_fz, double* g_f)
{
  //<<<3, 1024>>>
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int number_of_patches = (N - 1) / 1024 + 1;
  __shared__ double s_f[1024];
  double f = 0.0;

  switch (bid) {
    case 0:
      for (int patch = 0; patch < number_of_patches; ++patch) {
        int n = tid + patch * 1024;
        if (n < N)
          f += g_fx[n];
      }
      break;
    case 1:
      for (int patch = 0; patch < number_of_patches; ++patch) {
        int n = tid + patch * 1024;
        if (n < N)
          f += g_fy[n];
      }
      break;
    case 2:
      for (int patch = 0; patch < number_of_patches; ++patch) {
        int n = tid + patch * 1024;
        if (n < N)
          f += g_fz[n];
      }
      break;
  }
  s_f[tid] = f;
  __syncthreads();

#pragma unroll
  for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_f[tid] += s_f[tid + offset];
    }
    __syncthreads();
  }
  for (int offset = 32; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_f[tid] += s_f[tid + offset];
    }
    __syncwarp();
  }

  if (tid == 0) {
    g_f[bid] = s_f[0];
  }
}

// correct the total force
static __global__ void
gpu_correct_force(int N, double one_over_N, double* g_fx, double* g_fy, double* g_fz, double* g_f)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    g_fx[i] -= g_f[0] * one_over_N;
    g_fy[i] -= g_f[1] * one_over_N;
    g_fz[i] -= g_f[2] * one_over_N;
  }
}

static __global__ void initialize_properties(
  int N, double* g_fx, double* g_fy, double* g_fz, double* g_pe, double* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    g_fx[n1] = 0.0;
    g_fy[n1] = 0.0;
    g_fz[n1] = 0.0;
    g_pe[n1] = 0.0;
    g_virial[n1 + 0 * N] = 0.0;
    g_virial[n1 + 1 * N] = 0.0;
    g_virial[n1 + 2 * N] = 0.0;
    g_virial[n1 + 3 * N] = 0.0;
    g_virial[n1 + 4 * N] = 0.0;
    g_virial[n1 + 5 * N] = 0.0;
    g_virial[n1 + 6 * N] = 0.0;
    g_virial[n1 + 7 * N] = 0.0;
    g_virial[n1 + 8 * N] = 0.0;
  }
}

void Force::set_hnemd_parameters(
  const bool compute_hnemd,
  const double hnemd_fe_x,
  const double hnemd_fe_y,
  const double hnemd_fe_z)
{
  compute_hnemd_ = compute_hnemd;
  if (compute_hnemd) {
    hnemd_fe_[0] = hnemd_fe_x;
    hnemd_fe_[1] = hnemd_fe_y;
    hnemd_fe_[2] = hnemd_fe_z;
  }
}

void Force::set_hnemdec_parameters(
  const int compute_hnemdec,
  const double hnemd_fe_x,
  const double hnemd_fe_y,
  const double hnemd_fe_z,
  const std::vector<double>& mass,
  const std::vector<int>& type,
  const std::vector<int>& type_size,
  const double T)
{
  int N = mass.size();
  int N1 = type_size[0];
  int N2 = type_size[1];
  int number_of_types = type_size.size();
  compute_hnemdec_ = compute_hnemdec;
  temperature = T;
  std::vector<double> cpu_coefficient;

  // find 2 atom types' mass or fraction
  if (compute_hnemdec_ == 1) {
    std::vector<double> mass_type;
    mass_type.resize(number_of_types);
    int find_mass_type = 0;
    for (int i = 0; i < mass_type.size(); i++) {
      mass_type[i] = 0;
    }
    for (int i = 0; i < N; i++) {
      if (mass_type[type[i]] != mass[i]) {
        mass_type[type[i]] = mass[i];
        find_mass_type += 1;
      }
      if (find_mass_type == 2) {
        break;
      }
    }

    int COEFF_NUM = 4;
    cpu_coefficient.resize(COEFF_NUM);
    coefficient.resize(COEFF_NUM);

    double m1 = mass_type[0];
    double m2 = mass_type[1];
    double miu = 1 / m1 - 1 / m2;
    double tmp = m1 * m2 / (m1 * N1 + m2 * N2);
    double c1 = miu * N2 * tmp;
    double c2 = -1 * miu * N1 * tmp;
    double c11 = (c1 - 1) / N;
    double c12 = c1 * K_B * temperature;
    double c21 = (c2 - 1) / N;
    double c22 = c2 * K_B * temperature;

    cpu_coefficient[0] = c11;
    cpu_coefficient[1] = c21;
    cpu_coefficient[2] = c12;
    cpu_coefficient[3] = c22;
    coefficient.copy_from_host(cpu_coefficient.data());
  } else if (compute_hnemdec_ == 2) {
    int COEFF_NUM = 2;
    cpu_coefficient.resize(COEFF_NUM);
    cpu_coefficient[0] = N2 / double(N);
    cpu_coefficient[1] = -1 * N1 / double(N);
    coefficient.resize(COEFF_NUM);
    coefficient.copy_from_host(cpu_coefficient.data());
  }

  hnemd_fe_[0] = hnemd_fe_x;
  hnemd_fe_[1] = hnemd_fe_y;
  hnemd_fe_[2] = hnemd_fe_z;
}

static __global__ void gpu_apply_pbc(int N, Box box, double* g_x, double* g_y, double* g_z)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    if (box.triclinic == 0) {
      double lx = box.cpu_h[0];
      double ly = box.cpu_h[1];
      double lz = box.cpu_h[2];
      if (box.pbc_x == 1) {
        if (g_x[n] < 0) {
          g_x[n] += lx;
        } else if (g_x[n] > lx) {
          g_x[n] -= lx;
        }
      }
      if (box.pbc_y == 1) {
        if (g_y[n] < 0) {
          g_y[n] += ly;
        } else if (g_y[n] > ly) {
          g_y[n] -= ly;
        }
      }
      if (box.pbc_z == 1) {
        if (g_z[n] < 0) {
          g_z[n] += lz;
        } else if (g_z[n] > lz) {
          g_z[n] -= lz;
        }
      }
    } else {
      double x = g_x[n];
      double y = g_y[n];
      double z = g_z[n];
      double sx = box.cpu_h[9] * x + box.cpu_h[10] * y + box.cpu_h[11] * z;
      double sy = box.cpu_h[12] * x + box.cpu_h[13] * y + box.cpu_h[14] * z;
      double sz = box.cpu_h[15] * x + box.cpu_h[16] * y + box.cpu_h[17] * z;
      if (box.pbc_x == 1) {
        if (sx < 0.0) {
          sx += 1.0;
        } else if (sx > 1.0) {
          sx -= 1.0;
        }
      }
      if (box.pbc_y == 1) {
        if (sy < 0.0) {
          sy += 1.0;
        } else if (sy > 1.0) {
          sy -= 1.0;
        }
      }
      if (box.pbc_z == 1) {
        if (sz < 0.0) {
          sz += 1.0;
        } else if (sz > 1.0) {
          sz -= 1.0;
        }
      }
      g_x[n] = box.cpu_h[0] * sx + box.cpu_h[1] * sy + box.cpu_h[2] * sz;
      g_y[n] = box.cpu_h[3] * sx + box.cpu_h[4] * sy + box.cpu_h[5] * sz;
      g_z[n] = box.cpu_h[6] * sx + box.cpu_h[7] * sy + box.cpu_h[8] * sz;
    }
  }
}

void Force::compute(
  Box& box,
  GPU_Vector<double>& position_per_atom,
  GPU_Vector<int>& type,
  std::vector<Group>& group,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();
  if (!is_fcp) {
    gpu_apply_pbc<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, box, position_per_atom.data(), position_per_atom.data() + number_of_atoms,
      position_per_atom.data() + number_of_atoms * 2);
  }

  initialize_properties<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
    number_of_atoms, force_per_atom.data(), force_per_atom.data() + number_of_atoms,
    force_per_atom.data() + number_of_atoms * 2, potential_per_atom.data(), virial_per_atom.data());
  CUDA_CHECK_KERNEL

  potential->compute(
    box, type, position_per_atom, potential_per_atom, force_per_atom, virial_per_atom);

  if (compute_hnemd_) {
    // the virial tensor:
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    gpu_add_driving_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, hnemd_fe_[0], hnemd_fe_[1], hnemd_fe_[2],
      virial_per_atom.data() + 0 * number_of_atoms, virial_per_atom.data() + 3 * number_of_atoms,
      virial_per_atom.data() + 4 * number_of_atoms, virial_per_atom.data() + 6 * number_of_atoms,
      virial_per_atom.data() + 1 * number_of_atoms, virial_per_atom.data() + 5 * number_of_atoms,
      virial_per_atom.data() + 7 * number_of_atoms, virial_per_atom.data() + 8 * number_of_atoms,
      virial_per_atom.data() + 2 * number_of_atoms, force_per_atom.data(),
      force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms);

    GPU_Vector<double> ftot(3); // total force vector of the system

    gpu_sum_force<<<3, 1024>>>(
      number_of_atoms, force_per_atom.data(), force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms, ftot.data());
    CUDA_CHECK_KERNEL

    gpu_correct_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, 1.0 / number_of_atoms, force_per_atom.data(),
      force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms,
      ftot.data());
    CUDA_CHECK_KERNEL
  }

  // always correct the force when using the FCP potential
  if (is_fcp) {
    if (!compute_hnemd_) {
      GPU_Vector<double> ftot(3); // total force vector of the system
      gpu_sum_force<<<3, 1024>>>(
        number_of_atoms, force_per_atom.data(), force_per_atom.data() + number_of_atoms,
        force_per_atom.data() + 2 * number_of_atoms, ftot.data());
      CUDA_CHECK_KERNEL

      gpu_correct_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
        number_of_atoms, 1.0 / number_of_atoms, force_per_atom.data(),
        force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms,
        ftot.data());
      CUDA_CHECK_KERNEL
    }
  }
}

static __global__ void gpu_find_per_atom_tensor(
  int N,
  double* g_mass,
  double* g_potential,
  double* g_vx,
  double* g_vy,
  double* g_vz,
  double* g_sxx,
  double* g_sxy,
  double* g_sxz,
  double* g_syx,
  double* g_syy,
  double* g_syz,
  double* g_szx,
  double* g_szy,
  double* g_szz,
  double* g_tensor)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    double mass = g_mass[i];
    double potential = g_potential[i];
    double vx = g_vx[i];
    double vy = g_vy[i];
    double vz = g_vz[i];
    double energy = mass * (vx * vx + vy * vy + vz * vz) * 0.5 + potential;
    // the tensor:
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_tensor[i] = energy + g_sxx[i];
    g_tensor[i + 3 * N] = g_sxy[i];
    g_tensor[i + 4 * N] = g_sxz[i];
    g_tensor[i + 6 * N] = g_syx[i];
    g_tensor[i + N] = energy + g_syy[i];
    g_tensor[i + 5 * N] = g_syz[i];
    g_tensor[i + 7 * N] = g_szx[i];
    g_tensor[i + 8 * N] = g_szy[i];
    g_tensor[i + 2 * N] = energy + g_szz[i];
  }
}

static __global__ void gpu_sum_tensor(int N, double* g_tensor, double* g_sum_tensor)
{
  //<<<9,1024>>>
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int number_of_patches = (N - 1) / 1024 + 1;
  __shared__ double s_t[1024];
  double t = 0.0;

  for (int patch = 0; patch < number_of_patches; ++patch) {
    int n = tid + patch * 1024;
    if (n < N)
      t += g_tensor[bid * N + n];
  }
  s_t[tid] = t;
  __syncthreads();

#pragma unroll
  for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_t[tid] += s_t[tid + offset];
    }
    __syncthreads();
  }
  for (int offset = 32; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_t[tid] += s_t[tid + offset];
    }
    __syncwarp();
  }

  if (tid == 0) {
    g_sum_tensor[bid] = s_t[0];
  }
}

static __global__ void gpu_add_driving_force(
  int N,
  const double* coefficient,
  const int* g_type,
  double fe_x,
  double fe_y,
  double fe_z,
  double* g_sxx,
  double* g_sxy,
  double* g_sxz,
  double* g_syx,
  double* g_syy,
  double* g_syz,
  double* g_szx,
  double* g_szy,
  double* g_szz,
  double* g_tensor_tot,
  double* g_fx,
  double* g_fy,
  double* g_fz)
{
  // heat flow algorithm
  // coefficient: c11,c21,c12,c22
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    int type = g_type[i];
    double coefficient1 = coefficient[type];
    double coefficient2 = coefficient[type + 2];

    // the tensor:
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_fx[i] += fe_x * (g_sxx[i] + coefficient1 * g_tensor_tot[0] + coefficient2) +
               fe_y * (g_syx[i] + coefficient1 * g_tensor_tot[6]) +
               fe_z * (g_szx[i] + coefficient1 * g_tensor_tot[7]);

    g_fy[i] += fe_x * (g_sxy[i] + coefficient1 * g_tensor_tot[3]) +
               fe_y * (g_syy[i] + coefficient1 * g_tensor_tot[1] + coefficient2) +
               fe_z * (g_szy[i] + coefficient1 * g_tensor_tot[8]);

    g_fz[i] += fe_x * (g_sxz[i] + coefficient1 * g_tensor_tot[4]) +
               fe_y * (g_syz[i] + coefficient1 * g_tensor_tot[5]) +
               fe_z * (g_szz[i] + coefficient1 * g_tensor_tot[2] + coefficient2);
  }
}

static __global__ void gpu_add_driving_force(
  int N,
  const double* g_coefficient,
  const int* g_type,
  double fe_x,
  double fe_y,
  double fe_z,
  double* g_fx,
  double* g_fy,
  double* g_fz)
{
  // color conductivity algorithm
  // coefficient: c1,c2
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    int type = g_type[i];
    double coefficient = g_coefficient[type];

    g_fx[i] += fe_x * coefficient;
    g_fy[i] += fe_y * coefficient;
    g_fz[i] += fe_z * coefficient;
  }
}

void Force::compute(
  Box& box,
  GPU_Vector<double>& position_per_atom,
  GPU_Vector<int>& type,
  std::vector<Group>& group,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom,
  GPU_Vector<double>& velocity_per_atom,
  GPU_Vector<double>& mass_per_atom)
{
  const int number_of_atoms = type.size();
  if (!is_fcp) {
    gpu_apply_pbc<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, box, position_per_atom.data(), position_per_atom.data() + number_of_atoms,
      position_per_atom.data() + number_of_atoms * 2);
  }

  initialize_properties<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
    number_of_atoms, force_per_atom.data(), force_per_atom.data() + number_of_atoms,
    force_per_atom.data() + number_of_atoms * 2, potential_per_atom.data(), virial_per_atom.data());
  CUDA_CHECK_KERNEL

  potential->compute(
    box, type, position_per_atom, potential_per_atom, force_per_atom, virial_per_atom);

  if (compute_hnemd_) {
    // the virial tensor:
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    gpu_add_driving_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, hnemd_fe_[0], hnemd_fe_[1], hnemd_fe_[2],
      virial_per_atom.data() + 0 * number_of_atoms, virial_per_atom.data() + 3 * number_of_atoms,
      virial_per_atom.data() + 4 * number_of_atoms, virial_per_atom.data() + 6 * number_of_atoms,
      virial_per_atom.data() + 1 * number_of_atoms, virial_per_atom.data() + 5 * number_of_atoms,
      virial_per_atom.data() + 7 * number_of_atoms, virial_per_atom.data() + 8 * number_of_atoms,
      virial_per_atom.data() + 2 * number_of_atoms, force_per_atom.data(),
      force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms);

    GPU_Vector<double> ftot(3); // total force vector of the system

    gpu_sum_force<<<3, 1024>>>(
      number_of_atoms, force_per_atom.data(), force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms, ftot.data());
    CUDA_CHECK_KERNEL

    gpu_correct_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, 1.0 / number_of_atoms, force_per_atom.data(),
      force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms,
      ftot.data());
    CUDA_CHECK_KERNEL
  } else if (compute_hnemdec_ == 1) {
    // the tensor:
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    GPU_Vector<double> tensor_per_atom(number_of_atoms * 9);
    GPU_Vector<double> tensor_tot(9);

    gpu_find_per_atom_tensor<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, mass_per_atom.data(), potential_per_atom.data(), velocity_per_atom.data(),
      velocity_per_atom.data() + number_of_atoms, velocity_per_atom.data() + 2 * number_of_atoms,
      virial_per_atom.data() + 0 * number_of_atoms, virial_per_atom.data() + 3 * number_of_atoms,
      virial_per_atom.data() + 4 * number_of_atoms, virial_per_atom.data() + 6 * number_of_atoms,
      virial_per_atom.data() + 1 * number_of_atoms, virial_per_atom.data() + 5 * number_of_atoms,
      virial_per_atom.data() + 7 * number_of_atoms, virial_per_atom.data() + 8 * number_of_atoms,
      virial_per_atom.data() + 2 * number_of_atoms, tensor_per_atom.data());
    CUDA_CHECK_KERNEL

    gpu_sum_tensor<<<9, 1024>>>(number_of_atoms, tensor_per_atom.data(), tensor_tot.data());
    CUDA_CHECK_KERNEL

    gpu_add_driving_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, coefficient.data(), type.data(), hnemd_fe_[0], hnemd_fe_[1], hnemd_fe_[2],
      tensor_per_atom.data() + 0 * number_of_atoms, tensor_per_atom.data() + 3 * number_of_atoms,
      tensor_per_atom.data() + 4 * number_of_atoms, tensor_per_atom.data() + 6 * number_of_atoms,
      tensor_per_atom.data() + 1 * number_of_atoms, tensor_per_atom.data() + 5 * number_of_atoms,
      tensor_per_atom.data() + 7 * number_of_atoms, tensor_per_atom.data() + 8 * number_of_atoms,
      tensor_per_atom.data() + 2 * number_of_atoms, tensor_tot.data(), force_per_atom.data(),
      force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms);
    CUDA_CHECK_KERNEL

  } else if (compute_hnemdec_ == 2) {
    gpu_add_driving_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms, coefficient.data(), type.data(), hnemd_fe_[0], hnemd_fe_[1], hnemd_fe_[2],
      force_per_atom.data(), force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms);
  }

  // always correct the force when using the FCP potential
  if (is_fcp) {
    if (!compute_hnemd_) {
      GPU_Vector<double> ftot(3); // total force vector of the system
      gpu_sum_force<<<3, 1024>>>(
        number_of_atoms, force_per_atom.data(), force_per_atom.data() + number_of_atoms,
        force_per_atom.data() + 2 * number_of_atoms, ftot.data());
      CUDA_CHECK_KERNEL

      gpu_correct_force<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
        number_of_atoms, 1.0 / number_of_atoms, force_per_atom.data(),
        force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms,
        ftot.data());
      CUDA_CHECK_KERNEL
    }
  }
}
