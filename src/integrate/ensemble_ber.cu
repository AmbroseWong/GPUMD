#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The Berendsen thermostat:
[1] H. J. C. Berendsen et al. J. Chem. Phys. 81, 3684 (1984).
------------------------------------------------------------------------------*/

#include "ensemble_ber.cuh"

Ensemble_BER::Ensemble_BER(int t, int fg, double T, double Tc)
{
  type = t;
  fixed_group = fg;
  temperature = T;
  temperature_coupling = Tc;
}

Ensemble_BER::Ensemble_BER(
  int t,
  int fg,
  double T,
  double Tc,
  double px,
  double py,
  double pz,
  bool is_iso,
  double pc,
  int dx,
  int dy,
  int dz,
  double rate)
{
  type = t;
  fixed_group = fg;
  temperature = T;
  temperature_coupling = Tc;
  pressure_x = px;
  pressure_y = py;
  pressure_z = pz;
  is_isotropic_pressure = is_iso;
  pressure_coupling = pc;
  deform_x = dx;
  deform_y = dy;
  deform_z = dz;
  deform_rate = rate;
}

Ensemble_BER::~Ensemble_BER(void)
{
  // nothing now
}

static __global__ void gpu_berendsen_temperature(
  int N,
  double temperature,
  double coupling,
  double* g_prop,
  double* g_vx,
  double* g_vy,
  double* g_vz)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    double factor = sqrt(1.0 + coupling * (temperature / g_prop[0] - 1.0));
    g_vx[i] *= factor;
    g_vy[i] *= factor;
    g_vz[i] *= factor;
  }
}

static __global__ void gpu_berendsen_pressure(
  int deform_x,
  int deform_y,
  int deform_z,
  double deform_rate,
  int number_of_particles,
  Box box,
  double p0x,
  double p0y,
  double p0z,
  double p_coupling,
  double* g_prop,
  double* g_x,
  double* g_y,
  double* g_z)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < number_of_particles) {
    if (deform_x) {
      double scale_factor = box.cpu_h[0];
      scale_factor = (scale_factor + deform_rate) / scale_factor;
      g_x[i] *= scale_factor;
    } else if (box.pbc_x == 1) {
      double scale_factor = 1.0 - p_coupling * (p0x - g_prop[2]);
      g_x[i] *= scale_factor;
    }
    if (deform_y) {
      double scale_factor = box.cpu_h[1];
      scale_factor = (scale_factor + deform_rate) / scale_factor;
      g_y[i] *= scale_factor;
    } else if (box.pbc_y == 1) {
      double scale_factor = 1.0 - p_coupling * (p0y - g_prop[3]);
      g_y[i] *= scale_factor;
    }
    if (deform_z) {
      double scale_factor = box.cpu_h[2];
      scale_factor = (scale_factor + deform_rate) / scale_factor;
      g_z[i] *= scale_factor;
    } else if (box.pbc_z == 1) {
      double scale_factor = 1.0 - p_coupling * (p0z - g_prop[4]);
      g_z[i] *= scale_factor;
    }
  }
}

static __global__ void gpu_berendsen_pressure_isotropic(
  int number_of_particles,
  double p0x,
  double p_coupling,
  double* g_prop,
  double* g_x,
  double* g_y,
  double* g_z)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < number_of_particles) {
    double scale_factor =
      1.0 - p_coupling * (p0x - (g_prop[2] + g_prop[3] + g_prop[4]) * 0.3333333333333333);
    g_x[i] *= scale_factor;
    g_y[i] *= scale_factor;
    g_z[i] *= scale_factor;
  }
}

static void cpu_berendsen_pressure(
  int deform_x,
  int deform_y,
  int deform_z,
  double deform_rate,
  Box& box,
  double p0x,
  double p0y,
  double p0z,
  bool is_isotropic_pressure,
  double p_coupling,
  double* thermo)
{
  double p[3];
  CHECK(hipMemcpy(p, thermo + 2, sizeof(double) * 3, hipMemcpyDeviceToHost));

  if (is_isotropic_pressure) {
    if (box.pbc_x == 0 || box.pbc_y == 0 || box.pbc_z == 0) {
      PRINT_INPUT_ERROR(
        "Cannot use isotropic pressure with non-periodic boundary in any direction.");
    }
    if (deform_x != 0 || deform_y != 0 || deform_z != 0) {
      PRINT_INPUT_ERROR("Cannot use isotropic pressure and also deform the box.");
    }
    double scale_factor = 1.0 - p_coupling * (p0x - (p[0] + p[1] + p[2]) * 0.3333333333333333);
    box.cpu_h[0] *= scale_factor;
    box.cpu_h[1] *= scale_factor;
    box.cpu_h[2] *= scale_factor;
    box.cpu_h[3] = box.cpu_h[0] * 0.5;
    box.cpu_h[4] = box.cpu_h[1] * 0.5;
    box.cpu_h[5] = box.cpu_h[2] * 0.5;
  } else {
    if (deform_x) {
      double scale_factor = box.cpu_h[0];
      scale_factor = (scale_factor + deform_rate) / scale_factor;
      box.cpu_h[0] *= scale_factor;
      box.cpu_h[3] = box.cpu_h[0] * 0.5;
    } else if (box.pbc_x == 1) {
      double scale_factor = 1.0 - p_coupling * (p0x - p[0]);
      box.cpu_h[0] *= scale_factor;
      box.cpu_h[3] = box.cpu_h[0] * 0.5;
    }

    if (deform_y) {
      double scale_factor = box.cpu_h[1];
      scale_factor = (scale_factor + deform_rate) / scale_factor;
      box.cpu_h[1] *= scale_factor;
      box.cpu_h[4] = box.cpu_h[1] * 0.5;
    } else if (box.pbc_y == 1) {
      double scale_factor = 1.0 - p_coupling * (p0y - p[1]);
      box.cpu_h[1] *= scale_factor;
      box.cpu_h[4] = box.cpu_h[1] * 0.5;
    }

    if (deform_z) {
      double scale_factor = box.cpu_h[2];
      scale_factor = (scale_factor + deform_rate) / scale_factor;
      box.cpu_h[2] *= scale_factor;
      box.cpu_h[5] = box.cpu_h[2] * 0.5;
    } else if (box.pbc_z == 1) {
      double scale_factor = 1.0 - p_coupling * (p0z - p[2]);
      box.cpu_h[2] *= scale_factor;
      box.cpu_h[5] = box.cpu_h[2] * 0.5;
    }
  }
}

void Ensemble_BER::compute1(
  const double time_step,
  const std::vector<Group>& group,
  const GPU_Vector<double>& mass,
  const GPU_Vector<double>& potential_per_atom,
  const GPU_Vector<double>& force_per_atom,
  const GPU_Vector<double>& virial_per_atom,
  Box& box,
  GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& velocity_per_atom,
  GPU_Vector<double>& thermo)
{
  velocity_verlet(
    true, time_step, group, mass, force_per_atom, position_per_atom, velocity_per_atom);
}

void Ensemble_BER::compute2(
  const double time_step,
  const std::vector<Group>& group,
  const GPU_Vector<double>& mass,
  const GPU_Vector<double>& potential_per_atom,
  const GPU_Vector<double>& force_per_atom,
  const GPU_Vector<double>& virial_per_atom,
  Box& box,
  GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& velocity_per_atom,
  GPU_Vector<double>& thermo)
{
  const int number_of_atoms = mass.size();

  velocity_verlet(
    false, time_step, group, mass, force_per_atom, position_per_atom, velocity_per_atom);

  find_thermo(
    box.get_volume(), group, mass, potential_per_atom, velocity_per_atom, virial_per_atom, thermo);
  gpu_berendsen_temperature<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
    number_of_atoms, temperature, temperature_coupling, thermo.data(), velocity_per_atom.data(),
    velocity_per_atom.data() + number_of_atoms, velocity_per_atom.data() + 2 * number_of_atoms);
  CUDA_CHECK_KERNEL
  if (type == 11) {
    cpu_berendsen_pressure(
      deform_x, deform_y, deform_z, deform_rate, box, pressure_x, pressure_y, pressure_z,
      is_isotropic_pressure, pressure_coupling, thermo.data());
    if (is_isotropic_pressure) {
      gpu_berendsen_pressure_isotropic<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
        number_of_atoms, pressure_x, pressure_coupling, thermo.data(), position_per_atom.data(),
        position_per_atom.data() + number_of_atoms, position_per_atom.data() + number_of_atoms * 2);
    } else {
      gpu_berendsen_pressure<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
        deform_x, deform_y, deform_z, deform_rate, number_of_atoms, box, pressure_x, pressure_y,
        pressure_z, pressure_coupling, thermo.data(), position_per_atom.data(),
        position_per_atom.data() + number_of_atoms, position_per_atom.data() + number_of_atoms * 2);
      CUDA_CHECK_KERNEL
    }
  }
}
