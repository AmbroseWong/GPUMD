#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "ensemble_ti_spring.cuh"

namespace
{
static __global__ void gpu_add_spring_force(
  int number_of_atoms,
  double lambda,
  double* espring,
  double* k,
  double* x,
  double* y,
  double* z,
  double* x0,
  double* y0,
  double* z0,
  double* fx,
  double* fy,
  double* fz)
{
  double dx, dy, dz;
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < number_of_atoms) {
    dx = x[i] - x0[i];
    dy = y[i] - y0[i];
    dz = z[i] - z0[i];
    fx[i] = (1 - lambda) * fx[i] + lambda * (-k[i] * dx);
    fy[i] = (1 - lambda) * fy[i] + lambda * (-k[i] * dy);
    fz[i] = (1 - lambda) * fz[i] + lambda * (-k[i] * dz);
    espring[i] = k[i] * (dx * dx + dy * dy + dz * dz);
  }
}

static __global__ void gpu_get_espring_sum(const int N, double* espring)
{
  //<<<1, 1024>>>
  int tid = threadIdx.x;
  int patch, n;
  int number_of_patches = (N - 1) / 1024 + 1;
  __shared__ double s_data[1024];
  s_data[tid] = 0.0;

  for (patch = 0; patch < number_of_patches; patch++) {
    n = tid + patch * 1024;
    if (n < N)
      s_data[tid] += espring[n];
  }
  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset)
      s_data[tid] += s_data[tid + offset];
    __syncthreads();
  }
  if (tid == 0)
    espring[0] = s_data[0];
}

} // namespace

Ensemble_TI_Spring::Ensemble_TI_Spring(const char** params, int num_params)
{
  use_barostat = false;
  use_thermostat = true;
}

void Ensemble_TI_Spring::init()
{
  int N = atom->number_of_atoms;
  gpu_k.resize(N);
  gpu_espring.resize(N);
  position_0.resize(3 * N);
  CHECK(hipMemcpy(
    position_0.data(),
    atom->position_per_atom.data(),
    sizeof(double) * position_0.size(),
    hipMemcpyDeviceToDevice));
}

Ensemble_TI_Spring::~Ensemble_TI_Spring(void) {}

void Ensemble_TI_Spring::add_spring_force()
{
  int N = atom->number_of_atoms;
  gpu_add_spring_force(
    N,
    lambda,
    gpu_espring.data(),
    gpu_k.data(),
    atom->position_per_atom.data(),
    atom->position_per_atom.data() + N,
    atom->position_per_atom.data() + 2 * N,
    position_0.data(),
    position_0.data() + N,
    position_0.data() + 2 * N,
    atom->force_per_atom.data(),
    atom->force_per_atom.data() + N,
    atom->force_per_atom.data() + 2 * N);
}

double Ensemble_TI_Spring::get_espring_sum()
{
  double temp;
  gpu_get_espring_sum(atom->number_of_atoms, gpu_espring.data());
  gpu_espring.copy_to_host(&temp, sizeof(double));
  return temp;
}
void Ensemble_TI_Spring::compute1(
  const double time_step,
  const std::vector<Group>& group,
  Box& box,
  Atom& atoms,
  GPU_Vector<double>& thermo)
{
  if (*current_step == 0)
    init();
  Ensemble_MTTK::compute1(time_step, group, box, atoms, thermo);
}

void Ensemble_TI_Spring::compute2(
  const double time_step,
  const std::vector<Group>& group,
  Box& box,
  Atom& atoms,
  GPU_Vector<double>& thermo)
{
  // modify force by spring
  add_spring_force();
  double espring = get_espring_sum();
  Ensemble_MTTK::compute2(time_step, group, box, atoms, thermo);
}
