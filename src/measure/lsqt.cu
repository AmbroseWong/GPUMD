#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "force/neighbor.cuh"
#include "lsqt.cuh"
#include "model/atom.cuh"
#include "model/box.cuh"
#include "utilities/common.cuh"

/*----------------------------------------------------------------------------80
    This file implements the linear-scaling quantum transport method
    similar to our GPUQT code (https://github.com/brucefan1983/gpuqt)

    In this file, we use the unit system with
        length:      Angstrom
        charge:      e
        energy:      eV
        energy*time: hbar
------------------------------------------------------------------------------*/

namespace
{
#define BLOCK_SIZE_EC 512 // do not change this
#define TIGHT_BINDING_MODEL 1
#define TIME_CONVERSION 15.46692
#define E_MAX 10.1

// set a state s = 0
__global__ void gpu_set_zero(int N, double* sr, double* si)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    sr[n] = 0.0;
    si[n] = 0.0;
  }
}

// copy state: so = si
__global__ void gpu_copy_state(int N, double* sir, double* sii, double* sor, double* soi)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    sor[n] = sir[n];
    soi[n] = sii[n];
  }
}

// add state: so = so + si
__global__ void gpu_add_state(int N, double* sir, double* sii, double* sor, double* soi)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    sor[n] += sir[n];
    soi[n] += sii[n];
  }
}

// will be used for U(t)
__global__ void gpu_chebyshev_01(
  int N,
  double* s0r,
  double* s0i,
  double* s1r,
  double* s1i,
  double* sr,
  double* si,
  double b0,
  double b1,
  int direction)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    double bessel_0 = b0;
    double bessel_1 = b1 * direction;
    sr[n] = bessel_0 * s0r[n] + bessel_1 * s1i[n];
    si[n] = bessel_0 * s0i[n] - bessel_1 * s1r[n];
  }
}

// will be used for [X, U(t)]
__global__ void
gpu_chebyshev_1x(int N, double* s1xr, double* s1xi, double* sr, double* si, double g_bessel_1)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    double b1 = g_bessel_1;
    sr[n] = +b1 * s1xi[n];
    si[n] = -b1 * s1xr[n];
  }
}

// will be used for U(t)
__global__ void gpu_chebyshev_2(
  int N,
  double Em_inv,
  int* NN,
  int* NL,
  double* U,
  double* Hr,
  double* Hi,
  double* s0r,
  double* s0i,
  double* s1r,
  double* s1i,
  double* s2r,
  double* s2i,
  double* sr,
  double* si,
  double bessel_m,
  int label)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    double temp_real = U[n] * s1r[n]; // on-site
    double temp_imag = U[n] * s1i[n]; // on-site
    int neighbor_number = NN[n];
#pragma unroll
    for (int m = 0; m < neighbor_number; ++m) {
      int index_1 = m * N + n;
      int index_2 = NL[index_1];
      double a = Hr[index_1];
      double b = Hi[index_1];
      double c = s1r[index_2];
      double d = s1i[index_2];
      temp_real += a * c - b * d; // hopping
      temp_imag += a * d + b * c; // hopping
    }
    temp_real *= Em_inv; // scale
    temp_imag *= Em_inv; // scale

    temp_real = 2.0 * temp_real - s0r[n];
    temp_imag = 2.0 * temp_imag - s0i[n];
    switch (label) {
      case 1: {
        sr[n] += bessel_m * temp_real;
        si[n] += bessel_m * temp_imag;
        break;
      }
      case 2: {
        sr[n] -= bessel_m * temp_real;
        si[n] -= bessel_m * temp_imag;
        break;
      }
      case 3: {
        sr[n] += bessel_m * temp_imag;
        si[n] -= bessel_m * temp_real;
        break;
      }
      case 4: {
        sr[n] -= bessel_m * temp_imag;
        si[n] += bessel_m * temp_real;
        break;
      }
    }
    s2r[n] = temp_real;
    s2i[n] = temp_imag;
  }
}

// for KPM
__global__ void gpu_kernel_polynomial(
  int N,
  double Em_inv,
  int* NN,
  int* NL,
  double* U,
  double* Hr,
  double* Hi,
  double* s0r,
  double* s0i,
  double* s1r,
  double* s1i,
  double* s2r,
  double* s2i)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {

    double temp_real = U[n] * s1r[n]; // on-site
    double temp_imag = U[n] * s1i[n]; // on-site
    int neighbor_number = NN[n];
#pragma unroll
    for (int m = 0; m < neighbor_number; ++m) {
      int index_1 = m * N + n;
      int index_2 = NL[index_1];
      double a = Hr[index_1];
      double b = Hi[index_1];
      double c = s1r[index_2];
      double d = s1i[index_2];
      temp_real += a * c - b * d; // hopping
      temp_imag += a * d + b * c; // hopping
    }

    temp_real *= Em_inv; // scale
    temp_imag *= Em_inv; // scale

    temp_real = 2.0 * temp_real - s0r[n];
    temp_imag = 2.0 * temp_imag - s0i[n];
    s2r[n] = temp_real;
    s2i[n] = temp_imag;
  }
}

// will be used for [X, U(t)]
__global__ void gpu_chebyshev_2x(
  int N,
  double Em_inv,
  int* NN,
  int* NL,
  double* U,
  double* Hr,
  double* Hi,
  double* g_xx,
  double* s0r,
  double* s0i,
  double* s0xr,
  double* s0xi,
  double* s1r,
  double* s1i,
  double* s1xr,
  double* s1xi,
  double* s2r,
  double* s2i,
  double* s2xr,
  double* s2xi,
  double* sr,
  double* si,
  double g_bessel_m,
  int g_label)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    double temp_real = U[n] * s1r[n];    // on-site
    double temp_imag = U[n] * s1i[n];    // on-site
    double temp_x_real = U[n] * s1xr[n]; // on-site
    double temp_x_imag = U[n] * s1xi[n]; // on-site
    int neighbor_number = NN[n];
#pragma unroll
    for (int m = 0; m < neighbor_number; ++m) {
      int index_1 = m * N + n;
      int index_2 = NL[index_1];

      double a = Hr[index_1];
      double b = Hi[index_1];
      double c = s1r[index_2];
      double d = s1i[index_2];
      temp_real += a * c - b * d; // hopping
      temp_imag += a * d + b * c; // hopping

      double cx = s1xr[index_2];
      double dx = s1xi[index_2];
      temp_x_real += a * cx - b * dx; // hopping
      temp_x_imag += a * dx + b * cx; // hopping

      double xx = g_xx[index_1];
      temp_x_real -= (a * c - b * d) * xx; // hopping
      temp_x_imag -= (a * d + b * c) * xx; // hopping
    }

    temp_real *= Em_inv; // scale
    temp_imag *= Em_inv; // scale
    temp_real = 2.0 * temp_real - s0r[n];
    temp_imag = 2.0 * temp_imag - s0i[n];
    s2r[n] = temp_real;
    s2i[n] = temp_imag;

    temp_x_real *= Em_inv; // scale
    temp_x_imag *= Em_inv; // scale
    temp_x_real = 2.0 * temp_x_real - s0xr[n];
    temp_x_imag = 2.0 * temp_x_imag - s0xi[n];
    s2xr[n] = temp_x_real;
    s2xi[n] = temp_x_imag;

    double bessel_m = g_bessel_m;
    switch (g_label) {
      case 1: {
        sr[n] += bessel_m * temp_x_real;
        si[n] += bessel_m * temp_x_imag;
        break;
      }
      case 2: {
        sr[n] -= bessel_m * temp_x_real;
        si[n] -= bessel_m * temp_x_imag;
        break;
      }
      case 3: {
        sr[n] += bessel_m * temp_x_imag;
        si[n] -= bessel_m * temp_x_real;
        break;
      }
      case 4: {
        sr[n] -= bessel_m * temp_x_imag;
        si[n] += bessel_m * temp_x_real;
        break;
      }
    }
  }
}

// apply the Hamiltonian: H * si = so
__global__ void gpu_apply_hamiltonian(
  int N,
  double Em_inv,
  int* NN,
  int* NL,
  double* U,
  double* Hr,
  double* Hi,
  double* sir,
  double* sii,
  double* sor,
  double* soi)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    double temp_real = U[n] * sir[n]; // on-site
    double temp_imag = U[n] * sii[n]; // on-site
    int neighbor_number = NN[n];
#pragma unroll
    for (int m = 0; m < neighbor_number; ++m) {
      int index_1 = m * N + n;
      int index_2 = NL[index_1];
      double a = Hr[index_1];
      double b = Hi[index_1];
      double c = sir[index_2];
      double d = sii[index_2];
      temp_real += a * c - b * d; // hopping
      temp_imag += a * d + b * c; // hopping
    }
    temp_real *= Em_inv; // scale
    temp_imag *= Em_inv; // scale
    sor[n] = temp_real;
    soi[n] = temp_imag;
  }
}

// so = V * si (no scaling; no on-site)
__global__ void gpu_apply_current(
  int N,
  int* NN,
  int* NL,
  double* Hr,
  double* Hi,
  double* g_xx,
  double* sir,
  double* sii,
  double* sor,
  double* soi)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    double temp_real = 0.0;
    double temp_imag = 0.0;
    int neighbor_number = NN[n];
#pragma unroll
    for (int m = 0; m < neighbor_number; ++m) {
      int index_1 = m * N + n;
      int index_2 = NL[index_1];
      double a = Hr[index_1];
      double b = Hi[index_1];
      double c = sir[index_2];
      double d = sii[index_2];
      double xx = g_xx[index_1];
      temp_real += (a * c - b * d) * xx;
      temp_imag += (a * d + b * c) * xx;
    }
    sor[n] = +temp_imag;
    soi[n] = -temp_real;
  }
}

// so = [X, H] * si
__global__ void gpu_apply_commutator(
  int N,
  double Em_inv,
  int* NN,
  int* NL,
  double* Hr,
  double* Hi,
  double* g_xx,
  double* sir,
  double* sii,
  double* sor,
  double* soi)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    double temp_real = 0.0;
    double temp_imag = 0.0;
    int neighbor_number = NN[n];
#pragma unroll
    for (int m = 0; m < neighbor_number; ++m) {
      int index_1 = m * N + n;
      int index_2 = NL[index_1];
      double a = Hr[index_1];
      double b = Hi[index_1];
      double c = sir[index_2];
      double d = sii[index_2];
      double xx = g_xx[index_1];
      temp_real -= (a * c - b * d) * xx;
      temp_imag -= (a * d + b * c) * xx;
    }
    sor[n] = temp_real * Em_inv;
    soi[n] = temp_imag * Em_inv;
  }
}

// 1st step of <sl|sr>
static __global__ void gpu_find_inner_product_1(
  int N, double* srr, double* sri, double* slr, double* sli, double* moments, int offset)
{
  int tid = threadIdx.x;
  int n = blockIdx.x * blockDim.x + tid;
  __shared__ double s_data[BLOCK_SIZE_EC];
  s_data[tid] = 0.0;
  if (n < N) {
    s_data[tid] = (srr[n] * slr[n] + sri[n] * sli[n]);
  }
  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_data[tid] += s_data[tid + offset];
    }
    __syncthreads();
  }
  if (tid == 0) {
    moments[blockIdx.x + offset] = s_data[0];
  }
}

// 2nd step of <sl|sr>
__global__ void gpu_find_inner_product_2(
  int number_of_blocks, int number_of_patches, double* moments_tmp, double* moments)
{
  int tid = threadIdx.x;
  __shared__ double s_data[BLOCK_SIZE_EC];
  s_data[tid] = 0.0;
#pragma unroll
  for (int patch = 0; patch < number_of_patches; ++patch) {
    int n = tid + patch * BLOCK_SIZE_EC;
    if (n < number_of_blocks) {
      s_data[tid] += moments_tmp[blockIdx.x * number_of_blocks + n];
    }
  }
  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_data[tid] += s_data[tid + offset];
    }
    __syncthreads();
  }
  if (tid == 0)
    moments[blockIdx.x] = s_data[0];
}

// get the Chebyshev moments: <sl|T_m(H)|sr>
void find_moments_chebyshev(
  int N,
  int Nm,
  double Em,
  int* NN,
  int* NL,
  double* U,
  double* Hr,
  double* Hi,
  double* slr,
  double* sli,
  double* srr,
  double* sri,
  double* moments)
{
  int grid_size = (N - 1) / BLOCK_SIZE_EC + 1;
  int number_of_blocks = grid_size;
  int number_of_patches = (number_of_blocks - 1) / BLOCK_SIZE_EC + 1;

  int memory_moments = sizeof(double) * Nm;
  int memory_moments_tmp = memory_moments * grid_size;
  double Em_inv = 1.0 / Em;

  double *s0r, *s1r, *s2r, *s0i, *s1i, *s2i, *moments_tmp;
  hipMalloc((void**)&s0r, sizeof(double) * N);
  hipMalloc((void**)&s1r, sizeof(double) * N);
  hipMalloc((void**)&s2r, sizeof(double) * N);
  hipMalloc((void**)&s0i, sizeof(double) * N);
  hipMalloc((void**)&s1i, sizeof(double) * N);
  hipMalloc((void**)&s2i, sizeof(double) * N);
  hipMalloc((void**)&moments_tmp, memory_moments_tmp);

  // T_0(H)
  gpu_copy_state<<<grid_size, BLOCK_SIZE_EC>>>(N, srr, sri, s0r, s0i);
  gpu_find_inner_product_1<<<grid_size, BLOCK_SIZE_EC>>>(
    N, s0r, s0i, slr, sli, moments_tmp, 0 * grid_size);

  // T_1(H)
  gpu_apply_hamiltonian<<<grid_size, BLOCK_SIZE_EC>>>(
    N, Em_inv, NN, NL, U, Hr, Hi, s0r, s0i, s1r, s1i);
  gpu_find_inner_product_1<<<grid_size, BLOCK_SIZE_EC>>>(
    N, s1r, s1i, slr, sli, moments_tmp, 1 * grid_size);

  // T_m(H) (m >= 2)
  for (int m = 2; m < Nm; ++m) {
    gpu_kernel_polynomial<<<grid_size, BLOCK_SIZE_EC>>>(
      N, Em_inv, NN, NL, U, Hr, Hi, s0r, s0i, s1r, s1i, s2r, s2i);
    gpu_find_inner_product_1<<<grid_size, BLOCK_SIZE_EC>>>(
      N, s2r, s2i, slr, sli, moments_tmp, m * grid_size);
    // permute the pointers; do not need to copy the data
    double* temp_real;
    double* temp_imag;
    temp_real = s0r;
    temp_imag = s0i;
    s0r = s1r;
    s0i = s1i;
    s1r = s2r;
    s1i = s2i;
    s2r = temp_real;
    s2i = temp_imag;
  }

  gpu_find_inner_product_2<<<Nm, BLOCK_SIZE_EC>>>(
    number_of_blocks, number_of_patches, moments_tmp, moments);

  hipFree(s0r);
  hipFree(s0i);
  hipFree(s1r);
  hipFree(s1i);
  hipFree(s2r);
  hipFree(s2i);
  hipFree(moments_tmp);
}

// Jackson damping
void apply_damping(int Nm, double* moments)
{
  for (int k = 0; k < Nm; ++k) {
    double a = 1.0 / (Nm + 1.0);
    double damping = (1.0 - k * a) * cos(k * PI * a) + sin(k * PI * a) * a / tan(PI * a);
    moments[k] *= damping;
  }
}

// kernel polynomial summation
void perform_chebyshev_summation(
  int Nm, int Ne, double Em, double* E, double V, double* moments, double* correlation)
{
  for (int step1 = 0; step1 < Ne; ++step1) {
    double energy_scaled = E[step1] / Em;
    double chebyshev_0 = 1.0;
    double chebyshev_1 = energy_scaled;
    double chebyshev_2;
    double temp = moments[1] * chebyshev_1;
    for (int step2 = 2; step2 < Nm; ++step2) {
      chebyshev_2 = 2.0 * energy_scaled * chebyshev_1 - chebyshev_0;
      chebyshev_0 = chebyshev_1;
      chebyshev_1 = chebyshev_2;
      temp += moments[step2] * chebyshev_2;
    }
    temp *= 2.0;
    temp += moments[0];
    temp *= 2.0 / (PI * V * sqrt(1.0 - energy_scaled * energy_scaled));
    correlation[step1] = temp / Em;
  }
}

// direction = +1: U(+t) |state>
// direction = -1: U(-t) |state>
void evolve(
  int N,
  double Em,
  int direction,
  double time_step_scaled,
  int* NN,
  int* NL,
  double* U,
  double* Hr,
  double* Hi,
  double* sr,
  double* si)
{
  int grid_size = (N - 1) / BLOCK_SIZE_EC + 1;
  double Em_inv = 1.0 / Em;
  double* s0r;
  double* s1r;
  double* s2r;
  double* s0i;
  double* s1i;
  double* s2i;
  hipMalloc((void**)&s0r, sizeof(double) * N);
  hipMalloc((void**)&s0i, sizeof(double) * N);
  hipMalloc((void**)&s1r, sizeof(double) * N);
  hipMalloc((void**)&s1i, sizeof(double) * N);
  hipMalloc((void**)&s2r, sizeof(double) * N);
  hipMalloc((void**)&s2i, sizeof(double) * N);

  // T_0(H) |psi> = |psi>
  gpu_copy_state<<<grid_size, BLOCK_SIZE_EC>>>(N, sr, si, s0r, s0i);

  // T_1(H) |psi> = H |psi>
  gpu_apply_hamiltonian<<<grid_size, BLOCK_SIZE_EC>>>(
    N, Em_inv, NN, NL, U, Hr, Hi, sr, si, s1r, s1i);

  // |final_state> = c_0 * T_0(H) |psi> + c_1 * T_1(H) |psi>
  double bessel_0 = j0(time_step_scaled);
  double bessel_1 = 2.0 * j1(time_step_scaled);
  gpu_chebyshev_01<<<grid_size, BLOCK_SIZE_EC>>>(
    N, s0r, s0i, s1r, s1i, sr, si, bessel_0, bessel_1, direction);

  for (int m = 2; m < 1000000; ++m) {
    double bessel_m = jn(m, time_step_scaled);
    if (bessel_m < 1.0e-15 && bessel_m > -1.0e-15) {
      break;
    }
    bessel_m *= 2.0;
    int label;
    int m_mod_4 = m % 4;
    if (m_mod_4 == 0) {
      label = 1;
    } else if (m_mod_4 == 2) {
      label = 2;
    } else if ((m_mod_4 == 1 && direction == 1) || (m_mod_4 == 3 && direction == -1)) {
      label = 3;
    } else {
      label = 4;
    }
    gpu_chebyshev_2<<<grid_size, BLOCK_SIZE_EC>>>(
      N, Em_inv, NN, NL, U, Hr, Hi, s0r, s0i, s1r, s1i, s2r, s2i, sr, si, bessel_m, label);

    // permute the pointers; do not need to copy the data
    double *temp_real, *temp_imag;
    temp_real = s0r;
    temp_imag = s0i;
    s0r = s1r;
    s0i = s1i;
    s1r = s2r;
    s1i = s2i;
    s2r = temp_real;
    s2i = temp_imag;
  }
  hipFree(s0r);
  hipFree(s0i);
  hipFree(s1r);
  hipFree(s1i);
  hipFree(s2r);
  hipFree(s2i);
}

// direction = +1: [X, U(+t)] |state>
// direction = -1: [U(-t), X] |state>
void evolvex(
  int N,
  double Em,
  int direction,
  double time_step_scaled,
  int* NN,
  int* NL,
  double* U,
  double* Hr,
  double* Hi,
  double* xx,
  double* sr,
  double* si)
{
  int grid_size = (N - 1) / BLOCK_SIZE_EC + 1;
  double Em_inv = 1.0 / Em;
  double* s0r;
  double* s1r;
  double* s2r;
  double* s0i;
  double* s1i;
  double* s2i;
  double* s0xr;
  double* s1xr;
  double* s2xr;
  double* s0xi;
  double* s1xi;
  double* s2xi;
  hipMalloc((void**)&s0r, sizeof(double) * N);
  hipMalloc((void**)&s0i, sizeof(double) * N);
  hipMalloc((void**)&s1r, sizeof(double) * N);
  hipMalloc((void**)&s1i, sizeof(double) * N);
  hipMalloc((void**)&s2r, sizeof(double) * N);
  hipMalloc((void**)&s2i, sizeof(double) * N);
  hipMalloc((void**)&s0xr, sizeof(double) * N);
  hipMalloc((void**)&s0xi, sizeof(double) * N);
  hipMalloc((void**)&s1xr, sizeof(double) * N);
  hipMalloc((void**)&s1xi, sizeof(double) * N);
  hipMalloc((void**)&s2xr, sizeof(double) * N);
  hipMalloc((void**)&s2xi, sizeof(double) * N);

  // T_0(H) |psi> = |psi>
  gpu_copy_state<<<grid_size, BLOCK_SIZE_EC>>>(N, sr, si, s0r, s0i);

  // [X, T_0(H)] |psi> = 0
  gpu_set_zero<<<grid_size, BLOCK_SIZE_EC>>>(N, s0xr, s0xi);

  // T_1(H) |psi> = H |psi>
  gpu_apply_hamiltonian<<<grid_size, BLOCK_SIZE_EC>>>(
    N, Em_inv, NN, NL, U, Hr, Hi, sr, si, s1r, s1i);

  // [X, T_1(H)] |psi> = J |psi>
  gpu_apply_commutator<<<grid_size, BLOCK_SIZE_EC>>>(
    N, Em_inv, NN, NL, Hr, Hi, xx, sr, si, s1xr, s1xi);

  // |final_state> = c_1 * [X, T_1(H)] |psi>
  double bessel_1 = 2.0 * j1(time_step_scaled);
  gpu_chebyshev_1x<<<grid_size, BLOCK_SIZE_EC>>>(N, s1xr, s1xi, sr, si, bessel_1);

  for (int m = 2; m <= 1000000; ++m) {
    double bessel_m = jn(m, time_step_scaled);
    if (bessel_m < 1.0e-15 && bessel_m > -1.0e-15) {
      break;
    }
    bessel_m *= 2.0;
    int label;
    int m_mod_4 = m % 4;
    if (m_mod_4 == 1) {
      label = 3;
    } else if (m_mod_4 == 3) {
      label = 4;
    } else if ((m_mod_4 == 0 && direction == 1) || (m_mod_4 == 2 && direction == -1)) {
      label = 1;
    } else {
      label = 2;
    }

    gpu_chebyshev_2x<<<grid_size, BLOCK_SIZE_EC>>>(
      N,
      Em_inv,
      NN,
      NL,
      U,
      Hr,
      Hi,
      xx,
      s0r,
      s0i,
      s0xr,
      s0xi,
      s1r,
      s1i,
      s1xr,
      s1xi,
      s2r,
      s2i,
      s2xr,
      s2xi,
      sr,
      si,
      bessel_m,
      label);

    // Permute the pointers; do not need to copy the data
    double *temp_real, *temp_imag;
    temp_real = s0r;
    temp_imag = s0i;
    s0r = s1r;
    s0i = s1i;
    s1r = s2r;
    s1i = s2i;
    s2r = temp_real;
    s2i = temp_imag;

    temp_real = s0xr;
    temp_imag = s0xi;
    s0xr = s1xr;
    s0xi = s1xi;
    s1xr = s2xr;
    s1xi = s2xi;
    s2xr = temp_real;
    s2xi = temp_imag;
  }
  hipFree(s0r);
  hipFree(s0i);
  hipFree(s1r);
  hipFree(s1i);
  hipFree(s2r);
  hipFree(s2i);
  hipFree(s0xr);
  hipFree(s0xi);
  hipFree(s1xr);
  hipFree(s1xi);
  hipFree(s2xr);
  hipFree(s2xi);
}

// set up Hamiltonian and related quantities
__global__ void gpu_initialize_model(
  const Box box,
  const int N,
  const int direction,
  const double* x,
  const double* y,
  const double* z,
  const int* NN,
  const int* NL,
  double* U,
  double* Hr,
  double* Hi,
  double* xx)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    int neighbor_number = NN[n1];
    double x1 = x[n1];
    double y1 = y[n1];
    double z1 = z[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = n1 + N * i1;
      int n2 = NL[index];
      double x12 = x[n2] - x1;
      double y12 = y[n2] - y1;
      double z12 = z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);

      if (direction == 1) {
        xx[index] = x12;
      }
      if (direction == 2) {
        xx[index] = y12;
      }
      if (direction == 3) {
        xx[index] = z12;
      }
      // Hr[index] = -2.5; // test
      Hr[index] = -2.5 * 1.425 * 1.425 / (d12 * d12); // a CNT model
      Hi[index] = 0.0;                                // may be used in the future
    }
    U[n1] = 0.0; // may be used in the future
  }
}

// calculate the data and output
void output_data(
  int N,
  int Nm,
  int Ne,
  double Em,
  double* E,
  double V,
  int* NN,
  int* NL,
  double* U,
  double* Hr,
  double* Hi,
  double* sr,
  double* si)
{
  std::vector<double> moments_cpu(Nm);
  GPU_Vector<double> moments_gpu(Nm);
  std::vector<double> data(Ne);

  find_moments_chebyshev(N, Nm, Em, NN, NL, U, Hr, Hi, sr, si, sr, si, moments_gpu.data());
  moments_gpu.copy_to_host(moments_cpu.data());
  apply_damping(Nm, moments_cpu.data());
  perform_chebyshev_summation(Nm, Ne, Em, E, V, moments_cpu.data(), data.data());

  // output the data
  FILE* os = my_fopen("lsqt_dos.out", "a");
  for (int n = 0; n < Ne; ++n)
    fprintf(os, "%25.15e", data[n]);
  fprintf(os, "\n");
  fclose(os);
}

void initialize_state(int N, GPU_Vector<double>& sr, GPU_Vector<double>& si)
{
  std::vector<double> sr_cpu(N);
  std::vector<double> si_cpu(N);
  for (int n = 0; n < N; ++n) {
    double random_phase = rand() / double(RAND_MAX) * 2.0 * PI;
    sr_cpu[n] = cos(random_phase);
    si_cpu[n] = sin(random_phase);
  }
  sr.copy_from_host(sr_cpu.data());
  si.copy_from_host(si_cpu.data());
}
} // namespace

void LSQT::postprocess(Atom& atom, Box& box)
{
  printf("enter the function\n");
  int N = atom.number_of_atoms;
  int M = N * 10;    // number of pairs
  int direction = 1; // transport direction
  int Nm = 1000;     // number of moments
  int Ne = 1001;     // number of energy points
  double Em = 10.1;  // maximum energy
  std::vector<double> E(Ne);
  for (int n = 0; n < Ne; ++n) {
    E[n] = (n - (Ne - 1) / 2) * 0.02;
  }

  GPU_Vector<int> cell_count(N);
  GPU_Vector<int> cell_count_sum(N);
  GPU_Vector<int> cell_contents(N);
  GPU_Vector<int> NN(N);
  GPU_Vector<int> NL(M);
  double* x = atom.position_per_atom.data();
  double* y = atom.position_per_atom.data() + N;
  double* z = atom.position_per_atom.data() + N * 2;
  double V = box.get_volume();

  GPU_Vector<double> xx(M);
  GPU_Vector<double> Hr(M);
  GPU_Vector<double> Hi(M);
  GPU_Vector<double> U(N);
  GPU_Vector<double> sr(N);
  GPU_Vector<double> si(N);

  find_neighbor(
    0,
    N,
    2.1,
    box,
    atom.type,
    atom.position_per_atom,
    cell_count,
    cell_count_sum,
    cell_contents,
    NN,
    NL);

  gpu_initialize_model<<<(N - 1) / 64 + 1, 64>>>(
    box, N, direction, x, y, z, NN.data(), NL.data(), U.data(), Hr.data(), Hi.data(), xx.data());

  initialize_state(N, sr, si);

  output_data(
    N,
    Nm,
    Ne,
    Em,
    E.data(),
    V,
    NN.data(),
    NL.data(),
    U.data(),
    Hr.data(),
    Hi.data(),
    sr.data(),
    si.data());
}
