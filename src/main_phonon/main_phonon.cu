#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "phonon.cuh"
#include "utilities/error.cuh"
#include "utilities/main_common.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void print_welcome_information(void);

int main(int argc, char* argv[])
{
  print_welcome_information();
  print_compile_information();
  print_gpu_information();

  int number_of_inputs = get_number_of_input_directories();

  for (int n = 0; n < number_of_inputs; ++n) {
    char input_directory[200];
    int count = scanf("%s", input_directory);
    PRINT_SCANF_ERROR(count, 1, "Reading error for input directory.");

    print_line_1();
    printf("Run simulation for '%s'.\n", input_directory);
    print_line_2();

    CHECK(hipDeviceSynchronize());
    clock_t time_begin = clock();

    Phonon phonon(input_directory);

    CHECK(hipDeviceSynchronize());
    clock_t time_finish = clock();
    double time_used = (time_finish - time_begin) / double(CLOCKS_PER_SEC);

    print_line_1();
    printf("Time used for '%s' = %f s.\n", input_directory, time_used);
    print_line_2();
  }
  print_line_1();
  printf("Finished running phonon.\n");
  print_line_2();
  return EXIT_SUCCESS;
}

void print_welcome_information(void)
{
  printf("\n");
  printf("***************************************************************\n");
  printf("*                 Welcome to use GPUMD                        *\n");
  printf("*     (Graphics Processing Units Molecular Dynamics)          *\n");
  printf("*                     Version 2.6                             *\n");
  printf("*             This is the phonon executable                   *\n");
  printf("* Authors:                                                    *\n");
  printf("*     Zheyong Fan <brucenju@gmail.com>                        *\n");
  printf("*     Alexander J. Gabourie <gabourie@stanford.edu>           *\n");
  printf("*     Ville Vierimaa                                          *\n");
  printf("*     Mikko Ervasti                                           *\n");
  printf("*     Ari Harju                                               *\n");
  printf("***************************************************************\n");
  printf("\n");
}
