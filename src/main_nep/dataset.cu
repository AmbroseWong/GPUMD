#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "dataset.cuh"
#include "mic.cuh"
#include "parameters.cuh"
#include "utilities/error.cuh"

void Dataset::read_Nc(FILE* fid)
{
  int count = fscanf(fid, "%d", &Nc);
  PRINT_SCANF_ERROR(count, 1, "reading error for number of configurations in train.in.");
  if (Nc > 100000) {
    PRINT_INPUT_ERROR("Number of configurations should <= 100000");
  }
  printf("Number of configurations = %d.\n", Nc);

  structures.resize(Nc);
  h.resize(Nc * 18, Memory_Type::managed);
  pe_ref.resize(Nc, Memory_Type::managed);
  virial_ref.resize(Nc * 6, Memory_Type::managed);
  Na.resize(Nc, Memory_Type::managed);
  Na_original.resize(Nc);
  Na_sum.resize(Nc, Memory_Type::managed);
  error_cpu.resize(Nc);
  error_gpu.resize(Nc);
}

void Dataset::read_Na(FILE* fid)
{
  for (int nc = 0; nc < Nc; ++nc) {
    int count = fscanf(fid, "%d%d", &structures[nc].num_atom, &structures[nc].has_virial);
    PRINT_SCANF_ERROR(count, 2, "reading error for number of atoms and virial flag in train.in.");
    if (structures[nc].num_atom < 1) {
      PRINT_INPUT_ERROR("Number of atoms for one configuration should >= 1.");
    }
    if (structures[nc].num_atom > 1024) {
      PRINT_INPUT_ERROR("Number of atoms for one configuration should <=1024.");
    }
    Na_original[nc] = structures[nc].num_atom;
  }
}

void Dataset::read_energy_virial(FILE* fid, int nc)
{
  if (structures[nc].has_virial) {
    int count = fscanf(
      fid, "%f%f%f%f%f%f%f", &structures[nc].energy, &structures[nc].virial[0],
      &structures[nc].virial[1], &structures[nc].virial[2], &structures[nc].virial[3],
      &structures[nc].virial[4], &structures[nc].virial[5]);
    PRINT_SCANF_ERROR(count, 7, "reading error for energy and virial in train.in.");
    for (int k = 0; k < 6; ++k) {
      structures[nc].virial[k] /= structures[nc].num_atom;
    }
  } else {
    int count = fscanf(fid, "%f", &structures[nc].energy);
    PRINT_SCANF_ERROR(count, 1, "reading error for energy in train.in.");
  }
  structures[nc].energy /= structures[nc].num_atom;
}

static float get_area(const float* a, const float* b)
{
  float s1 = a[1] * b[2] - a[2] * b[1];
  float s2 = a[2] * b[0] - a[0] * b[2];
  float s3 = a[0] * b[1] - a[1] * b[0];
  return sqrt(s1 * s1 + s2 * s2 + s3 * s3);
}

static float get_det(const float* box)
{
  return box[0] * (box[4] * box[8] - box[5] * box[7]) +
         box[1] * (box[5] * box[6] - box[3] * box[8]) +
         box[2] * (box[3] * box[7] - box[4] * box[6]);
}

void Dataset::read_box(FILE* fid, int nc, Parameters& para)
{
  float a[3], b[3], c[3];
  int count = fscanf(
    fid, "%f%f%f%f%f%f%f%f%f", &a[0], &a[1], &a[2], &b[0], &b[1], &b[2], &c[0], &c[1], &c[2]);
  PRINT_SCANF_ERROR(count, 9, "reading error for box in train.in.");

  structures[nc].box_original[0] = a[0];
  structures[nc].box_original[3] = a[1];
  structures[nc].box_original[6] = a[2];
  structures[nc].box_original[1] = b[0];
  structures[nc].box_original[4] = b[1];
  structures[nc].box_original[7] = b[2];
  structures[nc].box_original[2] = c[0];
  structures[nc].box_original[5] = c[1];
  structures[nc].box_original[8] = c[2];

  float det = get_det(structures[nc].box_original);
  float volume = abs(det);
  structures[nc].num_cell_a = int(ceil(2.0f * para.rc_radial / (volume / get_area(b, c))));
  structures[nc].num_cell_b = int(ceil(2.0f * para.rc_radial / (volume / get_area(c, a))));
  structures[nc].num_cell_c = int(ceil(2.0f * para.rc_radial / (volume / get_area(a, b))));

  structures[nc].box[0] = structures[nc].box_original[0] * structures[nc].num_cell_a;
  structures[nc].box[3] = structures[nc].box_original[3] * structures[nc].num_cell_a;
  structures[nc].box[6] = structures[nc].box_original[6] * structures[nc].num_cell_a;
  structures[nc].box[1] = structures[nc].box_original[1] * structures[nc].num_cell_b;
  structures[nc].box[4] = structures[nc].box_original[4] * structures[nc].num_cell_b;
  structures[nc].box[7] = structures[nc].box_original[7] * structures[nc].num_cell_b;
  structures[nc].box[2] = structures[nc].box_original[2] * structures[nc].num_cell_c;
  structures[nc].box[5] = structures[nc].box_original[5] * structures[nc].num_cell_c;
  structures[nc].box[8] = structures[nc].box_original[8] * structures[nc].num_cell_c;

  structures[nc].box[9] =
    structures[nc].box[4] * structures[nc].box[8] - structures[nc].box[5] * structures[nc].box[7];
  structures[nc].box[10] =
    structures[nc].box[2] * structures[nc].box[7] - structures[nc].box[1] * structures[nc].box[8];
  structures[nc].box[11] =
    structures[nc].box[1] * structures[nc].box[5] - structures[nc].box[2] * structures[nc].box[4];
  structures[nc].box[12] =
    structures[nc].box[5] * structures[nc].box[6] - structures[nc].box[3] * structures[nc].box[8];
  structures[nc].box[13] =
    structures[nc].box[0] * structures[nc].box[8] - structures[nc].box[2] * structures[nc].box[6];
  structures[nc].box[14] =
    structures[nc].box[2] * structures[nc].box[3] - structures[nc].box[0] * structures[nc].box[5];
  structures[nc].box[15] =
    structures[nc].box[3] * structures[nc].box[7] - structures[nc].box[4] * structures[nc].box[6];
  structures[nc].box[16] =
    structures[nc].box[1] * structures[nc].box[6] - structures[nc].box[0] * structures[nc].box[7];
  structures[nc].box[17] =
    structures[nc].box[0] * structures[nc].box[4] - structures[nc].box[1] * structures[nc].box[3];

  det *= structures[nc].num_cell_a * structures[nc].num_cell_b * structures[nc].num_cell_c;
  for (int n = 9; n < 18; n++) {
    structures[nc].box[n] /= det;
  }
}

void Dataset::read_force(FILE* fid, int nc, Parameters& para)
{
  structures[nc].num_atom *=
    structures[nc].num_cell_a * structures[nc].num_cell_b * structures[nc].num_cell_c;
  if (structures[nc].num_atom > 1024) {
    PRINT_INPUT_ERROR("Number of atoms for one configuration after replication should <=1024; "
                      "consider using smaller cutoff.");
  }

  structures[nc].atomic_number.resize(structures[nc].num_atom);
  structures[nc].x.resize(structures[nc].num_atom);
  structures[nc].y.resize(structures[nc].num_atom);
  structures[nc].z.resize(structures[nc].num_atom);
  structures[nc].fx.resize(structures[nc].num_atom);
  structures[nc].fy.resize(structures[nc].num_atom);
  structures[nc].fz.resize(structures[nc].num_atom);

  for (int na = 0; na < Na_original[nc]; ++na) {
    int count = fscanf(
      fid, "%d%f%f%f%f%f%f", &structures[nc].atomic_number[na], &structures[nc].x[na],
      &structures[nc].y[na], &structures[nc].z[na], &structures[nc].fx[na], &structures[nc].fy[na],
      &structures[nc].fz[na]);
    PRINT_SCANF_ERROR(count, 7, "reading error for force in train.in.");
    if (para.nep_version == 1) {
      if (structures[nc].atomic_number[na] < 1) {
        PRINT_INPUT_ERROR("Atomic number should > 0.\n");
      }
    } else {
      if (structures[nc].atomic_number[na] < 0) {
        PRINT_INPUT_ERROR("Atom type should >= 0.\n");
      }
    }
  }

  for (int ia = 0; ia < structures[nc].num_cell_a; ++ia) {
    for (int ib = 0; ib < structures[nc].num_cell_b; ++ib) {
      for (int ic = 0; ic < structures[nc].num_cell_c; ++ic) {
        if (ia != 0 || ib != 0 || ic != 0) {
          for (int na = 0; na < Na_original[nc]; ++na) {
            int na_new =
              na + (ia + (ib + ic * structures[nc].num_cell_b) * structures[nc].num_cell_a) *
                     Na_original[nc];
            float delta_x = structures[nc].box_original[0] * ia +
                            structures[nc].box_original[1] * ib +
                            structures[nc].box_original[2] * ic;
            float delta_y = structures[nc].box_original[3] * ia +
                            structures[nc].box_original[4] * ib +
                            structures[nc].box_original[5] * ic;
            float delta_z = structures[nc].box_original[6] * ia +
                            structures[nc].box_original[7] * ib +
                            structures[nc].box_original[8] * ic;
            structures[nc].atomic_number[na_new] = structures[nc].atomic_number[na];
            structures[nc].x[na_new] = structures[nc].x[na] + delta_x;
            structures[nc].y[na_new] = structures[nc].y[na] + delta_y;
            structures[nc].z[na_new] = structures[nc].z[na] + delta_z;
            structures[nc].fx[na_new] = structures[nc].fx[na];
            structures[nc].fy[na_new] = structures[nc].fy[na];
            structures[nc].fz[na_new] = structures[nc].fz[na];
          }
        }
      }
    }
  }
}

void Dataset::read_train_in(char* input_dir, Parameters& para)
{
  print_line_1();
  printf("Started reading train.in.\n");
  print_line_2();

  char file_train[200];
  strcpy(file_train, input_dir);
  strcat(file_train, "/train.in");
  FILE* fid = my_fopen(file_train, "r");

  read_Nc(fid);
  read_Na(fid);
  for (int n = 0; n < Nc; ++n) {
    read_energy_virial(fid, n);
    read_box(fid, n, para);
    read_force(fid, n, para);
  }

  fclose(fid);
}

void Dataset::find_Na()
{
  N = 0;
  max_Na = 0;
  int num_virial_configurations = 0;
  for (int nc = 0; nc < Nc; ++nc) {
    Na[nc] = structures[nc].num_atom;
    Na_sum[nc] = 0;
  }

  for (int nc = 0; nc < Nc; ++nc) {
    N += structures[nc].num_atom;
    if (structures[nc].num_atom > max_Na) {
      max_Na = structures[nc].num_atom;
    }
    num_virial_configurations += structures[nc].has_virial;
  }

  for (int nc = 1; nc < Nc; ++nc) {
    Na_sum[nc] = Na_sum[nc - 1] + Na[nc - 1];
  }

  printf("Total number of atoms = %d.\n", N);
  printf("Number of atoms in the largest configuration = %d.\n", max_Na);
  printf("Number of configurations having virial = %d.\n", num_virial_configurations);
}

void Dataset::initialize_gpu_data(Parameters& para)
{
  if (para.nep_version == 1) {
    atomic_number.resize(N, Memory_Type::managed);
  } else {
    type.resize(N, Memory_Type::managed);
  }

  r.resize(N * 3, Memory_Type::managed);
  force.resize(N * 3, 0.0f, Memory_Type::managed);
  force_ref.resize(N * 3, Memory_Type::managed);
  pe.resize(N, 0.0f, Memory_Type::managed);
  virial.resize(N * 6, 0.0f, Memory_Type::managed);

  for (int n = 0; n < Nc; ++n) {
    pe_ref[n] = structures[n].energy;
    for (int k = 0; k < 6; ++k) {
      virial_ref[k * Nc + n] = structures[n].virial[k];
    }
    for (int k = 0; k < 18; ++k) {
      h[k + n * 18] = structures[n].box[k];
    }
    for (int na = 0; na < structures[n].num_atom; ++na) {
      r[Na_sum[n] + na] = structures[n].x[na];
      r[Na_sum[n] + na + N] = structures[n].y[na];
      r[Na_sum[n] + na + N * 2] = structures[n].z[na];
      force_ref[Na_sum[n] + na] = structures[n].fx[na];
      force_ref[Na_sum[n] + na + N] = structures[n].fy[na];
      force_ref[Na_sum[n] + na + N * 2] = structures[n].fz[na];
    }
  }
}

void Dataset::calculate_types_v1()
{
  int atomic_number_max = 0;
  std::vector<int> types;
  for (int nc = 0; nc < Nc; ++nc) {
    for (int na = 0; na < structures[nc].num_atom; ++na) {
      int atomic_number_tmp = structures[nc].atomic_number[na];
      if (atomic_number_tmp > atomic_number_max) {
        atomic_number_max = atomic_number_tmp;
      }
      bool find_a_new_type = true;
      for (int k = 0; k < types.size(); ++k) {
        if (types[k] == atomic_number_tmp) {
          find_a_new_type = false;
        }
      }
      if (find_a_new_type) {
        types.emplace_back(atomic_number_tmp);
      }
    }
  }

  for (int nc = 0; nc < Nc; ++nc) {
    for (int na = 0; na < structures[nc].num_atom; ++na) {
      atomic_number[Na_sum[nc] + na] =
        sqrt(float(structures[nc].atomic_number[na]) / atomic_number_max);
    }
  }

  num_types = types.size();
}

void Dataset::calculate_types_v2(Parameters& para)
{
  std::vector<int> types;
  for (int nc = 0; nc < Nc; ++nc) {
    for (int na = 0; na < structures[nc].num_atom; ++na) {
      type[Na_sum[nc] + na] = structures[nc].atomic_number[na];
      bool find_a_new_type = true;
      for (int k = 0; k < types.size(); ++k) {
        if (types[k] == structures[nc].atomic_number[na]) {
          find_a_new_type = false;
        }
      }
      if (find_a_new_type) {
        types.emplace_back(structures[nc].atomic_number[na]);
      }
    }
  }
  num_types = types.size();

  if (num_types != para.num_types) {
    PRINT_INPUT_ERROR("mismatching num_types in nep.in and train.in.");
  }
  for (int nc = 0; nc < Nc; ++nc) {
    for (int na = 0; na < structures[nc].num_atom; ++na) {
      if (structures[nc].atomic_number[na] >= num_types) {
        PRINT_INPUT_ERROR("detected atom type (in train.in) >= num_types (in nep.in).");
      }
    }
  }
}

static __global__ void gpu_find_neighbor_number(
  const int N,
  const int* Na,
  const int* Na_sum,
  const float rc2_radial,
  const float rc2_angular,
  const float* __restrict__ box,
  const float* x,
  const float* y,
  const float* z,
  int* NN_radial,
  int* NN_angular)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = box + 18 * blockIdx.x;
    float x1 = x[n1];
    float y1 = y[n1];
    float z1 = z[n1];
    int count_radial = 0;
    int count_angular = 0;
    for (int n2 = N1; n2 < N2; ++n2) {
      if (n2 == n1) {
        continue;
      }
      float x12 = x[n2] - x1;
      float y12 = y[n2] - y1;
      float z12 = z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);
      float distance_square = x12 * x12 + y12 * y12 + z12 * z12;
      if (distance_square < rc2_radial) {
        count_radial++;
      }
      if (distance_square < rc2_angular) {
        count_angular++;
      }
    }
    NN_radial[n1] = count_radial;
    NN_angular[n1] = count_angular;
  }
}

static __global__ void gpu_find_neighbor_list(
  const int N,
  const int* Na,
  const int* Na_sum,
  const float rc2_radial,
  const float rc2_angular,
  const float* __restrict__ box,
  const float* x,
  const float* y,
  const float* z,
  int* NN_radial,
  int* NL_radial,
  int* NN_angular,
  int* NL_angular,
  float* x12_radial,
  float* y12_radial,
  float* z12_radial,
  float* x12_angular,
  float* y12_angular,
  float* z12_angular)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = box + 18 * blockIdx.x;
    float x1 = x[n1];
    float y1 = y[n1];
    float z1 = z[n1];
    int count_radial = 0;
    int count_angular = 0;
    for (int n2 = N1; n2 < N2; ++n2) {
      if (n2 == n1) {
        continue;
      }
      float x12 = x[n2] - x1;
      float y12 = y[n2] - y1;
      float z12 = z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);
      float distance_square = x12 * x12 + y12 * y12 + z12 * z12;
      if (distance_square < rc2_radial) {
        NL_radial[count_radial * N + n1] = n2;
        x12_radial[count_radial * N + n1] = x12;
        y12_radial[count_radial * N + n1] = y12;
        z12_radial[count_radial * N + n1] = z12;
        count_radial++;
      }
      if (distance_square < rc2_angular) {
        NL_angular[count_angular * N + n1] = n2;
        x12_angular[count_angular * N + n1] = x12;
        y12_angular[count_angular * N + n1] = y12;
        z12_angular[count_angular * N + n1] = z12;
        count_angular++;
      }
    }
    NN_radial[n1] = count_radial;
    NN_angular[n1] = count_angular;
  }
}

void Dataset::find_neighbor(Parameters& para)
{
  NN_radial.resize(N, Memory_Type::managed);
  NN_angular.resize(N, Memory_Type::managed);
  float rc2_radial = para.rc_radial * para.rc_radial;
  float rc2_angular = para.rc_angular * para.rc_angular;

  gpu_find_neighbor_number<<<Nc, max_Na>>>(
    N, Na.data(), Na_sum.data(), rc2_radial, rc2_angular, h.data(), r.data(), r.data() + N,
    r.data() + N * 2, NN_radial.data(), NN_angular.data());
  CUDA_CHECK_KERNEL

  CHECK(hipDeviceSynchronize());
  int min_NN_radial = 10000;
  max_NN_radial = -1;
  for (int n = 0; n < N; ++n) {
    if (NN_radial[n] < min_NN_radial) {
      min_NN_radial = NN_radial[n];
    }
    if (NN_radial[n] > max_NN_radial) {
      max_NN_radial = NN_radial[n];
    }
  }
  int min_NN_angular = 10000;
  max_NN_angular = -1;
  for (int n = 0; n < N; ++n) {
    if (NN_angular[n] < min_NN_angular) {
      min_NN_angular = NN_angular[n];
    }
    if (NN_angular[n] > max_NN_angular) {
      max_NN_angular = NN_angular[n];
    }
  }

  printf("Radial descriptor with a cutoff of %g A:\n", para.rc_radial);
  printf("    Minimum number of neighbors for one atom = %d.\n", min_NN_radial);
  printf("    Maximum number of neighbors for one atom = %d.\n", max_NN_radial);
  printf("Angular descriptor with a cutoff of %g A:\n", para.rc_angular);
  printf("    Minimum number of neighbors for one atom = %d.\n", min_NN_angular);
  printf("    Maximum number of neighbors for one atom = %d.\n", max_NN_angular);

  NL_radial.resize(N * max_NN_radial);
  NL_angular.resize(N * max_NN_angular);
  x12_radial.resize(N * max_NN_radial);
  y12_radial.resize(N * max_NN_radial);
  z12_radial.resize(N * max_NN_radial);
  x12_angular.resize(N * max_NN_angular);
  y12_angular.resize(N * max_NN_angular);
  z12_angular.resize(N * max_NN_angular);

  gpu_find_neighbor_list<<<Nc, max_Na>>>(
    N, Na.data(), Na_sum.data(), rc2_radial, rc2_angular, h.data(), r.data(), r.data() + N,
    r.data() + N * 2, NN_radial.data(), NL_radial.data(), NN_angular.data(), NL_angular.data(),
    x12_radial.data(), y12_radial.data(), z12_radial.data(), x12_angular.data(), y12_angular.data(),
    z12_angular.data());
  CUDA_CHECK_KERNEL
}

void Dataset::construct(char* input_dir, Parameters& para)
{
  read_train_in(input_dir, para);
  find_Na();
  initialize_gpu_data(para);

  if (para.nep_version == 1) {
    calculate_types_v1();
  } else {
    calculate_types_v2(para);
  }

  find_neighbor(para);
}

static __global__ void gpu_sum_force_error(
  int N,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_fx_ref,
  float* g_fy_ref,
  float* g_fz_ref,
  float* g_error)
{
  int tid = threadIdx.x;
  int number_of_rounds = (N - 1) / blockDim.x + 1;
  extern __shared__ float s_error[];
  s_error[tid] = 0.0f;
  for (int round = 0; round < number_of_rounds; ++round) {
    int n = tid + round * blockDim.x;
    if (n < N) {
      float dx = g_fx[n] - g_fx_ref[n];
      float dy = g_fy[n] - g_fy_ref[n];
      float dz = g_fz[n] - g_fz_ref[n];
      s_error[tid] += dx * dx + dy * dy + dz * dz;
    }
  }

  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_error[tid] += s_error[tid + offset];
    }
    __syncthreads();
  }

  for (int offset = 32; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_error[tid] += s_error[tid + offset];
    }
    __syncwarp();
  }

  if (tid == 0) {
    g_error[0] = s_error[0];
  }
}

float Dataset::get_rmse_force()
{
  gpu_sum_force_error<<<1, 512, sizeof(float) * 512>>>(
    N, force.data(), force.data() + N, force.data() + N * 2, force_ref.data(), force_ref.data() + N,
    force_ref.data() + N * 2, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), sizeof(float), hipMemcpyDeviceToHost));
  return sqrt(error_cpu[0] / (N * 3));
}

static __global__ void
gpu_sum_pe_error(int* g_Na, int* g_Na_sum, float* g_pe, float* g_pe_ref, float* error_gpu)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int Na = g_Na[bid];
  int offset = g_Na_sum[bid];
  extern __shared__ float s_pe[];
  s_pe[tid] = 0.0f;
  if (tid < Na) {
    int n = offset + tid; // particle index
    s_pe[tid] += g_pe[n];
  }
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_pe[tid] += s_pe[tid + offset];
    }
    __syncthreads();
  }

  for (int offset = 32; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_pe[tid] += s_pe[tid + offset];
    }
    __syncwarp();
  }

  if (tid == 0) {
    float diff = s_pe[0] / Na - g_pe_ref[bid];
    error_gpu[bid] = diff * diff;
  }
}

static int get_block_size(int max_num_atom)
{
  int block_size = 64;
  for (int n = 64; n < 1024; n <<= 1) {
    if (max_num_atom > n) {
      block_size = n << 1;
    }
  }
  return block_size;
}

float Dataset::get_rmse_energy()
{
  int block_size = get_block_size(max_Na);
  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), pe.data(), pe_ref.data(), error_gpu.data());
  int mem = sizeof(float) * Nc;
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  float error_ave = 0.0;
  for (int n = 0; n < Nc; ++n) {
    error_ave += error_cpu[n];
  }
  return sqrt(error_ave / Nc);
}

float Dataset::get_rmse_virial()
{
  int num_virial_configurations = 0;
  for (int n = 0; n < Nc; ++n) {
    if (structures[n].has_virial) {
      ++num_virial_configurations;
    }
  }
  if (num_virial_configurations == 0) {
    return 0.0f;
  }

  float error_ave = 0.0;
  int mem = sizeof(float) * Nc;
  int block_size = get_block_size(max_Na);

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data(), virial_ref.data(), error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (structures[n].has_virial) {
      error_ave += error_cpu[n];
    }
  }

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data() + N, virial_ref.data() + Nc, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (structures[n].has_virial) {
      error_ave += error_cpu[n];
    }
  }

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data() + N * 2, virial_ref.data() + Nc * 2, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (structures[n].has_virial) {
      error_ave += error_cpu[n];
    }
  }

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data() + N * 3, virial_ref.data() + Nc * 3, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (structures[n].has_virial) {
      error_ave += error_cpu[n];
    }
  }

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data() + N * 4, virial_ref.data() + Nc * 4, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (structures[n].has_virial) {
      error_ave += error_cpu[n];
    }
  }

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data() + N * 5, virial_ref.data() + Nc * 5, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (structures[n].has_virial) {
      error_ave += error_cpu[n];
    }
  }

  return sqrt(error_ave / (num_virial_configurations * 6));
}
