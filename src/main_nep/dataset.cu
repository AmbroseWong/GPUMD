#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "dataset.cuh"
#include "parameters.cuh"
#include "utilities/error.cuh"

static void get_inverse(float* cpu_h)
{
  cpu_h[9] = cpu_h[4] * cpu_h[8] - cpu_h[5] * cpu_h[7];
  cpu_h[10] = cpu_h[2] * cpu_h[7] - cpu_h[1] * cpu_h[8];
  cpu_h[11] = cpu_h[1] * cpu_h[5] - cpu_h[2] * cpu_h[4];
  cpu_h[12] = cpu_h[5] * cpu_h[6] - cpu_h[3] * cpu_h[8];
  cpu_h[13] = cpu_h[0] * cpu_h[8] - cpu_h[2] * cpu_h[6];
  cpu_h[14] = cpu_h[2] * cpu_h[3] - cpu_h[0] * cpu_h[5];
  cpu_h[15] = cpu_h[3] * cpu_h[7] - cpu_h[4] * cpu_h[6];
  cpu_h[16] = cpu_h[1] * cpu_h[6] - cpu_h[0] * cpu_h[7];
  cpu_h[17] = cpu_h[0] * cpu_h[4] - cpu_h[1] * cpu_h[3];
  float volume = cpu_h[0] * (cpu_h[4] * cpu_h[8] - cpu_h[5] * cpu_h[7]) +
                 cpu_h[1] * (cpu_h[5] * cpu_h[6] - cpu_h[3] * cpu_h[8]) +
                 cpu_h[2] * (cpu_h[3] * cpu_h[7] - cpu_h[4] * cpu_h[6]);
  for (int n = 9; n < 18; n++) {
    cpu_h[n] /= volume;
  }
}

static void transpose(const int n, const float* h_tmp, float* h)
{
  h[0 + 18 * n] = h_tmp[0];
  h[3 + 18 * n] = h_tmp[1];
  h[6 + 18 * n] = h_tmp[2];
  h[1 + 18 * n] = h_tmp[3];
  h[4 + 18 * n] = h_tmp[4];
  h[7 + 18 * n] = h_tmp[5];
  h[2 + 18 * n] = h_tmp[6];
  h[5 + 18 * n] = h_tmp[7];
  h[8 + 18 * n] = h_tmp[8];
}

void Dataset::read_train_in(char* input_dir)
{
  print_line_1();
  printf("Started reading train.in.\n");
  print_line_2();

  char file_train[200];
  strcpy(file_train, input_dir);
  strcat(file_train, "/train.in");
  FILE* fid = my_fopen(file_train, "r");

  // get Nc
  read_Nc(fid);
  h.resize(Nc * 18, Memory_Type::managed);
  pe_ref.resize(Nc, Memory_Type::managed);
  virial_ref.resize(Nc * 6, Memory_Type::managed);
  Na.resize(Nc, Memory_Type::managed);
  Na_sum.resize(Nc, Memory_Type::managed);
  has_virial.resize(Nc);

  read_Na(fid);
  atomic_number.resize(N, Memory_Type::managed);
  r.resize(N * 3, Memory_Type::managed);
  force.resize(N * 3, Memory_Type::managed);
  force_ref.resize(N * 3, Memory_Type::managed);
  pe.resize(N, Memory_Type::managed);
  virial.resize(N * 6, Memory_Type::managed);

  int atomic_number_max = 0;

  for (int n = 0; n < Nc; ++n) {
    int count;

    // energy, virial
    if (has_virial[n]) {
      count = fscanf(
        fid, "%f%f%f%f%f%f%f", &pe_ref[n], &virial_ref[n + Nc * 0], &virial_ref[n + Nc * 1],
        &virial_ref[n + Nc * 2], &virial_ref[n + Nc * 3], &virial_ref[n + Nc * 4],
        &virial_ref[n + Nc * 5]);
      PRINT_SCANF_ERROR(count, 7, "reading error for train.in.");
      for (int k = 0; k < 6; ++k) {
        virial_ref[n + Nc * k] /= Na[n];
      }
    } else {
      count = fscanf(fid, "%f", &pe_ref[n]);
      PRINT_SCANF_ERROR(count, 1, "reading error for train.in.");
    }
    pe_ref[n] /= Na[n];

    // box (ax, ay, az, bx, by, bz, cx, cy, cz)
    float h_tmp[9];
    for (int k = 0; k < 9; ++k) {
      count = fscanf(fid, "%f", &h_tmp[k]);
      PRINT_SCANF_ERROR(count, 1, "reading error for train.in.");
    }
    transpose(n, h_tmp, h.data());
    get_inverse(h.data() + 18 * n);

    // atomic number, position, force
    for (int k = 0; k < Na[n]; ++k) {
      int atomic_number_tmp = 0;
      count = fscanf(
        fid, "%d%f%f%f%f%f%f", &atomic_number_tmp, &r[Na_sum[n] + k], &r[Na_sum[n] + k + N],
        &r[Na_sum[n] + k + N * 2], &force_ref[Na_sum[n] + k], &force_ref[Na_sum[n] + k + N],
        &force_ref[Na_sum[n] + k + N * 2]);
      PRINT_SCANF_ERROR(count, 7, "reading error for train.in.");
      if (atomic_number_tmp < 1) {
        PRINT_INPUT_ERROR("Atomic number should > 0.\n");
      } else {
        atomic_number[Na_sum[n] + k] = atomic_number_tmp;
        if (atomic_number_tmp > atomic_number_max) {
          atomic_number_max = atomic_number_tmp;
        }
      }
    }
  }

  fclose(fid);

  // normalize the atomic number by the largest one
  for (int n = 0; n < N; ++n) {
    atomic_number[n] /= atomic_number_max;
  }
}

void Dataset::read_Nc(FILE* fid)
{
  int count = fscanf(fid, "%d", &Nc);
  PRINT_SCANF_ERROR(count, 1, "reading error for xyz.in.");
  if (Nc < 10) {
    PRINT_INPUT_ERROR("Number of configurations should >= 10");
  }
  if (Nc > 100000) {
    PRINT_INPUT_ERROR("Number of configurations should <= 100000");
  }
  printf("Number of configurations = %d.\n", Nc);
}

void Dataset::read_Na(FILE* fid)
{
  N = 0;
  max_Na = 0;
  num_virial_configurations = 0;
  for (int nc = 0; nc < Nc; ++nc) {
    Na_sum[nc] = 0;
  }

  for (int nc = 0; nc < Nc; ++nc) {
    int count = fscanf(fid, "%d%d", &Na[nc], &has_virial[nc]);
    PRINT_SCANF_ERROR(count, 2, "reading error for train.in.");
    N += Na[nc];
    if (Na[nc] > max_Na) {
      max_Na = Na[nc];
    }
    if (Na[nc] < 2) {
      PRINT_INPUT_ERROR("Number of atoms for one configuration should >= 2.");
    }
    if (Na[nc] > 1024) {
      PRINT_INPUT_ERROR("Number of atoms for one configuration should <=1024.");
    }
    num_virial_configurations += has_virial[nc];
  }

  for (int nc = 1; nc < Nc; ++nc) {
    Na_sum[nc] = Na_sum[nc - 1] + Na[nc - 1];
  }

  printf("Total number of atoms = %d.\n", N);
  printf("Number of atoms in the largest configuration = %d.\n", max_Na);
  printf("Number of configurations having virial = %d.\n", num_virial_configurations);
}

static __global__ void gpu_sum_force_error(
  int N,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_fx_ref,
  float* g_fy_ref,
  float* g_fz_ref,
  float* g_error)
{
  int tid = threadIdx.x;
  int number_of_rounds = (N - 1) / blockDim.x + 1;
  extern __shared__ float s_error[];
  s_error[tid] = 0.0f;
  for (int round = 0; round < number_of_rounds; ++round) {
    int n = tid + round * blockDim.x;
    if (n < N) {
      float dx = g_fx[n] - g_fx_ref[n];
      float dy = g_fy[n] - g_fy_ref[n];
      float dz = g_fz[n] - g_fz_ref[n];
      s_error[tid] += dx * dx + dy * dy + dz * dz;
    }
  }

  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_error[tid] += s_error[tid + offset];
    }
    __syncthreads();
  }

  for (int offset = 32; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_error[tid] += s_error[tid + offset];
    }
    __syncwarp();
  }

  if (tid == 0) {
    g_error[0] = s_error[0];
  }
}

float Dataset::get_fitness_force(void)
{
  gpu_sum_force_error<<<1, 512, sizeof(float) * 512>>>(
    N, force.data(), force.data() + N, force.data() + N * 2, force_ref.data(), force_ref.data() + N,
    force_ref.data() + N * 2, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), sizeof(float), hipMemcpyDeviceToHost));
  return sqrt(error_cpu[0] / (N * 3));
}

static __global__ void
gpu_sum_pe_error(int* g_Na, int* g_Na_sum, float* g_pe, float* g_pe_ref, float* error_gpu)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int Na = g_Na[bid];
  int offset = g_Na_sum[bid];
  extern __shared__ float s_pe[];
  s_pe[tid] = 0.0f;
  if (tid < Na) {
    int n = offset + tid; // particle index
    s_pe[tid] += g_pe[n];
  }
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_pe[tid] += s_pe[tid + offset];
    }
    __syncthreads();
  }

  for (int offset = 32; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_pe[tid] += s_pe[tid + offset];
    }
    __syncwarp();
  }

  if (tid == 0) {
    float diff = s_pe[0] / Na - g_pe_ref[bid];
    error_gpu[bid] = diff * diff;
  }
}

static int get_block_size(int max_num_atom)
{
  int block_size = 64;
  for (int n = 64; n < 1024; n <<= 1) {
    if (max_num_atom > n) {
      block_size = n << 1;
    }
  }
  return block_size;
}

float Dataset::get_fitness_energy(void)
{
  int block_size = get_block_size(max_Na);
  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), pe.data(), pe_ref.data(), error_gpu.data());
  int mem = sizeof(float) * Nc;
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  float error_ave = 0.0;
  for (int n = 0; n < Nc; ++n) {
    error_ave += error_cpu[n];
  }
  return sqrt(error_ave / Nc);
}

float Dataset::get_fitness_stress(void)
{
  if (num_virial_configurations == 0) {
    return 0.0f;
  }

  float error_ave = 0.0;
  int mem = sizeof(float) * Nc;
  int block_size = get_block_size(max_Na);

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data(), virial_ref.data(), error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (has_virial[n]) {
      error_ave += error_cpu[n];
    }
  }

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data() + N, virial_ref.data() + Nc, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (has_virial[n]) {
      error_ave += error_cpu[n];
    }
  }

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data() + N * 2, virial_ref.data() + Nc * 2, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (has_virial[n]) {
      error_ave += error_cpu[n];
    }
  }

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data() + N * 3, virial_ref.data() + Nc * 3, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (has_virial[n]) {
      error_ave += error_cpu[n];
    }
  }

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data() + N * 4, virial_ref.data() + Nc * 4, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (has_virial[n]) {
      error_ave += error_cpu[n];
    }
  }

  gpu_sum_pe_error<<<Nc, block_size, sizeof(float) * block_size>>>(
    Na.data(), Na_sum.data(), virial.data() + N * 5, virial_ref.data() + Nc * 5, error_gpu.data());
  CHECK(hipMemcpy(error_cpu.data(), error_gpu.data(), mem, hipMemcpyDeviceToHost));
  for (int n = 0; n < Nc; ++n) {
    if (has_virial[n]) {
      error_ave += error_cpu[n];
    }
  }

  return sqrt(error_ave / (num_virial_configurations * 6));
}
