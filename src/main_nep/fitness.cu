#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Get the fitness
------------------------------------------------------------------------------*/

#include "fitness.cuh"
#include "nep.cuh"
#include "parameters.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_vector.cuh"
#include <algorithm>
#include <vector>

Fitness::Fitness(char* input_dir, Parameters& para)
{
  training_set.read_train_in(input_dir, para);
  potential.reset(new NEP2(para, training_set));

  char file_train_out[200];
  strcpy(file_train_out, input_dir);
  strcat(file_train_out, "/train.out");
  fid_train_out = my_fopen(file_train_out, "w");

  char file_potential_out[200];
  strcpy(file_potential_out, input_dir);
  strcat(file_potential_out, "/potential.out");
  fid_potential_out = my_fopen(file_potential_out, "w");
}

Fitness::~Fitness()
{
  fclose(fid_train_out);
  fclose(fid_potential_out);
}

void Fitness::compute(
  const int generation, Parameters& para, const float* population, float* fitness)
{
  const int num_of_batches = (training_set.Nc - 1) / para.batch_size + 1;
  const int batch_id = (generation / 1) % num_of_batches;
  const int configuration_start = batch_id * para.batch_size;
  const int configuration_end = std::min(training_set.Nc, configuration_start + para.batch_size);
  for (int n = 0; n < para.population_size; ++n) {
    const float* individual = population + n * para.number_of_variables;
    potential->find_force(configuration_start, configuration_end, individual, training_set);
    fitness[n + 0 * para.population_size] =
      training_set.get_rmse_energy(configuration_start, configuration_end) /
      training_set.energy_std;
    fitness[n + 1 * para.population_size] =
      training_set.get_rmse_force(
        training_set.Na_sum[configuration_start],
        training_set.Na_sum[configuration_end - 1] + training_set.Na[configuration_end - 1]) /
      training_set.force_std;
    fitness[n + 2 * para.population_size] =
      training_set.get_rmse_virial(configuration_start, configuration_end) /
      training_set.virial_std;
  }
}

void Fitness::report_error(
  char* input_dir,
  Parameters& para,
  const int generation,
  const float loss_total, // not used, but keep for a while
  const float loss_L1,
  const float loss_L2,
  const float loss_energy, // not used, but keep for a while
  const float loss_force,  // not used, but keep for a while
  const float loss_virial, // not used, but keep for a while
  const float* elite)
{
  if (0 == (generation + 1) % 1000) {
    for (int m = 0; m < para.number_of_variables; ++m) {
      fprintf(fid_potential_out, "%g ", elite[m]);
    }
    fprintf(fid_potential_out, "\n");
    fflush(fid_potential_out);

    // TODO: change to use test errors
    potential->find_force(0, training_set.Nc, elite, training_set);
    float rmse_energy_train = training_set.get_rmse_energy(0, training_set.Nc);
    float rmse_force_train = training_set.get_rmse_force(0, training_set.N);
    float rmse_virial_train = training_set.get_rmse_virial(0, training_set.Nc);
    float total_loss = loss_L1 + loss_L2 + rmse_energy_train + rmse_force_train + rmse_virial_train;

    printf(
      "%-8d%-11.5f%-13.5f%-13.5f%-13.5f\n", generation + 1, total_loss, rmse_energy_train,
      rmse_force_train, rmse_virial_train);
    fflush(stdout);
    fprintf(
      fid_train_out, "%-8d%-11.5f%-13.5f%-13.5f%-13.5f\n", generation + 1, total_loss,
      rmse_energy_train, rmse_force_train, rmse_virial_train);
    fflush(fid_train_out);

    // Synchronize
    CHECK(hipDeviceSynchronize());

    // update force.out
    char file_force[200];
    strcpy(file_force, input_dir);
    strcat(file_force, "/force.out");
    FILE* fid_force = my_fopen(file_force, "w");
    for (int n = 0; n < training_set.N; ++n) {
      fprintf(
        fid_force, "%g %g %g %g %g %g\n", training_set.force[n],
        training_set.force[n + training_set.N], training_set.force[n + training_set.N * 2],
        training_set.force_ref[n], training_set.force_ref[n + training_set.N],
        training_set.force_ref[n + training_set.N * 2]);
    }
    fclose(fid_force);

    // update energy.out
    char file_energy[200];
    strcpy(file_energy, input_dir);
    strcat(file_energy, "/energy.out");
    FILE* fid_energy = my_fopen(file_energy, "w");
    predict_energy_or_stress(fid_energy, training_set.pe.data(), training_set.pe_ref.data());
    fclose(fid_energy);

    // update virial.out
    char file_virial[200];
    strcpy(file_virial, input_dir);
    strcat(file_virial, "/virial.out");
    FILE* fid_virial = my_fopen(file_virial, "w");
    predict_energy_or_stress(
      fid_virial, training_set.virial.data(), training_set.virial_ref.data());
    predict_energy_or_stress(
      fid_virial, training_set.virial.data() + training_set.N,
      training_set.virial_ref.data() + training_set.Nc);
    predict_energy_or_stress(
      fid_virial, training_set.virial.data() + training_set.N * 2,
      training_set.virial_ref.data() + training_set.Nc * 2);
    predict_energy_or_stress(
      fid_virial, training_set.virial.data() + training_set.N * 3,
      training_set.virial_ref.data() + training_set.Nc * 3);
    predict_energy_or_stress(
      fid_virial, training_set.virial.data() + training_set.N * 4,
      training_set.virial_ref.data() + training_set.Nc * 4);
    predict_energy_or_stress(
      fid_virial, training_set.virial.data() + training_set.N * 5,
      training_set.virial_ref.data() + training_set.Nc * 5);
    fclose(fid_virial);
  }
}

void Fitness::predict_energy_or_stress(FILE* fid, float* data, float* ref)
{
  for (int nc = 0; nc < training_set.Nc; ++nc) {
    int offset = training_set.Na_sum[nc];
    float data_nc = 0.0;
    for (int m = 0; m < training_set.Na[nc]; ++m) {
      data_nc += data[offset + m];
    }
    fprintf(fid, "%g %g\n", data_nc / training_set.Na[nc], ref[nc]);
  }
}
