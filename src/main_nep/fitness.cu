/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Get the fitness
------------------------------------------------------------------------------*/

#include "fitness.cuh"
#include "neighbor.cuh"
#include "nep.cuh"
#include "nep2.cuh"
#include "parameters.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_vector.cuh"
#include <vector>

Fitness::Fitness(char* input_dir, Parameters& para)
{
  potential.reset(new NEP2(para.rc, para.num_neurons, para.n_max, para.L_max));
  neighbor.cutoff = para.rc;

  training_set.read_train_in(input_dir);
  neighbor.compute(
    training_set.Nc, training_set.N, training_set.max_Na, training_set.Na.data(),
    training_set.Na_sum.data(), training_set.r.data(), training_set.h.data());
  potential->initialize(training_set.N, training_set.max_Na);
  training_set.error_cpu.resize(training_set.Nc);
  training_set.error_gpu.resize(training_set.Nc);

  char file_train_out[200];
  strcpy(file_train_out, input_dir);
  strcat(file_train_out, "/train.out");
  fid_train_out = my_fopen(file_train_out, "w");
}

void Fitness::compute(Parameters& para, const float* population, float* fitness)
{
  for (int n = 0; n < para.population_size; ++n) {
    const float* individual = population + n * para.number_of_variables;
    potential->update_potential(individual);
    potential->find_force(
      training_set.Nc, training_set.N, training_set.Na.data(), training_set.Na_sum.data(),
      training_set.max_Na, training_set.atomic_number.data(), training_set.h.data(), &neighbor,
      training_set.r.data(), training_set.force, training_set.virial, training_set.pe);
    fitness[n + 0 * para.population_size] =
      para.weight_energy * training_set.get_fitness_energy() / training_set.potential_std;
    fitness[n + 1 * para.population_size] =
      para.weight_force * training_set.get_fitness_force() / training_set.force_std;
    fitness[n + 2 * para.population_size] =
      para.weight_stress * training_set.get_fitness_stress() / training_set.virial_std;
  }
}

void Fitness::report_error(
  char* input_dir,
  Parameters& para,
  const int generation,
  const float loss_total,
  const float loss_L1,
  const float loss_L2,
  const float* elite)
{
  if (0 == (generation + 1) % 100) {
    // save a potential
    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/potential.out");
    FILE* fid = my_fopen(file, "w");
    fprintf(fid, "nep 1\n");
    fprintf(fid, "cutoff %g\n", para.rc);
    fprintf(fid, "num_neurons %d\n", para.num_neurons);
    fprintf(fid, "n_max %d\n", para.n_max);
    fprintf(fid, "l_max %d\n", para.L_max);
    for (int m = 0; m < para.number_of_variables; ++m) {
      fprintf(fid, "%g ", elite[m]);
    }
    fprintf(fid, "\n");
    fclose(fid);

    // calculate force, energy, and virial
    potential->update_potential(elite);
    potential->find_force(
      training_set.Nc, training_set.N, training_set.Na.data(), training_set.Na_sum.data(),
      training_set.max_Na, training_set.atomic_number.data(), training_set.h.data(), &neighbor,
      training_set.r.data(), training_set.force, training_set.virial, training_set.pe);

    // report errors
    float rmse_energy = training_set.get_fitness_energy();
    float rmse_force = training_set.get_fitness_force();
    float rmse_virial = training_set.get_fitness_stress();
    printf(
      "%-7d%-10.2f%-10.2f%-10.2f%-10.2f%-10.2f%-10.2f%-12.2f%-10.2f%-12.2f\n", generation + 1,
      loss_total * 100.0f, loss_L1 * 100.0f, loss_L2 * 100.0f,
      rmse_energy / training_set.potential_std * 100.0f,
      rmse_force / training_set.force_std * 100.0f, rmse_virial / training_set.virial_std * 100.0f,
      rmse_energy * 1000.0f, rmse_force * 1000.0f, rmse_virial * 1000.0f);
    fprintf(
      fid_train_out, "%-7d%-10.2f%-10.2f%-10.2f%-10.2f%-10.2f%-10.2f%-12.2f%-10.2f%-12.2f\n",
      generation + 1, loss_total * 100.0f, loss_L1 * 100.0f, loss_L2 * 100.0f,
      rmse_energy / training_set.potential_std * 100.0f,
      rmse_force / training_set.force_std * 100.0f, rmse_virial / training_set.virial_std * 100.0f,
      rmse_energy * 1000.0f, rmse_force * 1000.0f, rmse_virial * 1000.0f);

    // Synchronize
    CHECK(hipDeviceSynchronize());

    // update force.out
    char file_force[200];
    strcpy(file_force, input_dir);
    strcat(file_force, "/force.out");
    FILE* fid_force = my_fopen(file_force, "w");
    for (int n = 0; n < training_set.N; ++n) {
      fprintf(
        fid_force, "%g %g %g %g %g %g\n", training_set.force[n],
        training_set.force[n + training_set.N], training_set.force[n + training_set.N * 2],
        training_set.force_ref[n], training_set.force_ref[n + training_set.N],
        training_set.force_ref[n + training_set.N * 2]);
    }
    fclose(fid_force);

    // update energy.out
    char file_energy[200];
    strcpy(file_energy, input_dir);
    strcat(file_energy, "/energy.out");
    FILE* fid_energy = my_fopen(file_energy, "w");
    predict_energy_or_stress(fid_energy, training_set.pe.data(), training_set.pe_ref.data());
    fclose(fid_energy);

    // update virial.out
    char file_virial[200];
    strcpy(file_virial, input_dir);
    strcat(file_virial, "/virial.out");
    FILE* fid_virial = my_fopen(file_virial, "w");
    predict_energy_or_stress(
      fid_virial, training_set.virial.data(), training_set.virial_ref.data());
    predict_energy_or_stress(
      fid_virial, training_set.virial.data() + training_set.N,
      training_set.virial_ref.data() + training_set.Nc);
    predict_energy_or_stress(
      fid_virial, training_set.virial.data() + training_set.N * 2,
      training_set.virial_ref.data() + training_set.Nc * 2);
    predict_energy_or_stress(
      fid_virial, training_set.virial.data() + training_set.N * 3,
      training_set.virial_ref.data() + training_set.Nc * 3);
    predict_energy_or_stress(
      fid_virial, training_set.virial.data() + training_set.N * 4,
      training_set.virial_ref.data() + training_set.Nc * 4);
    predict_energy_or_stress(
      fid_virial, training_set.virial.data() + training_set.N * 5,
      training_set.virial_ref.data() + training_set.Nc * 5);
    fclose(fid_virial);
  }
}

void Fitness::predict_energy_or_stress(FILE* fid, float* data, float* ref)
{
  for (int nc = 0; nc < training_set.Nc; ++nc) {
    int offset = training_set.Na_sum[nc];
    float data_nc = 0.0;
    for (int m = 0; m < training_set.Na[nc]; ++m) {
      data_nc += data[offset + m];
    }
    fprintf(fid, "%g %g\n", data_nc / training_set.Na[nc], ref[nc]);
  }
}
