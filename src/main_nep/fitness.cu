#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Get the fitness
------------------------------------------------------------------------------*/

#include "fitness.cuh"
#include "nep.cuh"
#include "parameters.cuh"
#include "structure.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_vector.cuh"
#include <algorithm>
#include <chrono>
#include <random>
#include <vector>

Fitness::Fitness(char* input_dir, Parameters& para)
{
  print_line_1();
  printf("Started reading train.in.\n");
  print_line_2();
  std::vector<Structure> structures_train;
  read_structures(true, input_dir, para, structures_train);
  num_batches = (structures_train.size() - 1) / para.batch_size + 1;
  printf("Number of batches = %d\n", num_batches);
  int batch_size_old = para.batch_size;
  para.batch_size = (structures_train.size() - 1) / num_batches + 1;
  if (batch_size_old != para.batch_size) {
    printf("Hello, I changed the batch_size from %d to %d.\n", batch_size_old, para.batch_size);
  }
  train_set.resize(num_batches);
  for (int batch_id = 0; batch_id < num_batches; ++batch_id) {
    printf("\nBatch %d:\n", batch_id);
    int n1 = batch_id * para.batch_size;
    int n2 = std::min(int(structures_train.size()), n1 + para.batch_size);
    printf("Number of configurations = %d.\n", n2 - n1);
    train_set[batch_id].construct(input_dir, para, structures_train, n1, n2);
  }

  print_line_1();
  printf("Started reading test.in.\n");
  print_line_2();
  std::vector<Structure> structures_test;
  read_structures(false, input_dir, para, structures_test);
  test_set.construct(input_dir, para, structures_test, 0, structures_test.size());

  int N = test_set.N;
  int N_times_max_NN_radial = test_set.N * test_set.max_NN_radial;
  int N_times_max_NN_angular = test_set.N * test_set.max_NN_angular;
  for (int n = 0; n < num_batches; ++n) {
    if (train_set[n].N > N) {
      N = train_set[n].N;
    };
    if (train_set[n].N * train_set[n].max_NN_radial > N_times_max_NN_radial) {
      N_times_max_NN_radial = train_set[n].N * train_set[n].max_NN_radial;
    };
    if (train_set[n].N * train_set[n].max_NN_angular > N_times_max_NN_angular) {
      N_times_max_NN_angular = train_set[n].N * train_set[n].max_NN_angular;
    };
  }
  potential.reset(new NEP2(input_dir, para, N, N_times_max_NN_radial, N_times_max_NN_angular));

  char file_loss_out[200];
  strcpy(file_loss_out, input_dir);
  strcat(file_loss_out, "/loss.out");
  fid_loss_out = my_fopen(file_loss_out, "w");
}

Fitness::~Fitness() { fclose(fid_loss_out); }

void Fitness::compute(
  const int generation, Parameters& para, const float* population, float* fitness)
{
  int batch_id = generation % num_batches;
  for (int n = 0; n < para.population_size; ++n) {
    const float* individual = population + n * para.number_of_variables;
    potential->find_force(para, individual, train_set[batch_id]);
    fitness[n + 0 * para.population_size] = train_set[batch_id].get_rmse_energy();
    fitness[n + 1 * para.population_size] = train_set[batch_id].get_rmse_force();
    fitness[n + 2 * para.population_size] = train_set[batch_id].get_rmse_virial();
  }
}

void Fitness::predict_energy_or_stress(FILE* fid, float* data, float* ref)
{
  for (int nc = 0; nc < test_set.Nc; ++nc) {
    int offset = test_set.Na_sum[nc];
    float data_nc = 0.0f;
    for (int m = 0; m < test_set.Na[nc]; ++m) {
      data_nc += data[offset + m];
    }
    fprintf(fid, "%g %g\n", data_nc / test_set.Na[nc], ref[nc]);
  }
}

void Fitness::report_error(
  char* input_dir,
  Parameters& para,
  const int generation,
  const float loss_total,
  const float loss_L1,
  const float loss_L2,
  const float loss_energy,
  const float loss_force,
  const float loss_virial,
  const float* elite)
{
  if (0 == (generation + 1) % 100) {

    potential->find_force(para, elite, test_set);
    float rmse_energy_test = test_set.get_rmse_energy();
    float rmse_force_test = test_set.get_rmse_force();
    float rmse_virial_test = test_set.get_rmse_virial();

    char file_nep[200];
    strcpy(file_nep, input_dir);
    strcat(file_nep, "/nep.txt");
    FILE* fid_nep = my_fopen(file_nep, "w");

    fprintf(fid_nep, "nep %d\n", para.num_types);
    fprintf(fid_nep, "cutoff %g %g\n", para.rc_radial, para.rc_angular);
    fprintf(fid_nep, "n_max %d %d\n", para.n_max_radial, para.n_max_angular);
    fprintf(fid_nep, "l_max %d\n", para.L_max);
    fprintf(fid_nep, "ANN %d %d\n", para.num_neurons1, 0);
    for (int m = 0; m < para.number_of_variables; ++m) {
      fprintf(fid_nep, "%15.7e ", elite[m]);
    }
    fprintf(fid_nep, "\n");
    para.q_scaler_gpu.copy_to_host(para.q_scaler_cpu.data());
    para.q_min_gpu.copy_to_host(para.q_min_cpu.data());
    for (int d = 0; d < para.q_scaler_cpu.size(); ++d) {
      fprintf(fid_nep, "%15.7e %15.7e\n", para.q_scaler_cpu[d], para.q_min_cpu[d]);
    }
    fclose(fid_nep);

    printf(
      "%-8d%-11.5f%-11.5f%-11.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f\n", generation + 1,
      loss_total, loss_L1, loss_L2, loss_energy, loss_force, loss_virial, rmse_energy_test,
      rmse_force_test, rmse_virial_test);
    fflush(stdout);
    fprintf(
      fid_loss_out, "%-8d%-11.5f%-11.5f%-11.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f%-13.5f\n",
      generation + 1, loss_total, loss_L1, loss_L2, loss_energy, loss_force, loss_virial,
      rmse_energy_test, rmse_force_test, rmse_virial_test);
    fflush(fid_loss_out);

    update_energy_force_virial(input_dir);
  }
}

void Fitness::update_energy_force_virial(char* input_dir)
{
  test_set.energy.copy_to_host(test_set.energy_cpu.data());
  test_set.virial.copy_to_host(test_set.virial_cpu.data());
  test_set.force.copy_to_host(test_set.force_cpu.data());

  // update force.out
  char file_force[200];
  strcpy(file_force, input_dir);
  strcat(file_force, "/force.out");
  FILE* fid_force = my_fopen(file_force, "w");
  for (int nc = 0; nc < test_set.Nc; ++nc) {
    int offset = test_set.Na_sum[nc];
    for (int m = 0; m < test_set.structures[nc].num_atom_original; ++m) {
      int n = offset + m;
      fprintf(
        fid_force, "%g %g %g %g %g %g\n", test_set.force_cpu[n], test_set.force_cpu[n + test_set.N],
        test_set.force_cpu[n + test_set.N * 2], test_set.force_ref_cpu[n],
        test_set.force_ref_cpu[n + test_set.N], test_set.force_ref_cpu[n + test_set.N * 2]);
    }
  }
  fclose(fid_force);

  // update energy.out
  char file_energy[200];
  strcpy(file_energy, input_dir);
  strcat(file_energy, "/energy.out");
  FILE* fid_energy = my_fopen(file_energy, "w");
  predict_energy_or_stress(fid_energy, test_set.energy_cpu.data(), test_set.energy_ref_cpu.data());
  fclose(fid_energy);

  // update virial.out
  char file_virial[200];
  strcpy(file_virial, input_dir);
  strcat(file_virial, "/virial.out");
  FILE* fid_virial = my_fopen(file_virial, "w");
  predict_energy_or_stress(fid_virial, test_set.virial_cpu.data(), test_set.virial_ref_cpu.data());

  predict_energy_or_stress(
    fid_virial, test_set.virial_cpu.data() + test_set.N,
    test_set.virial_ref_cpu.data() + test_set.Nc);

  predict_energy_or_stress(
    fid_virial, test_set.virial_cpu.data() + test_set.N * 2,
    test_set.virial_ref_cpu.data() + test_set.Nc * 2);

  predict_energy_or_stress(
    fid_virial, test_set.virial_cpu.data() + test_set.N * 3,
    test_set.virial_ref_cpu.data() + test_set.Nc * 3);

  predict_energy_or_stress(
    fid_virial, test_set.virial_cpu.data() + test_set.N * 4,
    test_set.virial_ref_cpu.data() + test_set.Nc * 4);

  predict_energy_or_stress(
    fid_virial, test_set.virial_cpu.data() + test_set.N * 5,
    test_set.virial_ref_cpu.data() + test_set.Nc * 5);

  fclose(fid_virial);
}

void Fitness::test(char* input_dir, Parameters& para, const float* elite)
{
  potential->find_force(para, elite, test_set);
  float rmse_energy_test = test_set.get_rmse_energy();
  float rmse_force_tes = test_set.get_rmse_force();
  float rmse_virial_tes = test_set.get_rmse_virial();
  printf("Energy RMSE = %g\n", rmse_energy_test);
  printf("Force RMSE = %g\n", rmse_force_tes);
  printf("Virial RMSE = %g\n", rmse_virial_tes);
  update_energy_force_virial(input_dir);
}
