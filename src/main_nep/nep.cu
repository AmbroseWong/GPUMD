#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The neuroevolution potential (NEP)
Ref: Zheyong Fan et al., in preparison.
------------------------------------------------------------------------------*/

#include "error.cuh"
#include "gpu_vector.cuh"
#include "mic.cuh"
#include "neighbor.cuh"
#include "nep.cuh"

const int NUM_OF_ABC = 10;               // 1 + 3 + 6 for L_max = 2
const int SIZE_BOX_AND_INVERSE_BOX = 18; // (3 * 3) * 2

NEP::NEP(
  int num_neurons_2b,
  float r1_2b,
  float r2_2b,
  int num_neurons_3b,
  float r1_3b,
  float r2_3b,
  int num_neurons_mb,
  int n_max,
  int L_max)
{
  // 2body
  ann2b.dim = 1;
  ann2b.num_neurons_per_layer = num_neurons_2b;
  para2b.r1 = r1_2b;
  para2b.r2 = r2_2b;
  para2b.r2inv = 1.0f / para2b.r2;
  para2b.pi_factor = 3.1415927f / (r2_2b - r1_2b);
  // 3body
  ann3b.dim = 3;
  ann3b.num_neurons_per_layer = num_neurons_3b;
  para3b.r1 = r1_3b;
  para3b.r2 = r2_3b;
  para3b.r2inv = 1.0f / para3b.r2;
  para3b.pi_factor = 3.1415927f / (r2_3b - r1_3b);
  // manybody
  paramb.n_max = n_max;
  paramb.L_max = L_max;
  paramb.r1 = 0.0f;  // inner cutoff for manybody is fixed to 0
  paramb.r2 = r2_2b; // manybody has the same outer cutoff as twobody
  paramb.r2inv = 1.0f / paramb.r2;
  paramb.pi_factor = 3.1415927f / (paramb.r2 - paramb.r1);
  paramb.delta_r = paramb.r2 / paramb.n_max;
  annmb.dim = (n_max + 1) * (L_max + 1);
  annmb.num_neurons_per_layer = num_neurons_mb;
};

void NEP::initialize(int N, int MAX_ATOM_NUMBER)
{
  if (ann3b.num_neurons_per_layer > 0) {
    nep_data.NN3b.resize(N);
    nep_data.NL3b.resize(N * MAX_ATOM_NUMBER);
  }
  if (annmb.num_neurons_per_layer > 0) {
    nep_data.Fp.resize(N * annmb.dim);
    nep_data.sum_fxyz.resize(N * (paramb.n_max + 1) * NUM_OF_ABC);
  }
  if (ann3b.num_neurons_per_layer > 0 || annmb.num_neurons_per_layer > 0) {
    nep_data.f12x.resize(N * MAX_ATOM_NUMBER);
    nep_data.f12y.resize(N * MAX_ATOM_NUMBER);
    nep_data.f12z.resize(N * MAX_ATOM_NUMBER);
  }
}

void NEP::update_potential(const float* parameters)
{
  if (ann2b.num_neurons_per_layer > 0) {
    int offset = 0;
    update_potential(parameters, offset, ann2b);
  }
  if (ann3b.num_neurons_per_layer > 0) {
    int offset = 0;
    if (ann2b.num_neurons_per_layer > 0) {
      offset += ann2b.num_neurons_per_layer * (ann2b.num_neurons_per_layer + 4) + 1;
    }
    update_potential(parameters, offset, ann3b);
  }
  if (annmb.num_neurons_per_layer > 0) {
    int offset = 0;
    if (ann2b.num_neurons_per_layer > 0) {
      offset += ann2b.num_neurons_per_layer * (ann2b.num_neurons_per_layer + 4) + 1;
    }
    if (ann3b.num_neurons_per_layer > 0) {
      offset += ann3b.num_neurons_per_layer * (ann3b.num_neurons_per_layer + 6) + 1;
    }
    update_potential(parameters, offset, annmb);
  }
}

void NEP::update_potential(const float* parameters, const int offset, NEP::ANN& ann)
{
  for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
    for (int d = 0; d < ann.dim; ++d) {
      ann.w0[n * ann.dim + d] = parameters[n * ann.dim + d + offset];
    }
    ann.b0[n] = parameters[n + ann.num_neurons_per_layer * ann.dim + offset];
    for (int m = 0; m < ann.num_neurons_per_layer; ++m) {
      int nm = n * ann.num_neurons_per_layer + m;
      ann.w1[nm] = parameters[nm + ann.num_neurons_per_layer * (ann.dim + 1) + offset];
    }
    ann.b1[n] = parameters
      [n + ann.num_neurons_per_layer * (ann.num_neurons_per_layer + (ann.dim + 1)) + offset];
    ann.w2[n] = parameters
      [n + ann.num_neurons_per_layer * (ann.num_neurons_per_layer + (ann.dim + 2)) + offset];
  }
  ann.b2 =
    parameters[ann.num_neurons_per_layer * (ann.num_neurons_per_layer + (ann.dim + 3)) + offset];
}

static __device__ void
apply_ann(const NEP::ANN& ann, float* q, float& energy, float* energy_derivative)
{
  // energy
  float x1[MAX_NUM_NEURONS_PER_LAYER] = {0.0f}; // states of the 1st hidden layer neurons
  float x2[MAX_NUM_NEURONS_PER_LAYER] = {0.0f}; // states of the 2nd hidden layer neurons
  for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
    float w0_times_q = 0.0f;
    for (int d = 0; d < ann.dim; ++d) {
      w0_times_q += ann.w0[n * ann.dim + d] * q[d];
    }
    x1[n] = tanh(w0_times_q - ann.b0[n]);
  }
  for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
    for (int m = 0; m < ann.num_neurons_per_layer; ++m) {
      x2[n] += ann.w1[n * ann.num_neurons_per_layer + m] * x1[m];
    }
    x2[n] = tanh(x2[n] - ann.b1[n]);
  }
  for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
    energy += ann.w2[n] * x2[n];
  }
  energy -= ann.b2;
  // energy gradient (compute it component by component)
  for (int d = 0; d < ann.dim; ++d) {
    float y1[MAX_NUM_NEURONS_PER_LAYER] = {0.0f}; // derivatives of the 1st hidden layer neurons
    float y2[MAX_NUM_NEURONS_PER_LAYER] = {0.0f}; // derivatives of the 2nd hidden layer neurons
    for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
      y1[n] = (1.0f - x1[n] * x1[n]) * ann.w0[n * ann.dim + d];
    }
    for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
      for (int m = 0; m < ann.num_neurons_per_layer; ++m) {
        y2[n] += ann.w1[n * ann.num_neurons_per_layer + m] * y1[m];
      }
      y2[n] *= 1.0f - x2[n] * x2[n];
    }
    for (int n = 0; n < ann.num_neurons_per_layer; ++n) {
      energy_derivative[d] += ann.w2[n] * y2[n];
    }
  }
}

static __device__ void find_fc(float r1, float r2, float pi_factor, float d12, float& fc)
{
  if (d12 < r1) {
    fc = 1.0f;
  } else if (d12 < r2) {
    fc = 0.5f * cos(pi_factor * (d12 - r1)) + 0.5f;
  } else {
    fc = 0.0f;
  }
}

static __device__ void
find_fc_and_fcp(float r1, float r2, float pi_factor, float d12, float& fc, float& fcp)
{
  if (d12 < r1) {
    fc = 1.0f;
    fcp = 0.0f;
  } else if (d12 < r2) {
    fc = 0.5f * cos(pi_factor * (d12 - r1)) + 0.5f;
    fcp = -sin(pi_factor * (d12 - r1)) * pi_factor * 0.5f;
  } else {
    fc = 0.0f;
    fcp = 0.0f;
  }
}

static __global__ void find_force_2body(
  int N,
  int* Na,
  int* Na_sum,
  int* g_NN2b,
  int* g_NL2b,
  int* g_type,
  NEP::Para2B para2b,
  NEP::ANN ann2b,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_virial,
  float* g_pe)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = g_box + SIZE_BOX_AND_INVERSE_BOX * blockIdx.x;
    int neighbor_number = g_NN2b[n1];
    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];
    float pe = 0.0f;
    float fx = 0.0f;
    float fy = 0.0f;
    float fz = 0.0f;
    float virial_xx = 0.0f;
    float virial_yy = 0.0f;
    float virial_zz = 0.0f;
    float virial_xy = 0.0f;
    float virial_yz = 0.0f;
    float virial_zx = 0.0f;
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_NL2b[n1 + N * i1];
      float x12 = g_x[n2] - x1;
      float y12 = g_y[n2] - y1;
      float z12 = g_z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      float p2 = 0.0f, f2[1] = {0.0f};
      float q[1] = {d12 * para2b.r2inv};
      apply_ann(ann2b, q, p2, f2);
      f2[0] *= para2b.r2inv;
      float fc, fcp;
      find_fc_and_fcp(para2b.r1, para2b.r2, para2b.pi_factor, d12, fc, fcp);
      p2 *= fc;
      f2[0] = (f2[0] * fc + p2 * fcp) / d12;
      fx += x12 * f2[0];
      fy += y12 * f2[0];
      fz += z12 * f2[0];
      virial_xx -= x12 * x12 * f2[0] * 0.5f;
      virial_yy -= y12 * y12 * f2[0] * 0.5f;
      virial_zz -= z12 * z12 * f2[0] * 0.5f;
      virial_xy -= x12 * y12 * f2[0] * 0.5f;
      virial_yz -= y12 * z12 * f2[0] * 0.5f;
      virial_zx -= z12 * x12 * f2[0] * 0.5f;
      pe += p2 * 0.5f;
    }
    g_fx[n1] = fx;
    g_fy[n1] = fy;
    g_fz[n1] = fz;
    g_virial[n1 + N * 0] = virial_xx;
    g_virial[n1 + N * 1] = virial_yy;
    g_virial[n1 + N * 2] = virial_zz;
    g_virial[n1 + N * 3] = virial_xy;
    g_virial[n1 + N * 4] = virial_yz;
    g_virial[n1 + N * 5] = virial_zx;
    g_pe[n1] = pe;
  }
}

static __global__ void find_neighbor_list_3body(
  int N,
  int* Na,
  int* Na_sum,
  int* g_NN2b,
  int* g_NL2b,
  NEP::Para3B para3b,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  int* g_NN3b,
  int* g_NL3b)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = g_box + SIZE_BOX_AND_INVERSE_BOX * blockIdx.x;
    int neighbor_number = g_NN2b[n1];
    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];
    int count = 0;
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_NL2b[n1 + N * i1];
      float x12 = g_x[n2] - x1;
      float y12 = g_y[n2] - y1;
      float z12 = g_z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);
      float d12sq = x12 * x12 + y12 * y12 + z12 * z12;
      if (d12sq < para3b.r2 * para3b.r2) {
        g_NL3b[n1 + N * (count++)] = n2;
      }
    }
    g_NN3b[n1] = count;
  }
}

static __global__ void find_partial_force_3body(
  int N,
  int* Na,
  int* Na_sum,
  int* g_neighbor_number,
  int* g_neighbor_list,
  int* g_type,
  NEP::Para3B para3b,
  NEP::ANN ann3b,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  float* g_potential,
  float* g_f12x,
  float* g_f12y,
  float* g_f12z)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = g_box + SIZE_BOX_AND_INVERSE_BOX * blockIdx.x;
    int neighbor_number = g_neighbor_number[n1];
    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];
    float pot_energy = 0.0f;
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_neighbor_list[index];
      float x12 = g_x[n2] - x1;
      float y12 = g_y[n2] - y1;
      float z12 = g_z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      float d12inv = 1.0f / d12;
      float fc12, fcp12;
      find_fc_and_fcp(para3b.r1, para3b.r2, para3b.pi_factor, d12, fc12, fcp12);
      float p12 = 0.0f, f12[3] = {0.0f, 0.0f, 0.0f};
      for (int i2 = 0; i2 < neighbor_number; ++i2) {
        int n3 = g_neighbor_list[n1 + N * i2];
        if (n3 == n2) {
          continue;
        }
        float x13 = g_x[n3] - x1;
        float y13 = g_y[n3] - y1;
        float z13 = g_z[n3] - z1;
        dev_apply_mic(h, x13, y13, z13);
        float d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
        float fc13;
        find_fc(para3b.r1, para3b.r2, para3b.pi_factor, d13, fc13);
        float x23 = x13 - x12;
        float y23 = y13 - y12;
        float z23 = z13 - z12;
        float d23 = sqrt(x23 * x23 + y23 * y23 + z23 * z23);
        float d23inv = 1.0f / d23;
        float q[3] = {d12 + d13, (d12 - d13) * (d12 - d13), d23};
        float p123 = 0.0f, f123[3] = {0.0f};
        apply_ann(ann3b, q, p123, f123);
        p12 += p123 * fc12 * fc13;
        float tmp = p123 * fcp12 * fc13 + (f123[0] + f123[1] * (d12 - d13) * 2.0f) * fc12 * fc13;
        f12[0] += 2.0f * (tmp * x12 * d12inv - f123[2] * fc12 * fc13 * x23 * d23inv);
        f12[1] += 2.0f * (tmp * y12 * d12inv - f123[2] * fc12 * fc13 * y23 * d23inv);
        f12[2] += 2.0f * (tmp * z12 * d12inv - f123[2] * fc12 * fc13 * z23 * d23inv);
      }
      pot_energy += p12;
      g_f12x[index] = f12[0];
      g_f12y[index] = f12[1];
      g_f12z[index] = f12[2];
    }
    g_potential[n1] += pot_energy;
  }
}

static __global__ void find_force_3body_or_manybody(
  int N,
  int* Na,
  int* Na_sum,
  int* g_neighbor_number,
  int* g_neighbor_list,
  const float* __restrict__ g_f12x,
  const float* __restrict__ g_f12y,
  const float* __restrict__ g_f12z,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_virial)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_virial_xx = 0.0f;
    float s_virial_yy = 0.0f;
    float s_virial_zz = 0.0f;
    float s_virial_xy = 0.0f;
    float s_virial_yz = 0.0f;
    float s_virial_zx = 0.0f;
    const float* __restrict__ h = g_box + SIZE_BOX_AND_INVERSE_BOX * blockIdx.x;
    int neighbor_number = g_neighbor_number[n1];
    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_neighbor_list[index];
      int neighbor_number_2 = g_neighbor_number[n2];
      float x12 = g_x[n2] - x1;
      float y12 = g_y[n2] - y1;
      float z12 = g_z[n2] - z1;
      dev_apply_mic(h, x12, y12, z12);
      float f12x = g_f12x[index];
      float f12y = g_f12y[index];
      float f12z = g_f12z[index];
      int offset = 0;
      for (int k = 0; k < neighbor_number_2; ++k) {
        if (n1 == g_neighbor_list[n2 + N * k]) {
          offset = k;
          break;
        }
      }
      index = offset * N + n2;
      float f21x = g_f12x[index];
      float f21y = g_f12y[index];
      float f21z = g_f12z[index];
      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;
      s_virial_xx += x12 * f21x;
      s_virial_yy += y12 * f21y;
      s_virial_zz += z12 * f21z;
      s_virial_xy += x12 * f21y;
      s_virial_yz += y12 * f21z;
      s_virial_zx += z12 * f21x;
    }
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    g_virial[n1] += s_virial_xx;
    g_virial[n1 + N] += s_virial_yy;
    g_virial[n1 + N * 2] += s_virial_zz;
    g_virial[n1 + N * 3] += s_virial_xy;
    g_virial[n1 + N * 4] += s_virial_yz;
    g_virial[n1 + N * 5] += s_virial_zx;
  }
}

static __device__ void find_fn(const int n, const float delta_r, const int d12, float& fn)
{
  float tmp = d12 - n * delta_r;
  fn = exp(-tmp * tmp);
}

static __device__ void
find_fn_and_fnp(const int n, const float delta_r, const int d12, float& fn, float& fnp)
{
  float tmp = d12 - n * delta_r;
  fn = exp(-tmp * tmp);
  fnp = -2.0f * tmp * fn;
}

static __global__ void find_energy_manybody(
  int N,
  int* Na,
  int* Na_sum,
  int* g_NN,
  int* g_NL,
  int* g_type,
  NEP::ParaMB paramb,
  NEP::ANN annmb,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  float* g_pe,
  float* g_Fp,
  float* g_sum_fxyz)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = g_box + SIZE_BOX_AND_INVERSE_BOX * blockIdx.x;
    int neighbor_number = g_NN[n1];
    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];
    float q[MAX_DIM] = {0.0f};
    for (int n = 0; n <= paramb.n_max; ++n) {
      float sum_xyz[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < neighbor_number; ++i1) {
        int n2 = g_NL[n1 + N * i1];
        float x12 = g_x[n2] - x1;
        float y12 = g_y[n2] - y1;
        float z12 = g_z[n2] - z1;
        dev_apply_mic(h, x12, y12, z12);
        float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
        float fc12;
        find_fc(paramb.r1, paramb.r2, paramb.pi_factor, d12, fc12);
        float fn;
        find_fn(n, paramb.delta_r, d12, fn);
        fn *= fc12;
        float d12inv = 1.0f / d12;
        x12 *= d12inv;
        y12 *= d12inv;
        z12 *= d12inv;
        sum_xyz[0] += fn;
        sum_xyz[1] += x12 * fn;
        sum_xyz[2] += y12 * fn;
        sum_xyz[3] += z12 * fn;
        sum_xyz[4] += x12 * x12 * fn;
        sum_xyz[5] += y12 * y12 * fn;
        sum_xyz[6] += z12 * z12 * fn;
        sum_xyz[7] += x12 * y12 * fn;
        sum_xyz[8] += x12 * z12 * fn;
        sum_xyz[9] += y12 * z12 * fn;
      }
      q[n * 3 + 0] = sum_xyz[0] * sum_xyz[0];
      q[n * 3 + 1] = sum_xyz[1] * sum_xyz[1] + sum_xyz[2] * sum_xyz[2] + sum_xyz[3] * sum_xyz[3];
      q[n * 3 + 2] = sum_xyz[7] * sum_xyz[7] + sum_xyz[8] * sum_xyz[8] + sum_xyz[9] * sum_xyz[9];
      q[n * 3 + 2] *= 2.0f;
      q[n * 3 + 2] += sum_xyz[4] * sum_xyz[4] + sum_xyz[5] * sum_xyz[5] + sum_xyz[6] * sum_xyz[6];
      for (int abc = 0; abc < NUM_OF_ABC; ++abc) {
        g_sum_fxyz[(n * NUM_OF_ABC + abc) * N + n1] = sum_xyz[abc];
      }
    }
    float F = 0.0f, Fp[MAX_DIM] = {0.0f};
    apply_ann(annmb, q, F, Fp);
    g_pe[n1] += F;
    for (int d = 0; d < annmb.dim; ++d) {
      g_Fp[d * N + n1] = Fp[d];
    }
  }
}

static __global__ void find_partial_force_manybody(
  int N,
  int* Na,
  int* Na_sum,
  int* g_NN,
  int* g_NL,
  int* g_type,
  NEP::ParaMB paramb,
  NEP::ANN annmb,
  const float* __restrict__ g_x,
  const float* __restrict__ g_y,
  const float* __restrict__ g_z,
  const float* __restrict__ g_box,
  const float* __restrict__ g_Fp,
  const float* __restrict__ g_sum_fxyz,
  float* g_f12x,
  float* g_f12y,
  float* g_f12z)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    const float* __restrict__ h = g_box + SIZE_BOX_AND_INVERSE_BOX * blockIdx.x;
    int neighbor_number = g_NN[n1];
    float x1 = g_x[n1];
    float y1 = g_y[n1];
    float z1 = g_z[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL[index];
      float r12[3] = {g_x[n2] - x1, g_y[n2] - y1, g_z[n2] - z1};
      dev_apply_mic(h, r12[0], r12[1], r12[2]);
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float fc12, fcp12;
      find_fc_and_fcp(paramb.r1, paramb.r2, paramb.pi_factor, d12, fc12, fcp12);
      float d12inv = 1.0f / d12;
      float f12[3] = {0.0f};
      for (int n = 0; n <= paramb.n_max; ++n) {
        float fn;
        float fnp;
        find_fn_and_fnp(n, paramb.delta_r, d12, fn, fnp);
        // l=0
        float fn0 = fn * fc12;
        float fn0p = fnp * fc12 + fn * fcp12;
        float Fp0 = g_Fp[(n * 3 + 0) * N + n1];
        float sum_f0 = g_sum_fxyz[(n * NUM_OF_ABC + 0) * N + n1];
        float tmp = Fp0 * sum_f0 * fn0p * d12inv;
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp * r12[d];
        }
        // l=1
        float fn1 = fn0 * d12inv;
        float fn1p = fn0p * d12inv - fn0 * d12inv * d12inv;
        float Fp1 = g_Fp[(n * 3 + 1) * N + n1];
        float sum_f1[3] = {
          g_sum_fxyz[(n * NUM_OF_ABC + 1) * N + n1], g_sum_fxyz[(n * NUM_OF_ABC + 2) * N + n1],
          g_sum_fxyz[(n * NUM_OF_ABC + 3) * N + n1]};
        float tmp1 =
          Fp1 * fn1p * (sum_f1[0] * r12[0] + sum_f1[1] * r12[1] + sum_f1[2] * r12[2]) * d12inv;
        float tmp2 = Fp1 * fn1;
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp1 * r12[d] + tmp2 * sum_f1[d];
        }
        // l=2
        float fn2 = fn1 * d12inv;
        float fn2p = fn1p * d12inv - fn1 * d12inv * d12inv;
        float Fp2 = g_Fp[(n * 3 + 2) * N + n1];
        float sum_f2[6] = {
          g_sum_fxyz[(n * NUM_OF_ABC + 4) * N + n1], g_sum_fxyz[(n * NUM_OF_ABC + 5) * N + n1],
          g_sum_fxyz[(n * NUM_OF_ABC + 6) * N + n1], g_sum_fxyz[(n * NUM_OF_ABC + 7) * N + n1],
          g_sum_fxyz[(n * NUM_OF_ABC + 8) * N + n1], g_sum_fxyz[(n * NUM_OF_ABC + 9) * N + n1]};
        tmp1 = Fp2 * fn2p *
               (sum_f2[0] * r12[0] * r12[0] + sum_f2[1] * r12[1] * r12[1] +
                sum_f2[2] * r12[2] * r12[2] + 2.0f * sum_f2[3] * r12[0] * r12[1] +
                2.0f * sum_f2[4] * r12[0] * r12[2] + 2.0f * sum_f2[5] * r12[1] * r12[2]) *
               d12inv;
        tmp2 = 2.0f * Fp2 * fn2;
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp1 * r12[d] + tmp2 * sum_f2[d] * r12[d];
        }
        f12[0] += tmp2 * (sum_f2[3] * r12[1] + sum_f2[4] * r12[2]);
        f12[1] += tmp2 * (sum_f2[3] * r12[0] + sum_f2[5] * r12[2]);
        f12[2] += tmp2 * (sum_f2[4] * r12[0] + sum_f2[5] * r12[1]);
      }
      g_f12x[index] = f12[0] * 2.0f;
      g_f12y[index] = f12[1] * 2.0f;
      g_f12z[index] = f12[2] * 2.0f;
    }
  }
}

static __global__ void
initialize_properties(int N, float* g_pe, float* g_fx, float* g_fy, float* g_fz, float* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (n1 < N) {
    g_pe[n1] = 0.0f;
    g_fx[n1] = 0.0f;
    g_fy[n1] = 0.0f;
    g_fz[n1] = 0.0f;
    g_virial[n1 + 0 * N] = 0.0f;
    g_virial[n1 + 1 * N] = 0.0f;
    g_virial[n1 + 2 * N] = 0.0f;
    g_virial[n1 + 3 * N] = 0.0f;
    g_virial[n1 + 4 * N] = 0.0f;
    g_virial[n1 + 5 * N] = 0.0f;
  }
}

void NEP::find_force(
  int Nc,
  int N,
  int* Na,
  int* Na_sum,
  int max_Na,
  int* type,
  float* h,
  Neighbor* neighbor,
  float* r,
  GPU_Vector<float>& f,
  GPU_Vector<float>& virial,
  GPU_Vector<float>& pe)
{
  if (ann2b.num_neurons_per_layer > 0) {
    find_force_2body<<<Nc, max_Na>>>(
      N, Na, Na_sum, neighbor->NN, neighbor->NL, type, para2b, ann2b, r, r + N, r + N * 2, h,
      f.data(), f.data() + N, f.data() + N * 2, virial.data(), pe.data());
    CUDA_CHECK_KERNEL
  } else {
    initialize_properties<<<(N - 1) / 64 + 1, 64>>>(
      N, pe.data(), f.data(), f.data() + N, f.data() + N * 2, virial.data());
    CUDA_CHECK_KERNEL
  }
  if (ann3b.num_neurons_per_layer > 0) {
    find_neighbor_list_3body<<<Nc, max_Na>>>(
      N, Na, Na_sum, neighbor->NN, neighbor->NL, para3b, r, r + N, r + N * 2, h,
      nep_data.NN3b.data(), nep_data.NL3b.data());
    CUDA_CHECK_KERNEL
    find_partial_force_3body<<<Nc, max_Na>>>(
      N, Na, Na_sum, nep_data.NN3b.data(), nep_data.NL3b.data(), type, para3b, ann3b, r, r + N,
      r + N * 2, h, pe.data(), nep_data.f12x.data(), nep_data.f12y.data(), nep_data.f12z.data());
    CUDA_CHECK_KERNEL
    find_force_3body_or_manybody<<<Nc, max_Na>>>(
      N, Na, Na_sum, nep_data.NN3b.data(), nep_data.NL3b.data(), nep_data.f12x.data(),
      nep_data.f12y.data(), nep_data.f12z.data(), r, r + N, r + N * 2, h, f.data(), f.data() + N,
      f.data() + N * 2, virial.data());
    CUDA_CHECK_KERNEL
  }
  if (annmb.num_neurons_per_layer > 0) {
    find_energy_manybody<<<Nc, max_Na>>>(
      N, Na, Na_sum, neighbor->NN, neighbor->NL, type, paramb, annmb, r, r + N, r + N * 2, h,
      pe.data(), nep_data.Fp.data(), nep_data.sum_fxyz.data());
    CUDA_CHECK_KERNEL
    find_partial_force_manybody<<<Nc, max_Na>>>(
      N, Na, Na_sum, neighbor->NN, neighbor->NL, type, paramb, annmb, r, r + N, r + N * 2, h,
      nep_data.Fp.data(), nep_data.sum_fxyz.data(), nep_data.f12x.data(), nep_data.f12y.data(),
      nep_data.f12z.data());
    CUDA_CHECK_KERNEL
    find_force_3body_or_manybody<<<Nc, max_Na>>>(
      N, Na, Na_sum, neighbor->NN, neighbor->NL, nep_data.f12x.data(), nep_data.f12y.data(),
      nep_data.f12z.data(), r, r + N, r + N * 2, h, f.data(), f.data() + N, f.data() + N * 2,
      virial.data());
    CUDA_CHECK_KERNEL
  }
}
