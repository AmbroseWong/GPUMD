#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The neuroevolution potential (NEP)
Ref: Zheyong Fan et al., Neuroevolution machine learning potentials:
Combining high accuracy and low cost in atomistic simulations and application to
heat transport, Phys. Rev. B. 104, 104309 (2021).
------------------------------------------------------------------------------*/

#include "dataset.cuh"
#include "mic.cuh"
#include "nep.cuh"
#include "parameters.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_vector.cuh"
#include "utilities/nep_utilities.cuh"

static __global__ void find_descriptors_radial(
  const int N,
  const int* Na,
  const int* Na_sum,
  const int* g_NN,
  const int* g_NL,
  const NEP2::ParaMB paramb,
  const NEP2::ANN annmb,
  const int* __restrict__ g_type,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  float* g_descriptors)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    int t1 = g_type[n1];
    int neighbor_number = g_NN[n1];
    float q[MAX_DIM] = {0.0f};
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = n1 + N * i1;
      int n2 = g_NL[index];
      float x12 = g_x12[index];
      float y12 = g_y12[index];
      float z12 = g_z12[index];
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      float fc12;
      find_fc(paramb.rc_radial, paramb.rcinv_radial, d12, fc12);
      int t2 = g_type[n2];
      float fn12[MAX_NUM_N];
      find_fn(paramb.n_max_radial, paramb.rcinv_radial, d12, fc12, fn12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float c = (paramb.num_types == 1)
                    ? 1.0f
                    : annmb.c[(n * paramb.num_types + t1) * paramb.num_types + t2];
        q[n] += fn12[n] * c;
      }
    }
    for (int n = 0; n <= paramb.n_max_radial; ++n) {
      g_descriptors[n1 + n * N] = q[n];
    }
  }
}

static __global__ void find_descriptors_angular(
  const int N,
  const int* Na,
  const int* Na_sum,
  const int* g_NN,
  const int* g_NL,
  NEP2::ParaMB paramb,
  const NEP2::ANN annmb,
  const int* __restrict__ g_type,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  float* g_descriptors,
  float* g_sum_fxyz)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  int n1 = N1 + threadIdx.x;
  if (n1 < N2) {
    int t1 = g_type[n1];
    int neighbor_number = g_NN[n1];
    float q[MAX_DIM] = {0.0f};

    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      float s[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < neighbor_number; ++i1) {
        int index = n1 + N * i1;
        int n2 = g_NL[n1 + N * i1];
        float x12 = g_x12[index];
        float y12 = g_y12[index];
        float z12 = g_z12[index];
        float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
        float fc12;
        find_fc(paramb.rc_angular, paramb.rcinv_angular, d12, fc12);
        int t2 = g_type[n2];
        float fn;
        find_fn(n, paramb.rcinv_angular, d12, fc12, fn);
        fn *=
          (paramb.num_types == 1)
            ? 1.0f
            : annmb
                .c[((paramb.n_max_radial + 1 + n) * paramb.num_types + t1) * paramb.num_types + t2];
        accumulate_s(d12, x12, y12, z12, fn, s);
      }
      find_q(paramb.n_max_angular + 1, n, s, q);
      for (int abc = 0; abc < NUM_OF_ABC; ++abc) {
        g_sum_fxyz[(n * NUM_OF_ABC + abc) * N + n1] = s[abc] * YLM[abc];
      }
    }

    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      for (int l = 0; l < paramb.L_max; ++l) {
        int ln = l * (paramb.n_max_angular + 1) + n;
        g_descriptors[n1 + ((paramb.n_max_radial + 1) + ln) * N] = q[ln];
      }
    }
  }
}

static void __global__
find_max_min(const int N, const float* g_q, float* g_q_scaler, float* g_q_min)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  __shared__ float s_max[1024];
  __shared__ float s_min[1024];
  s_max[tid] = -1000000.0f; // a small number
  s_min[tid] = +1000000.0f; // a large number
  const int stride = 1024;
  const int number_of_rounds = (N - 1) / stride + 1;
  for (int round = 0; round < number_of_rounds; ++round) {
    const int n = round * stride + tid;
    if (n < N) {
      const int m = n + N * bid;
      float q = g_q[m];
      if (q > s_max[tid]) {
        s_max[tid] = q;
      }
      if (q < s_min[tid]) {
        s_min[tid] = q;
      }
    }
  }
  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      if (s_max[tid] < s_max[tid + offset]) {
        s_max[tid] = s_max[tid + offset];
      }
      if (s_min[tid] > s_min[tid + offset]) {
        s_min[tid] = s_min[tid + offset];
      }
    }
    __syncthreads();
  }
  if (tid == 0) {
    g_q_scaler[bid] = 1.0f / (s_max[0] - s_min[0]);
    g_q_min[bid] = s_min[0];
  }
}

static void __global__ normalize_descriptors(
  NEP2::ANN annmb, const int N, const float* g_q_scaler, const float* g_q_min, float* g_q)
{
  int n1 = blockDim.x * blockIdx.x + threadIdx.x;
  if (n1 < N) {
    for (int d = 0; d < annmb.dim; ++d) {
      g_q[n1 + d * N] = (g_q[n1 + d * N] - g_q_min[d]) * g_q_scaler[d];
    }
  }
}

NEP2::NEP2(char* input_dir, Parameters& para, Dataset& dataset)
{
  paramb.rc_radial = para.rc_radial;
  paramb.rcinv_radial = 1.0f / paramb.rc_radial;
  paramb.rc_angular = para.rc_angular;
  paramb.rcinv_angular = 1.0f / paramb.rc_angular;
  annmb.dim = (para.n_max_radial + 1) + (para.n_max_angular + 1) * para.L_max;
  annmb.num_neurons1 = para.num_neurons1;
  paramb.num_types = dataset.num_types;
  annmb.num_para = para.number_of_variables;
  paramb.n_max_radial = para.n_max_radial;
  paramb.n_max_angular = para.n_max_angular;
  paramb.L_max = para.L_max;
  nep_data.f12x.resize(dataset.N * dataset.max_NN_angular);
  nep_data.f12y.resize(dataset.N * dataset.max_NN_angular);
  nep_data.f12z.resize(dataset.N * dataset.max_NN_angular);
  nep_data.descriptors.resize(dataset.N * annmb.dim);
  nep_data.Fp.resize(dataset.N * annmb.dim);
  nep_data.sum_fxyz.resize(dataset.N * (paramb.n_max_angular + 1) * NUM_OF_ABC);

  if (paramb.num_types == 1) {
    // use radial neighbor list
    find_descriptors_radial<<<dataset.Nc, dataset.max_Na>>>(
      dataset.N, dataset.Na.data(), dataset.Na_sum.data(), dataset.NN_radial.data(),
      dataset.NL_radial.data(), paramb, annmb, dataset.type.data(), dataset.x12_radial.data(),
      dataset.y12_radial.data(), dataset.z12_radial.data(), nep_data.descriptors.data());
    CUDA_CHECK_KERNEL

    // use angular neighbor list
    find_descriptors_angular<<<dataset.Nc, dataset.max_Na>>>(
      dataset.N, dataset.Na.data(), dataset.Na_sum.data(), dataset.NN_angular.data(),
      dataset.NL_angular.data(), paramb, annmb, dataset.type.data(), dataset.x12_angular.data(),
      dataset.y12_angular.data(), dataset.z12_angular.data(), nep_data.descriptors.data(),
      nep_data.sum_fxyz.data());
    CUDA_CHECK_KERNEL

    find_max_min<<<annmb.dim, 1024>>>(
      dataset.N, nep_data.descriptors.data(), para.q_scaler.data(), para.q_min.data());
    CUDA_CHECK_KERNEL
    normalize_descriptors<<<(dataset.N - 1) / 64 + 1, 64>>>(
      annmb, dataset.N, para.q_scaler.data(), para.q_min.data(), nep_data.descriptors.data());
    CUDA_CHECK_KERNEL
  }
}

void NEP2::update_potential(const float* parameters, ANN& ann)
{
  ann.w0 = parameters;
  ann.b0 = ann.w0 + ann.num_neurons1 * ann.dim;
  ann.w1 = ann.b0 + ann.num_neurons1;
  ann.b1 = ann.w1 + ann.num_neurons1;
  if (paramb.num_types > 1) {
    ann.c = ann.b1 + 1;
  }
}

static __device__ void
apply_ann_one_layer(const NEP2::ANN& ann, float* q, float& energy, float* energy_derivative)
{
  for (int n = 0; n < ann.num_neurons1; ++n) {
    float w0_times_q = 0.0f;
    for (int d = 0; d < ann.dim; ++d) {
      w0_times_q += ann.w0[n * ann.dim + d] * q[d];
    }
    float x1 = tanh(w0_times_q - ann.b0[n]);
    energy += ann.w1[n] * x1;
    for (int d = 0; d < ann.dim; ++d) {
      float y1 = (1.0f - x1 * x1) * ann.w0[n * ann.dim + d];
      energy_derivative[d] += ann.w1[n] * y1;
    }
  }
  energy -= ann.b1[0];
}

static __global__ void apply_ann(
  const int N,
  const NEP2::ParaMB paramb,
  const NEP2::ANN annmb,
  const float* __restrict__ g_descriptors,
  const float* __restrict__ g_q_scaler,
  float* g_pe,
  float* g_Fp)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    // get descriptors
    float q[MAX_DIM] = {0.0f};
    for (int d = 0; d < annmb.dim; ++d) {
      q[d] = g_descriptors[n1 + d * N];
    }
    // get energy and energy gradient
    float F = 0.0f, Fp[MAX_DIM] = {0.0f};
    apply_ann_one_layer(annmb, q, F, Fp);
    g_pe[n1] = F;
    for (int d = 0; d < annmb.dim; ++d) {
      g_Fp[n1 + d * N] = Fp[d] * g_q_scaler[d];
    }
  }
}

static __global__ void find_force_radial(
  const int N,
  const int* g_NN,
  const int* g_NL,
  const NEP2::ParaMB paramb,
  const NEP2::ANN annmb,
  const int* __restrict__ g_type,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  const float* __restrict__ g_Fp,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_virial)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    int neighbor_number = g_NN[n1];
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_virial_xx = 0.0f;
    float s_virial_yy = 0.0f;
    float s_virial_zz = 0.0f;
    float s_virial_xy = 0.0f;
    float s_virial_yz = 0.0f;
    float s_virial_zx = 0.0f;
    int t1 = g_type[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL[index];
      int t2 = g_type[n2];
      float r12[3] = {g_x12[index], g_y12[index], g_z12[index]};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float fc12, fcp12;
      find_fc_and_fcp(paramb.rc_radial, paramb.rcinv_radial, d12, fc12, fcp12);
      float fn12[MAX_NUM_N];
      float fnp12[MAX_NUM_N];
      find_fn_and_fnp(paramb.n_max_radial, paramb.rcinv_radial, d12, fc12, fcp12, fn12, fnp12);
      float f12[3] = {0.0f};
      float f21[3] = {0.0f};
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float tmp12 = g_Fp[n1 + n * N] * fnp12[n] * d12inv;
        float tmp21 = g_Fp[n2 + n * N] * fnp12[n] * d12inv;
        tmp12 *= (paramb.num_types == 1)
                   ? 1.0f
                   : annmb.c[(n * paramb.num_types + t1) * paramb.num_types + t2];
        tmp21 *= (paramb.num_types == 1)
                   ? 1.0f
                   : annmb.c[(n * paramb.num_types + t2) * paramb.num_types + t1];
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp12 * r12[d];
          f21[d] -= tmp21 * r12[d];
        }
      }
      s_fx += f12[0] - f21[0];
      s_fy += f12[1] - f21[1];
      s_fz += f12[2] - f21[2];
      s_virial_xx += r12[0] * f21[0];
      s_virial_yy += r12[1] * f21[1];
      s_virial_zz += r12[2] * f21[2];
      s_virial_xy += r12[0] * f21[1];
      s_virial_yz += r12[1] * f21[2];
      s_virial_zx += r12[2] * f21[0];
    }
    g_fx[n1] = s_fx;
    g_fy[n1] = s_fy;
    g_fz[n1] = s_fz;
    g_virial[n1] = s_virial_xx;
    g_virial[n1 + N] = s_virial_yy;
    g_virial[n1 + N * 2] = s_virial_zz;
    g_virial[n1 + N * 3] = s_virial_xy;
    g_virial[n1 + N * 4] = s_virial_yz;
    g_virial[n1 + N * 5] = s_virial_zx;
  }
}

static __global__ void find_partial_force_angular(
  const int N,
  const int* g_NN,
  const int* g_NL,
  const NEP2::ParaMB paramb,
  const NEP2::ANN annmb,
  const int* __restrict__ g_type,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  const float* __restrict__ g_Fp,
  const float* __restrict__ g_sum_fxyz,
  float* g_f12x,
  float* g_f12y,
  float* g_f12z)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    float Fp[MAX_DIM_ANGULAR] = {0.0f};
    float sum_fxyz[NUM_OF_ABC * MAX_NUM_N];
    for (int d = 0; d < (paramb.n_max_angular + 1) * paramb.L_max; ++d) {
      Fp[d] = g_Fp[(paramb.n_max_radial + 1 + d) * N + n1];
    }
    for (int d = 0; d < (paramb.n_max_angular + 1) * NUM_OF_ABC; ++d) {
      sum_fxyz[d] = g_sum_fxyz[d * N + n1];
    }
    int neighbor_number = g_NN[n1];
    int t1 = g_type[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL[index];
      float r12[3] = {g_x12[index], g_y12[index], g_z12[index]};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float fc12, fcp12;
      find_fc_and_fcp(paramb.rc_angular, paramb.rcinv_angular, d12, fc12, fcp12);
      int t2 = g_type[n2];
      float f12[3] = {0.0f};
      for (int n = 0; n <= paramb.n_max_angular; ++n) {
        float fn;
        float fnp;
        find_fn_and_fnp(n, paramb.rcinv_angular, d12, fc12, fcp12, fn, fnp);
        const float c =
          (paramb.num_types == 1)
            ? 1.0f
            : annmb
                .c[((paramb.n_max_radial + 1 + n) * paramb.num_types + t1) * paramb.num_types + t2];
        fn *= c;
        fnp *= c;
        accumulate_f12(
          n, n1, paramb.n_max_radial + 1, paramb.n_max_angular + 1, d12, r12, fn, fnp, Fp, sum_fxyz,
          f12);
      }

      g_f12x[index] = f12[0] * 2.0f;
      g_f12y[index] = f12[1] * 2.0f;
      g_f12z[index] = f12[2] * 2.0f;
    }
  }
}

static __global__ void find_force_manybody(
  const int N,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const float* __restrict__ g_f12x,
  const float* __restrict__ g_f12y,
  const float* __restrict__ g_f12z,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  float* g_fx,
  float* g_fy,
  float* g_fz,
  float* g_virial)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_virial_xx = 0.0f;
    float s_virial_yy = 0.0f;
    float s_virial_zz = 0.0f;
    float s_virial_xy = 0.0f;
    float s_virial_yz = 0.0f;
    float s_virial_zx = 0.0f;
    int neighbor_number = g_neighbor_number[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_neighbor_list[index];
      int neighbor_number_2 = g_neighbor_number[n2];
      float x12 = g_x12[index];
      float y12 = g_y12[index];
      float z12 = g_z12[index];
      float f12x = g_f12x[index];
      float f12y = g_f12y[index];
      float f12z = g_f12z[index];
      int offset = 0;
      for (int k = 0; k < neighbor_number_2; ++k) {
        if (n1 == g_neighbor_list[n2 + N * k]) {
          offset = k;
          break;
        }
      }
      index = offset * N + n2;
      float f21x = g_f12x[index];
      float f21y = g_f12y[index];
      float f21z = g_f12z[index];
      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;
      s_virial_xx += x12 * f21x;
      s_virial_yy += y12 * f21y;
      s_virial_zz += z12 * f21z;
      s_virial_xy += x12 * f21y;
      s_virial_yz += y12 * f21z;
      s_virial_zx += z12 * f21x;
    }
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    g_virial[n1] += s_virial_xx;
    g_virial[n1 + N] += s_virial_yy;
    g_virial[n1 + N * 2] += s_virial_zz;
    g_virial[n1 + N * 3] += s_virial_xy;
    g_virial[n1 + N * 4] += s_virial_yz;
    g_virial[n1 + N * 5] += s_virial_zx;
  }
}

void NEP2::find_force(Parameters& para, const float* parameters, Dataset& dataset)
{
  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_parameters), parameters, sizeof(float) * annmb.num_para));
  float* address_c_parameters;
  CHECK(hipGetSymbolAddress((void**)&address_c_parameters, c_parameters));
  update_potential(address_c_parameters, annmb);

  const int block_size = 32;
  const int grid_size = (dataset.N - 1) / block_size + 1;

  if (paramb.num_types > 1) {
    // use radial neighbor list
    find_descriptors_radial<<<dataset.Nc, dataset.max_Na>>>(
      dataset.N, dataset.Na.data(), dataset.Na_sum.data(), dataset.NN_radial.data(),
      dataset.NL_radial.data(), paramb, annmb, dataset.type.data(), dataset.x12_radial.data(),
      dataset.y12_radial.data(), dataset.z12_radial.data(), nep_data.descriptors.data());
    CUDA_CHECK_KERNEL

    // use angular neighbor list
    find_descriptors_angular<<<dataset.Nc, dataset.max_Na>>>(
      dataset.N, dataset.Na.data(), dataset.Na_sum.data(), dataset.NN_angular.data(),
      dataset.NL_angular.data(), paramb, annmb, dataset.type.data(), dataset.x12_angular.data(),
      dataset.y12_angular.data(), dataset.z12_angular.data(), nep_data.descriptors.data(),
      nep_data.sum_fxyz.data());
    CUDA_CHECK_KERNEL

    find_max_min<<<annmb.dim, 1024>>>(
      dataset.N, nep_data.descriptors.data(), para.q_scaler.data(), para.q_min.data());
    CUDA_CHECK_KERNEL
    normalize_descriptors<<<(dataset.N - 1) / 64 + 1, 64>>>(
      annmb, dataset.N, para.q_scaler.data(), para.q_min.data(), nep_data.descriptors.data());
    CUDA_CHECK_KERNEL
  }

  apply_ann<<<grid_size, block_size>>>(
    dataset.N, paramb, annmb, nep_data.descriptors.data(), para.q_scaler.data(), dataset.pe.data(),
    nep_data.Fp.data());
  CUDA_CHECK_KERNEL

  // use radial neighbor list
  find_force_radial<<<grid_size, block_size>>>(
    dataset.N, dataset.NN_radial.data(), dataset.NL_radial.data(), paramb, annmb,
    dataset.type.data(), dataset.x12_radial.data(), dataset.y12_radial.data(),
    dataset.z12_radial.data(), nep_data.Fp.data(), dataset.force.data(),
    dataset.force.data() + dataset.N, dataset.force.data() + dataset.N * 2, dataset.virial.data());
  CUDA_CHECK_KERNEL

  // use angular neighbor list
  find_partial_force_angular<<<grid_size, block_size>>>(
    dataset.N, dataset.NN_angular.data(), dataset.NL_angular.data(), paramb, annmb,
    dataset.type.data(), dataset.x12_angular.data(), dataset.y12_angular.data(),
    dataset.z12_angular.data(), nep_data.Fp.data(), nep_data.sum_fxyz.data(), nep_data.f12x.data(),
    nep_data.f12y.data(), nep_data.f12z.data());
  CUDA_CHECK_KERNEL

  // use angular neighbor list
  find_force_manybody<<<grid_size, block_size>>>(
    dataset.N, dataset.NN_angular.data(), dataset.NL_angular.data(), nep_data.f12x.data(),
    nep_data.f12y.data(), nep_data.f12z.data(), dataset.x12_angular.data(),
    dataset.y12_angular.data(), dataset.z12_angular.data(), dataset.force.data(),
    dataset.force.data() + dataset.N, dataset.force.data() + dataset.N * 2, dataset.virial.data());
  CUDA_CHECK_KERNEL
}
